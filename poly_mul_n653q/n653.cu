#include "hip/hip_runtime.h"
#include "n653.h"

#include "iostream"


__device__ int32_t zetas_n653[N_N653 / 7] = {
    10676304, 16761025, 7394872, 16761025, 11121377, 10949150, 7394872, 16761025, 3850100, 7874710, 7862122, 664151, 11121377, 10949150, 7394872, 16761025, 7912042, 14765008, 12738274, 10067325, 16116200, 1189525, 9719104, 16499514, 3850100, 7874710, 7862122, 664151, 11121377, 10949150, 7394872, 16761025, 98522, 4692110, 1288047, 3170012, 16116200, 7862122, 11436071, 12085043, 16212669, 2963283, 9719104, 664151, 10655357, 3627434, 10377718, 13813870, 7912042, 3850100, 1620141, 9051474, 14325862, 12901574, 14765008, 12927053, 15588451, 7583861, 8878623, 290849, 12738274, 7874710, 1572208, 9193292, 4692110, 1126178, 7862122, 10949150, 2963283, 12075328, 3627434, 15650975, 3850100, 11121377, 12901574, 14329275, 7583861, 8673499, 7874710, 5655776, 1126178, 13522707, 12075328, 10649318, 11121377, 7394872, 8673499, 3254446, 13522707, 2596593, 7394872, 16761025, 2596593, 2580465, 2580465, 14180560, 2981589, 15393923, 12412334, 12519308, 14110054, 1590746, 3731610, 15588906, 11857296, 10676304, 864588, 6965437, 11091924, 9920313, 15605542, 15532409, 6191036, 7435780, 600298, 9622180, 9021882, 10966002, 12884094, 1918092, 16647180, 10403679, 10533652, 11973559, 12998707, 1025148, 4383065, 13802193, 9419128, 14587874, 1361894, 3551173, 4768624, 14519764, 9751140, 10588057, 6370682, 12559778, 13982193, 11735271, 14530231, 14772227, 9387670, 11392596, 98522, 1288047, 1189525, 16116200, 11436071, 12097024, 16212669, 9719104, 10283588, 10655357, 10377718, 16499514, 7912042, 1620141, 10485252, 14325862, 14765008, 439146, 15588451, 8878623, 10067325, 12738274, 1572208, 5611087, 4692110, 7862122, 3170012, 2963283, 3627434, 664151, 3850100, 12901574, 9051474, 7583861, 7874710, 290849, 1126178, 12075328, 10949150, 11121377, 8673499, 14329275, 13522707, 7394872, 10649318, 2596593, 2580465, 16761025};

__device__ int32_t zetas_inv_n653[N_N653 / 7] = {
    5384557, 7389483, 2004926, 2246922, 5041882, 2794960, 4217375, 10406471, 6189096, 7026013, 2257389, 12008529, 13225980, 15415259, 2189279, 7358025, 2974960, 12394088, 15752005, 3778446, 4803594, 6243501, 6373474, 129973, 14859061, 3893059, 5811151, 7755271, 7154973, 16176855, 9341373, 10586117, 1244744, 1171611, 6856840, 5685229, 9811716, 15912565, 6100849, 4919857, 1188247, 13045543, 15186407, 2667099, 4257845, 4364819, 1383230, 13795564, 11166066, 15204945, 4038879, 6709828, 7898530, 1188702, 16338007, 2012145, 2451291, 6291901, 15157012, 8865111, 277639, 6399435, 6121796, 6493565, 7058049, 564484, 4680129, 5341082, 660953, 15587628, 15489106, 16678631, 16486304, 8902443, 9193292, 7725679, 3875579, 12927053, 16113002, 13149719, 13813870, 13607141, 8915031, 12085043, 2447878, 8103654, 5655776, 5828003, 4701825, 15650975, 6127835, 9382281, 3254446, 14196688, 14180560, 16761025, 11166066, 16486304, 15204945, 9193292, 6709828, 8902443, 7898530, 290849, 16338007, 7725679, 2012145, 12927053, 6291901, 3875579, 15157012, 9051474, 277639, 16113002, 6399435, 13813870, 6493565, 13149719, 7058049, 664151, 4680129, 13607141, 5341082, 12085043, 15587628, 8915031, 15489106, 3170012, 16486304, 2447878, 8902443, 5655776, 7725679, 8103654, 3875579, 14329275, 16113002, 5828003, 13149719, 15650975, 13607141, 4701825, 8915031, 10949150, 2447878, 6127835, 8103654, 3254446, 5828003, 9382281, 4701825, 10649318, 6127835, 14196688, 9382281, 16761025, 14196688, 14180560, 14180560, 2580465, 277639, 7058049, 15587628, 660953, 6709828, 4038879, 2012145, 8865111, 16113002, 8915031, 8902443, 12927053, 5828003, 5655776, 9382281, 16761025, 16113002, 8915031, 8902443, 12927053, 5828003, 5655776, 9382281, 16761025, 5828003, 5655776, 9382281, 16761025, 9382281, 16761025, 16761025};


    //6层基2,1层基3
__device__ int32_t zetas_n653_v2[N_N653 / 7] = {10676304,16761025, 7394872, 16761025, 11121377, 10949150, 7394872, 16761025, 3850100, 7874710, 7862122, 664151, 11121377, 10949150, 7394872, 16761025, 7912042, 14765008, 12738274, 10067325, 16116200, 1189525, 9719104, 16499514, 3850100, 7874710, 7862122, 664151, 11121377, 10949150, 7394872, 16761025, 16647180, 12998707, 14587874, 9419128, 13982193, 9387670, 14519764, 12559778, 3731610, 864588, 15393923, 1590746, 15532409, 15605542, 9622180, 1918092, 7912042, 14765008, 12738274, 10067325, 16116200, 1189525, 9719104, 16499514, 3850100, 7874710, 7862122, 664151, 11121377, 10949150, 7394872, 16761025, 2981589, 98522, 15393923, 1189525, 12519308, 1288047, 14110054, 16678631, 3731610, 16116200, 15588906, 12097024, 10676304, 11436071, 864588, 660953, 11091924, 16212669, 9920313, 10283588, 15532409, 9719104, 6191036, 564484, 600298, 10655357, 9622180, 16499514, 10966002, 10377718, 12884094, 6121796, 16647180, 7912042, 10403679, 10485252, 11973559, 1620141, 12998707, 8865111, 4383065, 14325862, 13802193, 439146, 14587874, 14765008, 1361894, 2451291, 4768624, 15588451, 14519764, 10067325, 10588057, 8878623, 6370682, 1188702, 13982193, 12738274, 11735271, 5611087, 14772227, 1572208, 9387670, 4038879, 98522, 4692110, 1288047, 3170012, 16116200, 7862122, 11436071, 12085043, 16212669, 2963283, 9719104, 664151, 10655357, 3627434, 10377718, 13813870, 7912042, 3850100, 1620141, 9051474, 14325862, 12901574, 14765008, 12927053, 15588451, 7583861, 8878623, 290849, 12738274, 7874710, 1572208, 9193292, 4692110, 1126178, 7862122, 10949150, 2963283, 12075328, 3627434, 15650975, 3850100, 11121377, 12901574, 14329275, 7583861, 8673499, 7874710, 5655776, 1126178, 13522707, 12075328, 10649318, 11121377, 7394872, 8673499, 3254446, 13522707, 2596593, 7394872, 16761025, 2596593, 2580465, 2580465, 14180560};


__device__ int32_t zetas_inv_n653_v2[N_N653 / 7] = {5384557, 11166066, 7389483, 4038879, 2246922, 15204945, 5041882, 5611087, 4217375, 6709828, 10406471, 1188702, 7026013, 7898530, 2257389, 10067325, 13225980, 16338007, 15415259, 2451291, 7358025, 2012145, 2974960, 439146, 15752005, 6291901, 3778446, 8865111, 6243501, 15157012, 6373474, 10485252, 14859061, 277639, 3893059, 6121796, 7755271, 6399435, 7154973, 16499514, 9341373, 6493565, 10586117, 564484, 1171611, 7058049, 6856840, 10283588, 9811716, 4680129, 15912565, 660953, 4919857, 5341082, 1188247, 12097024, 15186407, 15587628, 2667099, 16678631, 4364819, 15489106, 1383230, 1189525, 11166066, 16486304, 15204945, 9193292, 6709828, 8902443, 7898530, 290849, 16338007, 7725679, 2012145, 12927053, 6291901, 3875579, 15157012, 9051474, 277639, 16113002, 6399435, 13813870, 6493565, 13149719, 7058049, 664151, 4680129, 13607141, 5341082, 12085043, 15587628, 8915031, 15489106, 3170012, 16486304, 2447878, 8902443, 5655776, 7725679, 8103654, 3875579, 14329275, 16113002, 5828003, 13149719, 15650975, 13607141, 4701825, 8915031, 10949150, 2447878, 6127835, 8103654, 3254446, 5828003, 9382281, 4701825, 10649318, 6127835, 14196688, 9382281, 16761025, 14196688, 14180560, 14180560, 2580465, 14859061, 7154973, 1171611, 1244744, 15186407, 1383230, 15912565, 13045543, 4217375, 2257389, 7389483, 2794960, 7358025, 2189279, 3778446, 129973, 277639, 7058049, 15587628, 660953, 6709828, 4038879, 2012145, 8865111, 16113002, 8915031, 8902443, 12927053, 5828003, 5655776, 9382281, 16761025, 277639, 7058049, 15587628, 660953, 6709828, 4038879, 2012145, 8865111, 16113002, 8915031, 8902443, 12927053, 5828003, 5655776, 9382281, 16761025, 16113002, 8915031, 8902443, 12927053, 5828003, 5655776, 9382281, 16761025, 5828003, 5655776, 9382281, 16761025, 9382281, 16761025, 16761025};

__device__ int32_t root3_n653[3] = {16128, 2580465, 14180560};



__device__ int32_t montgomery_reduce_n653_cuda(int64_t a)
{
    int32_t t;
    t = (int32_t)a * NUMBER_ACONFI;//t是低32位置
    t = (a - (int64_t)t * Q_N653) >> 32;
    return t;
}

/*2024-7-11
蒙格玛丽约减法的ptx版本:
*/
__device__ int32_t montgomery_reduce_n653_ptx(int64_t a){
    int64_t res;
    asm(
        "{\n\t"
        //".reg .s64 b;\n\t"
        "mul.lo.s64 %0,%1,%2;\n\t"
        //"and.b64 %0,%0,0xffffffff;\n\t"
        "shr.s64 %0,%0,32;\n\t"
        "mul.lo.s64 %0,%0,%3;\n\t"
        "sub.s64 %0,%1,%0;\n\t"
        "shr.s64 %0,%0,32;\n\t" 
        //"add.s64 %0,%0,%3;\n\t"       
        "}"
    :"=l"(res):"l"(a),"n"(-4610542247161626624),"n"(Q_N653));  //-4610542247161626624 = NUMBER_ACONFI << 32
    return (int32_t)res;
}

__device__ int32_t pseudomersenne_reduce_single_n653_cuda(int32_t a)
{
    int32_t t0, t1;

    t0 = a & 0xffffff;//a的低24bit
    t1 = a >> 24;//右移24bit后的结果
    t0 = (t1 << 6) + t0 - t1 - Q_N653;

    return t0;
}

__global__ void ntt_big_n653_kernel(int32_t a[N_N653]){
    __shared__ int32_t poly_a[N_N653];//shared memory
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x,idx;//idx指向线程tid处理的数
    unsigned int len;
    unsigned int zeta_start=1;

    int32_t t;
    int32_t zeta;


    if(tid < N_N653/2){
        poly_a[tid] = a[tid];
        poly_a[tid+N_N653/2]=a[tid+N_N653/2];

        //第一层基2NTT
        __syncthreads();//控制从全局内存到共享内存的读写是完成的->用于控制线程块内的同步
        len = N_N653 / 2;
        idx = tid;
        t = poly_a[idx + len];//访问bank的一行
        poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
        poly_a[idx] = poly_a[idx] - t;//写入bank的一行


        //四层基2NTT

        for(len = N_N653 / 4 ; len >= 42;len >>= 1 ){
            zeta_start = zeta_start << 1;
            __syncthreads();

            zeta = zetas_n653[zeta_start + ( tid / len )];//不同的tid可能访问相同的idx TODO:这里能不能改成常量内存
            //printf("%d\n",zeta);
            //这里存在分支执行,能不能把不做乘法的放到一个warp里面->如果实现不了，那为了避免空转，都用蒙哥马利乘法
            idx = ( tid / len ) * len * 2 + tid % len;//TODO:这里的取模操作是不是可以优化一下,调用inverse.cpp里面已经实现的代码
            //printf("%d %d\n",tid,idx);
            /*
            if(zeta == 16761025){
                t = poly_a[idx + len];//访问bank的一行
                poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
                poly_a[idx] = poly_a[idx] - t;//写入bank的一行
            }
            else{*/
                t = montgomery_reduce_n653_cuda((int64_t)zeta *poly_a[idx + len]);//访问bank的一行
                poly_a[idx + len]=poly_a[idx] - t;//写入bank的一行
                poly_a[idx] = poly_a[idx] + t;//写入bank的一行
            //}
            
        }

        //一层基3NTT
        int32_t tb, tc, tpho, zeta1, zeta2;
        zeta_start = zeta_start << 1;
        //if (tid == 0 ) printf("%d\n",zeta_start);
        len = 14;

        int32_t a_value;
        __syncthreads();
        if(tid < N_N653 / 3){
            zeta1 = zetas_n653[zeta_start + (tid / len) * 2];//TODO:注意考虑这里是否会发生bank的冲突
            zeta2 = zetas_n653[zeta_start + (tid / len) * 2 + 1];
    
            idx = ( tid / len ) * len * 3 + tid % len;
            
            tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * poly_a[idx + len]);
            tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * poly_a[idx + len + len]);
            tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));
            
            a_value = poly_a[idx];
            poly_a[idx + len + len] = (a_value - tb -tpho);//TODO:这里是否能够将a[idx]存储到寄存器，这样减少访问的开销
            poly_a[idx + len] = (a_value - tc + tpho);
            poly_a[idx] = (a_value + tb + tc);
        }


        //最后一层基2NTT
        len = 7 ;
        __syncthreads();
        zeta_start += 64;//基3NTT共32组，每组使用2个zeta
        //printf("my pid is %d %d %d\n",tid, zeta_start, zeta_start + ( tid / len ));
        zeta = zetas_n653[zeta_start + ( tid / len )];
        idx = ( tid / len ) * len * 2 + tid % len;//TODO:这里的取模操作是不是可以优化一下,调用inverse.cpp里面已经实现的代码
        /*if(zeta == 16761025){
            t = poly_a[idx + len];//访问bank的一行
            poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
            poly_a[idx] = poly_a[idx] - t;//写入bank的一行
        }
        else{*/
            t = montgomery_reduce_n653_cuda((int64_t)zeta *poly_a[idx + len]);
            poly_a[idx + len]=pseudomersenne_reduce_single_n653_cuda( poly_a[idx] - t );//TODO:这里是否会发生bank的冲突
            poly_a[idx] = pseudomersenne_reduce_single_n653_cuda(poly_a[idx] + t);
        //}
        
        
        //将数据写回全局内存
        __syncthreads();
        a[tid] = poly_a[tid];
        a[tid+N_N653/2]=poly_a[tid+N_N653/2];
    }
    
}

/*2024-4-19:
输入:
    c:设备端多项式，用于存储相乘的结果
    a:设备端多项式
    b:设备端多项式
    
处理流程:先将素阶数域上的多项式转化到伪梅森数环上，进行NTT操作后，做点乘，然后再做逆NTT操作，最后返回到素阶数域上
*/
void poly_mul_653_batch_q1(poly *array_c,poly * array_a,poly *array_b,hipStream_t stream,int batch_size){
    //printf("in poly_mul_653_batch_q1\n");
    nttpoly_n653 * array_ntta;
    nttpoly_n653 * array_nttb;
    nttpoly_n653 * array_nttc;


    HANDLE_ERROR(hipMalloc((void **)&array_ntta,sizeof(nttpoly_n653)*batch_size));
    HANDLE_ERROR(hipMalloc((void **)&array_nttb,sizeof(nttpoly_n653)*batch_size));
    HANDLE_ERROR(hipMalloc((void **)&array_nttc,sizeof(nttpoly_n653)*batch_size));

    
    poly_extend_n653_batch<<<batch_size,1,0,stream>>>(array_ntta,array_a);

    poly_extend_n653_batch<<<batch_size,1,0,stream>>>(array_nttb,array_b);

    poly_ntt_big_n653_batch<<<batch_size,N_N653/2,0,stream>>>(array_ntta);
    poly_ntt_small_n653_batch<<<batch_size,N_N653/2,0,stream>>>(array_nttb);

    poly_basemul_n653_batch<<<batch_size,1,0,stream>>>(array_nttc,array_ntta,array_nttb);

    poly_invntt_n653_batch<<<batch_size,N_N653/2,0,stream>>>(array_nttc);

    poly_extract_n653_q1_batch<<<batch_size,1,0,stream>>>(array_c,array_nttc);

    hipStreamSynchronize(stream);//等待kernel完成再释放内存
    
    HANDLE_ERROR(hipFree(array_ntta));
    HANDLE_ERROR(hipFree(array_nttb));
    HANDLE_ERROR(hipFree(array_nttc));


}

/*2024-5-4:
用于测试cuda版本的下述功能为什么错误*/
__device__ void poly_extract_n653_q1_test(poly *b, const nttpoly_n653 *a)
{
    b->coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (a->coeffs[0] + a->coeffs[FPTRU_N])));
    for (int i = 1; i <= FPTRU_N - 2; i++)
        b->coeffs[i] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (a->coeffs[i] + a->coeffs[i + FPTRU_N - 1] + a->coeffs[i + FPTRU_N])));
    b->coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (a->coeffs[FPTRU_N - 1] + a->coeffs[2 * FPTRU_N - 2])));
}

/*2024-4-30:
输入:array_c:全局内存
    array_a:全局内存
    array_b:全局内存
<<<BATCH_SIZE,N_N653/2>>>

增加了进行调试的接口*/
__global__ void poly_mul_653_batch_q1_v2_test(poly *array_c,poly * array_a,poly *array_b,nttpoly_n653 * array_ntta,nttpoly_n653 * array_nttb){
    //共享内存用作需要传递的数据
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    
    poly a;
    poly b;
    int idx = threadIdx.x;
    int32_t zeta;

    //只要一个线程来做
    a = array_a[blockIdx.x];
    b = array_b[blockIdx.x];

    //做poly_extend
    ntta.coeffs[idx*2]=0;
    ntta.coeffs[idx*2+1]=0;

    nttb.coeffs[idx*2]=0;
    nttb.coeffs[idx*2+1]=0;
    
    if(idx*2 < FPTRU_N) {
        ntta.coeffs[idx*2]=a.coeffs[idx*2];
        nttb.coeffs[idx*2]=b.coeffs[idx*2];
    }
    if(idx*2 + 1< FPTRU_N){
        ntta.coeffs[idx*2 + 1]=a.coeffs[idx*2 + 1];
        nttb.coeffs[idx*2 + 1]=b.coeffs[idx*2 + 1];
    }

    __syncthreads();//上述正确性已验证


    ntt_big_n653(ntta.coeffs);
    ntt_small_n653(nttb.coeffs);


    __syncthreads();

    if(idx < N_N653 /14){ //空转的改进一下
        zeta = zetas_n653[N_N653 /14 + idx];
        basemul_n653((&nttc)->coeffs + 14 *idx, (&ntta)->coeffs + 14 * idx, (&nttb)->coeffs + 14 * idx, zeta);

        basemul_n653((&nttc)->coeffs + 14 *idx + 7, (&ntta)->coeffs + 14 * idx + 7, (&nttb)->coeffs + 14 * idx + 7 , -zeta);
    }
    
    __syncthreads();//basemul验证完成


    invntt_tomont_n653(nttc.coeffs);//求逆正确

    /*正确性测试
    array_ntta[blockIdx.x] = nttc; 
    return;*/

    //TODO:为什么这里出现了错误
    //经过验证，应该是代码当下函数关于poly_extract_n653_q1_test的改写出现了问题
    /*
    if(idx == 0){
        //printf("use poly_extract_n653_q1_test\n");
        poly_extract_n653_q1_test(&array_c[blockIdx.x],&nttc);
    }

    return;*/

    //__syncthreads();//会一直变化，可能是线程组织出现了问题,
    /*if(blockIdx.x ==0) {
        printf("start idx=%d\n",idx);
    }*/
    //__syncthreads();
    if(idx == 0){
        //array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
        /*if(blockIdx.x ==0) {
            printf("in[0] idx=%d\n",idx);
        }*/

        array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
    }
    if(idx == FPTRU_N-1){
        //array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
        /*if(blockIdx.x ==0) {
            printf("in[654] idx=%d\n",idx);
        }*/
        array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
    }
    if( 1<= idx){
        if(idx <=FPTRU_N-2 ){
            /*if(blockIdx.x ==0) {
            printf("in[m] idx=%d\n",idx);//这里为什么能输出in[m] idx=665
        }*/
        //array_c[blockIdx.x].coeffs[idx] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N])));

        array_c[blockIdx.x].coeffs[idx] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N])));
        }
    }
    /*else{
        if(blockIdx.x ==0) {
            printf("end idx=%d\n",idx);
        }
    }*/
}

/*2024-5-5:
根绝正确的的poly_mul_653_batch_q1_v2改编得到*/
__device__ poly poly_mul_653_batch_q1_v2_device_test(poly * array_a,poly *array_b){
    //共享内存用作需要传递的数据
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;
    __shared__ poly f_res;

    
    poly a;
    poly b;
    int idx = threadIdx.x;
    int32_t zeta;

    //只要一个线程来做
    a = array_a[blockIdx.x];
    b = array_b[blockIdx.x];

    //做poly_extend
    ntta.coeffs[idx*2]=0;
    ntta.coeffs[idx*2+1]=0;

    nttb.coeffs[idx*2]=0;
    nttb.coeffs[idx*2+1]=0;
    
    if(idx*2 < FPTRU_N) {
        ntta.coeffs[idx*2]=a.coeffs[idx*2];
        nttb.coeffs[idx*2]=b.coeffs[idx*2];
    }
    if(idx*2 + 1< FPTRU_N){
        ntta.coeffs[idx*2 + 1]=a.coeffs[idx*2 + 1];
        nttb.coeffs[idx*2 + 1]=b.coeffs[idx*2 + 1];
    }

    __syncthreads();
    ntt_big_n653(ntta.coeffs);
    ntt_small_n653(nttb.coeffs);


    __syncthreads();

    if(idx < N_N653 /14){ //空转的改进一下
        zeta = zetas_n653[N_N653 /14 + idx];
        basemul_n653((&nttc)->coeffs + 14 *idx, (&ntta)->coeffs + 14 * idx, (&nttb)->coeffs + 14 * idx, zeta);

        basemul_n653((&nttc)->coeffs + 14 *idx + 7, (&ntta)->coeffs + 14 * idx + 7, (&nttb)->coeffs + 14 * idx + 7 , -zeta);
    }
    
    __syncthreads();

    invntt_tomont_n653(nttc.coeffs);

    if(idx == 0){
        f_res.coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
    }
    if(idx == FPTRU_N-1){
        f_res.coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
    }
    if( 1<= idx){
        if(idx <=FPTRU_N-2 ){
            f_res.coeffs[idx] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N])));
        }
    }
    __syncthreads();//TODO:由于是共享内存，会不会提前返回呢？
    return f_res;
}

/*2024-5-4:
根绝正确的的poly_mul_653_batch_q1_v2改编得到*/
__device__ void poly_mul_653_batch_q1_v2_device(poly *array_c,poly * array_a,poly *array_b){
    //共享内存用作需要传递的数据
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    __shared__ poly res;

    
    poly a;
    poly b;
    int idx = threadIdx.x;
    int32_t zeta;

    //只要一个线程来做
    a = array_a[blockIdx.x];
    //__syncthreads();//TODO:防止读写冲突->感觉是没有必要的
    b = array_b[blockIdx.x];
    /*if(idx == 0){//已经验证成功
        for(int i=0;i<FPTRU_N;i++){
            printf("%d,",a.coeffs[i]);
        }
        printf("\n");

        for(int i=0;i<FPTRU_N;i++){
            printf("%d,",b.coeffs[i]);
        }
        printf("\n");
    }*/

    //做poly_extend
    ntta.coeffs[idx*2]=0;
    ntta.coeffs[idx*2+1]=0;

    nttb.coeffs[idx*2]=0;
    nttb.coeffs[idx*2+1]=0;
    
    if(idx*2 < FPTRU_N) {
        ntta.coeffs[idx*2]=a.coeffs[idx*2];
        nttb.coeffs[idx*2]=b.coeffs[idx*2];
    }
    if(idx*2 + 1< FPTRU_N){
        ntta.coeffs[idx*2 + 1]=a.coeffs[idx*2 + 1];
        nttb.coeffs[idx*2 + 1]=b.coeffs[idx*2 + 1];
    }

    __syncthreads();
    ntt_big_n653(ntta.coeffs);
    ntt_small_n653(nttb.coeffs);


    __syncthreads();

    if(idx < N_N653 /14){ //空转的改进一下
        zeta = zetas_n653[N_N653 /14 + idx];
        basemul_n653((&nttc)->coeffs + 14 *idx, (&ntta)->coeffs + 14 * idx, (&nttb)->coeffs + 14 * idx, zeta);

        basemul_n653((&nttc)->coeffs + 14 *idx + 7, (&ntta)->coeffs + 14 * idx + 7, (&nttb)->coeffs + 14 * idx + 7 , -zeta);
    }
    
    __syncthreads();

    invntt_tomont_n653(nttc.coeffs);

    /*if(idx == 0){ //输出查看一下
        printf("invntt\n");
        for(int i=0;i<N_N653;i++){
            printf("%d,",nttc.coeffs[i]);
        }
        printf("\n");
    }*/
    __syncthreads();

    if(idx == 0){
        array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
        //res.coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
    }
    if(idx == FPTRU_N-1){
        array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
        //res.coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));

    }
    if( 1<= idx){
        if(idx <=FPTRU_N-2 ){
            array_c[blockIdx.x].coeffs[idx] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N])));
            //res.coeffs[idx] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N])));
        }
    }
    //__syncthreads();

    /*if(idx == 0){//已经验证成功
        for(int i=0;i<FPTRU_N;i++){
            printf("%d,",res.coeffs[i]);
        }
        printf("\n");
    }*/

    //__syncthreads();//TODO:由于是共享内存，会不会提前返回呢？
}

/*2024-4-24:
输入:array_c:全局内存
    array_a:全局内存
    array_b:全局内存
<<<BATCH_SIZE,N_N653/2>>>

2024-5-4修改:已经验证正确性*/
__global__ void poly_mul_653_batch_q1_v2(poly *array_c,poly * array_a,poly *array_b){
    //共享内存用作需要传递的数据
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    
    poly a;
    poly b;
    int idx = threadIdx.x;
    int32_t zeta;

    //只要一个线程来做
    a = array_a[blockIdx.x];
    b = array_b[blockIdx.x];

    //做poly_extend
    ntta.coeffs[idx*2]=0;
    ntta.coeffs[idx*2+1]=0;

    nttb.coeffs[idx*2]=0;
    nttb.coeffs[idx*2+1]=0;
    
    if(idx*2 < FPTRU_N) {
        ntta.coeffs[idx*2]=a.coeffs[idx*2];
        nttb.coeffs[idx*2]=b.coeffs[idx*2];
    }
    if(idx*2 + 1< FPTRU_N){
        ntta.coeffs[idx*2 + 1]=a.coeffs[idx*2 + 1];
        nttb.coeffs[idx*2 + 1]=b.coeffs[idx*2 + 1];
    }

    __syncthreads();
    ntt_big_n653(ntta.coeffs);
    ntt_small_n653(nttb.coeffs);


    __syncthreads();

    if(idx < N_N653 /14){ //空转的改进一下
        zeta = zetas_n653[N_N653 /14 + idx];
        basemul_n653((&nttc)->coeffs + 14 *idx, (&ntta)->coeffs + 14 * idx, (&nttb)->coeffs + 14 * idx, zeta);

        basemul_n653((&nttc)->coeffs + 14 *idx + 7, (&ntta)->coeffs + 14 * idx + 7, (&nttb)->coeffs + 14 * idx + 7 , -zeta);
    }
    
    __syncthreads();

    invntt_tomont_n653(nttc.coeffs);

    //TODO:这里要不要加上线程同步呢？
    __syncthreads();//TODO:device函数不加会错误

    if(idx == 0){
        array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
    }
    if(idx == FPTRU_N-1){
        array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
    }
    if( 1<= idx){
        if(idx <=FPTRU_N-2 ){
            array_c[blockIdx.x].coeffs[idx] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N])));
        }
    }
    
}

/*2024-5-18:
查看ntt的速度，寻找性能瓶颈*/
__global__ void poly_mul_653_batch_q1_zhc(poly *array_c,poly * array_a,poly *array_b){
    //共享内存用作需要传递的数据
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    
    poly a;
    poly b;
    int idx = threadIdx.x;
    int32_t zeta;

    //只要一个线程来做
    a = array_a[blockIdx.x];
    b = array_b[blockIdx.x];

    //做poly_extend
    
    ntta.coeffs[idx*2]=0;
    ntta.coeffs[idx*2+1]=0;

    nttb.coeffs[idx*2]=0;
    nttb.coeffs[idx*2+1]=0;
    
    if(idx*2 < FPTRU_N) {
        ntta.coeffs[idx*2]=a.coeffs[idx*2];
        nttb.coeffs[idx*2]=b.coeffs[idx*2];
    }
    if(idx*2 + 1< FPTRU_N){
        ntta.coeffs[idx*2 + 1]=a.coeffs[idx*2 + 1];
        nttb.coeffs[idx*2 + 1]=b.coeffs[idx*2 + 1];
    }

    __syncthreads();
    ntt_big_n653_v2(nttc.coeffs);
    /*ntt_small_n653(nttb.coeffs);


    __syncthreads();

    if(idx < N_N653 /14){ //空转的改进一下
        zeta = zetas_n653[N_N653 /14 + idx];
        basemul_n653((&nttc)->coeffs + 14 *idx, (&ntta)->coeffs + 14 * idx, (&nttb)->coeffs + 14 * idx, zeta);

        basemul_n653((&nttc)->coeffs + 14 *idx + 7, (&ntta)->coeffs + 14 * idx + 7, (&nttb)->coeffs + 14 * idx + 7 , -zeta);
    }
    
    __syncthreads();

    invntt_tomont_n653(nttc.coeffs);

    //TODO:这里要不要加上线程同步呢？
    __syncthreads();//TODO:device函数不加会错误

    if(idx == 0){
        array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
    }
    if(idx == FPTRU_N-1){
        array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
    }
    if( 1<= idx){
        if(idx <=FPTRU_N-2 ){
            array_c[blockIdx.x].coeffs[idx] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N])));
        }
    }*/
    

}

/*2024-4-27:
输入:c:共享内存
    a:共享内存
    array_b:全局内存，只含一个多项式

输出:计算fmul*array_b并存储到fmul中*/
__device__ void poly_mul_653_batch_q1_v2_device_v2(poly*array_c, poly* array_a,poly *array_b){
    return;
    /*
    //共享内存用作需要传递的数据
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    
    poly a;
    poly b;
    int idx = threadIdx.x;
    int32_t zeta;

    //只要一个线程来做
    a = array_a[blockIdx.x];
    b = array_b[blockIdx.x];

    //做poly_extend
    ntta.coeffs[idx*2]=0;
    ntta.coeffs[idx*2+1]=0;

    nttb.coeffs[idx*2]=0;
    nttb.coeffs[idx*2+1]=0;
    
    if(idx*2 < FPTRU_N) {
        ntta.coeffs[idx*2]=a.coeffs[idx*2];
        nttb.coeffs[idx*2]=b.coeffs[idx*2];
    }
    if(idx*2 + 1< FPTRU_N){
        ntta.coeffs[idx*2 + 1]=a.coeffs[idx*2 + 1];
        nttb.coeffs[idx*2 + 1]=b.coeffs[idx*2 + 1];
    }

    __syncthreads();
    
    ntt_big_n653(ntta.coeffs);
    ntt_small_n653(nttb.coeffs);


    __syncthreads();

    if(idx < N_N653 /14){ //空转的改进一下
        zeta = zetas_n653[N_N653 /14 + idx];
        basemul_n653((&nttc)->coeffs + 14 *idx, (&ntta)->coeffs + 14 * idx, (&nttb)->coeffs + 14 * idx, zeta);

        basemul_n653((&nttc)->coeffs + 14 *idx + 7, (&ntta)->coeffs + 14 * idx + 7, (&nttb)->coeffs + 14 * idx + 7 , -zeta);
    }
    
    __syncthreads();

    invntt_tomont_n653(nttc.coeffs);*/
    /*
    if(idx == 0){
        array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
    }
    else if(idx == FPTRU_N-1){
        array_c[blockIdx.x].coeffs[FPTRU_N - 1] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2]))) & (FPTRU_Q2 - 1);
    }
    else if(0<idx<FPTRU_N-1){
        array_c[blockIdx.x].coeffs[idx] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N]))) & (FPTRU_Q2 - 1);
    }*/
}
/*
__device__ void poly_mul_653_batch_q1_v2_device(poly array_c, poly array_a,poly *array_b){
    //共享内存用作需要传递的数据
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    
    poly a;
    poly b;
    int idx = threadIdx.x;
    int32_t zeta;

    //只要一个线程来做
    a = array_a;
    b = array_b[blockIdx.x];

    //做poly_extend
    ntta.coeffs[idx*2]=0;
    ntta.coeffs[idx*2+1]=0;

    nttb.coeffs[idx*2]=0;
    nttb.coeffs[idx*2+1]=0;
    
    if(idx*2 < FPTRU_N) {
        ntta.coeffs[idx*2]=a.coeffs[idx*2];
        nttb.coeffs[idx*2]=b.coeffs[idx*2];
    }
    if(idx*2 + 1< FPTRU_N){
        ntta.coeffs[idx*2 + 1]=a.coeffs[idx*2 + 1];
        nttb.coeffs[idx*2 + 1]=b.coeffs[idx*2 + 1];
    }

    __syncthreads();
    
    ntt_big_n653(ntta.coeffs);
    ntt_small_n653(nttb.coeffs);


    __syncthreads();

    if(idx < N_N653 /14){ //空转的改进一下
        zeta = zetas_n653[N_N653 /14 + idx];
        basemul_n653((&nttc)->coeffs + 14 *idx, (&ntta)->coeffs + 14 * idx, (&nttb)->coeffs + 14 * idx, zeta);

        basemul_n653((&nttc)->coeffs + 14 *idx + 7, (&ntta)->coeffs + 14 * idx + 7, (&nttb)->coeffs + 14 * idx + 7 , -zeta);
    }
    
    __syncthreads();

    invntt_tomont_n653(nttc.coeffs);

    if(idx == 0){
        array_c.coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
    }
    else if(idx == FPTRU_N-1){
        array_c.coeffs[FPTRU_N - 1] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2]))) & (FPTRU_Q2 - 1);
    }
    else if(0<idx<FPTRU_N-1){
        array_c.coeffs[idx] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx] + nttc.coeffs[idx + FPTRU_N - 1] + nttc.coeffs[idx + FPTRU_N]))) & (FPTRU_Q2 - 1);
    }
}
*/

/*2024-4-18
输入:array_nttb:设备端多项式数组，用于存储拓展后的结果
    array_a:设备端多项式数组

输出:存储在nttpoly_n653中

线程组织形式:<<<BATCH_SIZE,1>>>

处理流程:进行数据分块，使得每一个poly_extend_n653函数处理一个多项式的拓展
*/
__global__ void poly_extend_n653_batch(nttpoly_n653 * array_nttb,poly * array_a){
    poly_extend_n653(&(array_nttb[blockIdx.x]),&(array_a[blockIdx.x]));
}

/*2024-4-18:
输入:b:设备端多项式,用于存储拓展后的结果
    a:设备端多项式，待处理的多项式

输出:存储在b中

//TODO:这里是否需要变成多个kernel的形式
*/
__device__ void poly_extend_n653(nttpoly_n653 * b,poly * a){
    int i;
    for (i = 0; i < FPTRU_N; i++)
        b->coeffs[i] = a->coeffs[i];
    for (; i < N_N653; i++)
        b->coeffs[i] = 0;
}

/*2024-4-18:
输入:array_a:设备端待进行NTT处理的多项式数组
    
输出:存放在array_a中

线程组织形式:<<<BATCH_SIZE,N_N653/2>>>
*/
__global__ void poly_ntt_big_n653_batch(nttpoly_n653 * array_a){
    ntt_big_n653((array_a[blockIdx.x]).coeffs);//array_a[BATCH_SIZE]
}


/*2024-4-18:
输入:a:设备端待进行NTT处理的多项式，以数组的形式进行表示

输出:存储在输入中

处理流程:N_N653/2个线程协同处理一个大多项式的NTT操作*/
__device__ void ntt_big_n653(int32_t a[N_N653]){
    __shared__ int32_t poly_a[N_N653];//shared memory
    unsigned int tid = threadIdx.x,idx;//idx指向线程tid处理的数
    unsigned int len;
    unsigned int zeta_start=1;

    int32_t t;
    int32_t zeta;


    if(tid < N_N653/2){
        poly_a[tid] = a[tid];
        poly_a[tid+N_N653/2]=a[tid+N_N653/2];

        //第一层基2NTT
        __syncthreads();//控制从全局内存到共享内存的读写是完成的->用于控制线程块内的同步
        len = N_N653 / 2;
        idx = tid;
        t = poly_a[idx + len];//访问bank的一行
        poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
        poly_a[idx] = poly_a[idx] - t;//写入bank的一行


        //四层基2NTT

        for(len = N_N653 / 4 ; len >= 42;len >>= 1 ){
            zeta_start = zeta_start << 1;
            __syncthreads();

            zeta = zetas_n653[zeta_start + ( tid / len )];//不同的tid可能访问相同的idx TODO:这里能不能改成常量内存
            //printf("%d\n",zeta);
            //这里存在分支执行,能不能把不做乘法的放到一个warp里面->如果实现不了，那为了避免空转，都用蒙哥马利乘法
            idx = ( tid / len ) * len * 2 + tid % len;//TODO:这里的取模操作是不是可以优化一下,调用inverse.cpp里面已经实现的代码
            //printf("%d %d\n",tid,idx);
            
            if(zeta == 16761025){
                t = poly_a[idx + len];//访问bank的一行
                poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
                poly_a[idx] = poly_a[idx] - t;//写入bank的一行
            }
            else{
                t = montgomery_reduce_n653_cuda((int64_t)zeta *poly_a[idx + len]);//访问bank的一行
                poly_a[idx + len]=poly_a[idx] - t;//写入bank的一行
                poly_a[idx] = poly_a[idx] + t;//写入bank的一行
            }
            
        }

        //一层基3NTT
        int32_t tb, tc, tpho, zeta1, zeta2;
        zeta_start = zeta_start << 1;
        //if (tid == 0 ) printf("%d\n",zeta_start);
        len = 14;

        int32_t a_value;
        __syncthreads();
        if(tid < N_N653 / 3){
            zeta1 = zetas_n653[zeta_start + (tid / len) * 2];//TODO:注意考虑这里是否会发生bank的冲突
            zeta2 = zetas_n653[zeta_start + (tid / len) * 2 + 1];
    
            idx = ( tid / len ) * len * 3 + tid % len;
            
            tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * poly_a[idx + len]);
            tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * poly_a[idx + len + len]);
            tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));
            
            a_value = poly_a[idx];
            poly_a[idx + len + len] = (a_value - tb -tpho);//TODO:这里是否能够将a[idx]存储到寄存器，这样减少访问的开销
            poly_a[idx + len] = (a_value - tc + tpho);
            poly_a[idx] = (a_value + tb + tc);
        }


        //最后一层基2NTT
        len = 7 ;
        __syncthreads();
        zeta_start += 64;//基3NTT共32组，每组使用2个zeta
        //printf("my pid is %d %d %d\n",tid, zeta_start, zeta_start + ( tid / len ));
        zeta = zetas_n653[zeta_start + ( tid / len )];
        idx = ( tid / len ) * len * 2 + tid % len;//TODO:这里的取模操作是不是可以优化一下,调用inverse.cpp里面已经实现的代码
        if(zeta == 16761025){
            t = poly_a[idx + len];//访问bank的一行
            poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
            poly_a[idx] = poly_a[idx] - t;//写入bank的一行
        }
        else{
            t = montgomery_reduce_n653_cuda((int64_t)zeta *poly_a[idx + len]);
            poly_a[idx + len]=pseudomersenne_reduce_single_n653_cuda( poly_a[idx] - t );//TODO:这里是否会发生bank的冲突
            poly_a[idx] = pseudomersenne_reduce_single_n653_cuda(poly_a[idx] + t);
        }
        
        
        //将数据写回全局内存
        __syncthreads();
        a[tid] = poly_a[tid];
        a[tid+N_N653/2]=poly_a[tid+N_N653/2];
    }
}

__device__ void ntt_big_n653_v2(int32_t poly_a[N_N653]){
    //__shared__ int32_t poly_a[N_N653];//shared memory
    unsigned int tid = threadIdx.x,idx;//idx指向线程tid处理的数
    unsigned int len;
    unsigned int zeta_start=1;

    int32_t t;
    int32_t zeta;


    if(tid < N_N653/2){// 
        /*poly_a[tid] = a[tid];
        poly_a[tid+N_N653/2]=a[tid+N_N653/2];*/

        /*poly_a[tid * 2] = a[tid * 2];
        poly_a[tid * 2 + 1]=a[tid * 2 + 1];*/

        
        //第一层基2NTT
        //__syncthreads();//控制从全局内存到共享内存的读写是完成的->用于控制线程块内的同步
        len = N_N653 / 2;
        idx = tid;
        t = poly_a[idx + len];//访问bank的一行
        poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
        poly_a[idx] = poly_a[idx] - t;//写入bank的一行


        //四层基2NTT

        for(len = N_N653 / 4 ; len >= 42;len >>= 1 ){
            zeta_start = zeta_start << 1;
            __syncthreads();

            zeta = zetas_n653[zeta_start + ( tid / len )];//不同的tid可能访问相同的idx TODO:这里能不能改成常量内存
            //printf("%d\n",zeta);
            //这里存在分支执行,能不能把不做乘法的放到一个warp里面->如果实现不了，那为了避免空转，都用蒙哥马利乘法
            idx = ( tid / len ) * len * 2 + tid % len;//TODO:这里的取模操作是不是可以优化一下,调用inverse.cpp里面已经实现的代码
            //printf("%d %d\n",tid,idx);
            
            if(zeta == 16761025){
                t = poly_a[idx + len];//访问bank的一行
                poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
                poly_a[idx] = poly_a[idx] - t;//写入bank的一行
            }
            else{
                t = montgomery_reduce_n653_cuda((int64_t)zeta *poly_a[idx + len]);//访问bank的一行
                poly_a[idx + len]=poly_a[idx] - t;//写入bank的一行
                poly_a[idx] = poly_a[idx] + t;//写入bank的一行
            }
            
        }

        //一层基3NTT
        int32_t tb, tc, tpho, zeta1, zeta2;
        zeta_start = zeta_start << 1;
        //if (tid == 0 ) printf("%d\n",zeta_start);
        len = 14;

        int32_t a_value;
        __syncthreads();
        if(tid < N_N653 / 3){
            zeta1 = zetas_n653[zeta_start + (tid / len) * 2];//TODO:注意考虑这里是否会发生bank的冲突
            zeta2 = zetas_n653[zeta_start + (tid / len) * 2 + 1];
    
            idx = ( tid / len ) * len * 3 + tid % len;
            
            tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * poly_a[idx + len]);
            tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * poly_a[idx + len + len]);
            tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));
            
            a_value = poly_a[idx];
            poly_a[idx + len + len] = (a_value - tb -tpho);//TODO:这里是否能够将a[idx]存储到寄存器，这样减少访问的开销
            poly_a[idx + len] = (a_value - tc + tpho);
            poly_a[idx] = (a_value + tb + tc);
        }


        //最后一层基2NTT
        len = 7 ;
        __syncthreads();
        zeta_start += 64;//基3NTT共32组，每组使用2个zeta
        //printf("my pid is %d %d %d\n",tid, zeta_start, zeta_start + ( tid / len ));
        zeta = zetas_n653[zeta_start + ( tid / len )];
        idx = ( tid / len ) * len * 2 + tid % len;//TODO:这里的取模操作是不是可以优化一下,调用inverse.cpp里面已经实现的代码
        if(zeta == 16761025){
            t = poly_a[idx + len];//访问bank的一行
            poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
            poly_a[idx] = poly_a[idx] - t;//写入bank的一行
        }
        else{
            t = montgomery_reduce_n653_cuda((int64_t)zeta *poly_a[idx + len]);
            poly_a[idx + len]=pseudomersenne_reduce_single_n653_cuda( poly_a[idx] - t );//TODO:这里是否会发生bank的冲突
            poly_a[idx] = pseudomersenne_reduce_single_n653_cuda(poly_a[idx] + t);
        }
        
        
        //将数据写回全局内存
        //__syncthreads();
        //a[tid] = poly_a[tid];
        //a[tid+N_N653/2]=poly_a[tid+N_N653/2];
    }
}

/*2024-4-18:
输入:array_a:设备端待进行NTT处理的多项式数组
    
输出:存放在array_a中

线程组织形式:<<<BATCH_SIZE,N_N653/2>>>
*/
__global__ void poly_ntt_small_n653_batch(nttpoly_n653 * array_a){
    ntt_small_n653((array_a[blockIdx.x]).coeffs);//array_a[BATCH_SIZE]
}


/*2024-4-18:
输入:a:设备端待进行NTT处理的多项式，以数组的形式进行表示

输出:存储在输入中

处理流程:N_N653/2个线程协同处理一个小多项式的NTT操作*/

__device__ void ntt_small_n653(int32_t a[N_N653]){
    __shared__ int32_t poly_a[N_N653];//shared memory
    unsigned int tid = threadIdx.x,idx;//idx指向线程tid处理的数
    unsigned int len;
    unsigned int zeta_start=1;

    int32_t t;
    int32_t zeta;


    if(tid < N_N653/2){
        poly_a[tid] = a[tid];
        poly_a[tid+N_N653/2]=a[tid+N_N653/2];

        //第一层基2NTT
        __syncthreads();//控制从全局内存到共享内存的读写是完成的->用于控制线程块内的同步
        len = N_N653 / 2;
        idx = tid;
        t = poly_a[idx + len];//访问bank的一行
        poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
        poly_a[idx] = poly_a[idx] - t;//写入bank的一行

        //第二层基2NTT
        __syncthreads();
        len = N_N653 / 4;
        idx = ( tid / len ) * len * 2 + tid % len;
        // tid /len = 0 ,则成乘上FACTOR_CNSHHUI；否则乘上 -1
        //zeta = tid / len - 1;//tid /len =0 时，全1；
        //zeta = (zeta & FACTOR_CNSHHUI) + (zeta ^ (-1)); //TODO:这里的zeta可能会存在问题
        if(tid / len == 0){
            zeta = FACTOR_CNSHHUI;
        }
        else zeta = -1;

        t = poly_a[idx + len] * zeta;
        poly_a[idx + len] = poly_a[idx] - t;
        poly_a[idx] = poly_a[idx] + t;//DEBUG:这里符号出错

        //3层基2NTT
        zeta_start = zeta_start << 1;
        for(len = N_N653 / 8 ; len >= 42;len >>= 1 ){
            zeta_start = zeta_start << 1;
            __syncthreads();

            zeta = zetas_n653[zeta_start + ( tid / len )];//不同的tid可能访问相同的idx TODO:这里能不能改成常量内存
            //printf("%d\n",zeta);
            //这里存在分支执行,能不能把不做乘法的放到一个warp里面->如果实现不了，那为了避免空转，都用蒙哥马利乘法
            idx = ( tid / len ) * len * 2 + tid % len;//TODO:这里的取模操作是不是可以优化一下,调用inverse.cpp里面已经实现的代码
            //printf("%d %d\n",tid,idx);
            
            if(zeta == 16761025){
                t = poly_a[idx + len];//访问bank的一行
                poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
                poly_a[idx] = poly_a[idx] - t;//写入bank的一行
            }
            else{
                t = montgomery_reduce_n653_cuda((int64_t)zeta *poly_a[idx + len]);//访问bank的一行
                poly_a[idx + len]=poly_a[idx] - t;//写入bank的一行
                poly_a[idx] = poly_a[idx] + t;//写入bank的一行
            }  
        }

        //一层基3NTT
        int32_t tb, tc, tpho, zeta1, zeta2;
        zeta_start = zeta_start << 1;
        //if (tid == 0 ) printf("%d\n",zeta_start);
        len = 14;

        int32_t a_value;
        __syncthreads();
        if(tid < N_N653 / 3){
            zeta1 = zetas_n653[zeta_start + (tid / len) * 2];//TODO:注意考虑这里是否会发生bank的冲突
            zeta2 = zetas_n653[zeta_start + (tid / len) * 2 + 1];
    
            idx = ( tid / len ) * len * 3 + tid % len;
            
            tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * poly_a[idx + len]);
            tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * poly_a[idx + len + len]);
            tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));
            
            a_value = poly_a[idx];
            poly_a[idx + len + len] = (a_value - tb -tpho);//TODO:这里是否能够将a[idx]存储到寄存器，这样减少访问的开销
            poly_a[idx + len] = (a_value - tc + tpho);
            poly_a[idx] = (a_value + tb + tc);
        }


        //最后一层基2NTT
        len = 7 ;
        __syncthreads();
        zeta_start += 64;//基3NTT共32组，每组使用2个zeta
        //printf("my pid is %d %d %d\n",tid, zeta_start, zeta_start + ( tid / len ));
        zeta = zetas_n653[zeta_start + ( tid / len )];
        idx = ( tid / len ) * len * 2 + tid % len;//TODO:这里的取模操作是不是可以优化一下,调用inverse.cpp里面已经实现的代码
        if(zeta == 16761025){
            t = poly_a[idx + len];//访问bank的一行
            poly_a[idx + len]=poly_a[idx] + t;//写入bank的一行
            poly_a[idx] = poly_a[idx] - t;//写入bank的一行
        }
        else{
            t = montgomery_reduce_n653_cuda((int64_t)zeta *poly_a[idx + len]);
            poly_a[idx + len]=pseudomersenne_reduce_single_n653_cuda( poly_a[idx] - t );//TODO:这里是否会发生bank的冲突
            poly_a[idx] = pseudomersenne_reduce_single_n653_cuda(poly_a[idx] + t);
        }
        
        
        //将数据写回全局内存
        __syncthreads();
        a[tid] = poly_a[tid];
        a[tid+N_N653/2]=poly_a[tid+N_N653/2];
    }
}

/*2024-4-18:
输入:array_c:设备端多项式数组
    array_a:设备端多项式数组
    array_b:设备端多项式数组
    
输出:封装BATCH_SIZE个多项式的乘法,最终使得array_c=array_a*array_b
*/
__global__ void poly_basemul_n653_batch(nttpoly_n653 *array_c, nttpoly_n653 *array_a, nttpoly_n653 *array_b){
    poly_basemul_n653(&(array_c[blockIdx.x]),&(array_a[blockIdx.x]),&(array_b[blockIdx.x]));
}

__device__ void poly_basemul_n653(nttpoly_n653 *c, nttpoly_n653 *a, nttpoly_n653 *b){
    int i;
    int32_t zeta;
    for (i = 0; i < N_N653 / 14; i++)
    {
        zeta = zetas_n653[N_N653 / 14 + i];//TODO:这里为什么还需要使用zetas？->对应了多项式所在的域->位置是如何确定的
        basemul_n653(c->coeffs + 14 * i, a->coeffs + 14 * i, b->coeffs + 14 * i, zeta);
        basemul_n653(c->coeffs + 14 * i + 7, a->coeffs + 14 * i + 7, b->coeffs + 14 * i + 7, -zeta);
    }
}

#define KARA(a, b, x, y, d) ((montgomery_reduce_n653_cuda((int64_t)(a[x] + a[y]) * (b[x] + b[y])) - d[x] - d[y]))
/*2024-4-7:
输入:c:存储计算结果 c=a*b mod x^7-zeta
    a:待处理的多项式a
    b:待处理的多项式b
输出:存储到c上
处理流程:依据原理c[k]=add(a[i]b[j],i+j=k) + zeta * add(a[i]b[j],i+j=k+7),利用karasuba技巧，先计算a[i]b[i],然后依次处理每一个系数*/
__device__ void basemul_n653(int32_t c[7], const int32_t a[7], const int32_t b[7], const int32_t zeta)
{
    int i;
    int32_t d[7];

    for (i = 0; i < 7; i++)
        d[i] = montgomery_reduce_n653_cuda((int64_t)a[i] * b[i]);

    c[0] = (d[0] + montgomery_reduce_n653_cuda((int64_t)zeta * (KARA(a, b, 1, 6, d) + KARA(a, b, 2, 5, d) + KARA(a, b, 3, 4, d))));//感觉这里使用了karasuba技巧
    c[1] = (KARA(a, b, 0, 1, d) + montgomery_reduce_n653_cuda((int64_t)zeta * (KARA(a, b, 2, 6, d) + KARA(a, b, 3, 5, d) + d[4])));
    c[2] = (KARA(a, b, 0, 2, d) + d[1] + montgomery_reduce_n653_cuda((int64_t)zeta * (KARA(a, b, 3, 6, d) + KARA(a, b, 4, 5, d))));
    c[3] = (KARA(a, b, 0, 3, d) + KARA(a, b, 1, 2, d) + montgomery_reduce_n653_cuda((int64_t)zeta * (KARA(a, b, 4, 6, d) + d[5])));
    c[4] = (KARA(a, b, 0, 4, d) + KARA(a, b, 1, 3, d) + d[2] + montgomery_reduce_n653_cuda((int64_t)zeta * (KARA(a, b, 5, 6, d))));
    c[5] = (KARA(a, b, 0, 5, d) + KARA(a, b, 1, 4, d) + KARA(a, b, 2, 3, d) + montgomery_reduce_n653_cuda((int64_t)zeta * d[6]));
    c[6] = (KARA(a, b, 0, 6, d) + KARA(a, b, 1, 5, d) + KARA(a, b, 2, 4, d) + d[3]);
}

/*2024-4-19:
输入:array_a:设备端多项式数组array_a

输出:存储到array_a中

处理流程:数据分块后，调用invntt_tomont_n653对每一个多项式进行数据处理

线程组织形式:<<<BATCH_SIZE,N_N_653/2>>>*/
__global__ void poly_invntt_n653_batch(nttpoly_n653 * array_a){
    invntt_tomont_n653((array_a[blockIdx.x]).coeffs);//TODO:这里很奇怪,array_a是数组指针的形式吗？
}

/*2024-4-19:
输入:a:设备端以数组表示的多项式

输出:存储到a中

处理流程:N_N653/2个线程协同处理一个多项式的逆NTT操作*/
__device__ void invntt_tomont_n653(int32_t a[N_N653]){
    __shared__ int32_t poly_a[N_N653];//shared memory
    unsigned int tid = threadIdx.x,idx;//idx指向线程tid处理的数
    unsigned int len;
    unsigned int zeta_start = 0;

    int32_t t;
    int32_t zeta;


    if(tid < N_N653/2){
        poly_a[tid] = a[tid];
        poly_a[tid+N_N653/2]=a[tid+N_N653/2];

        //倒数1层基2NTT
        __syncthreads();

        len=7;
        zeta=zetas_inv_n653[zeta_start + (tid / len)];
        idx = ( tid / len ) * len * 2 + tid % len;
        if(zeta == NUMBER_AKNCF){ //注意:这里一定要有条件判断，否则会出现错误
            t = poly_a[idx];
            poly_a[idx] = (poly_a[idx + len] + t);
            poly_a[idx + len] = (poly_a[idx + len] -t);
        }
        else{
            t=poly_a[idx];
            poly_a[idx] = (t + poly_a[idx + len]);
            poly_a[idx + len] = (t - poly_a[idx + len]);
            poly_a[idx + len] = montgomery_reduce_n653_cuda((int64_t)zeta * poly_a[idx + len]);
        }
        

        //倒数第2层基3NTT
        int32_t tb, tc, tpho, zeta1, zeta2;
        len =14;
        zeta_start = 96;

        __syncthreads();
        if(tid < N_N653 / 3){
            
            zeta1 = zetas_inv_n653[zeta_start + (tid / len) * 2];//TODO:注意考虑这里是否会发生bank的冲突
            //if(tid == 0 ) printf("%d\n",zeta1);
            zeta2 = zetas_inv_n653[zeta_start + (tid / len) * 2 + 1];
            //if(tid == 0 ) printf("%d\n",zeta2);

            idx = ( tid / len ) * len * 3 + tid % len;

            tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (poly_a[idx + len] - poly_a[idx + len + len]));
            tb = (poly_a[idx] - poly_a[idx + len] - tpho);
            tc = (poly_a[idx] - poly_a[idx + len + len] + tpho);

            /*if(idx == 50){
                printf("[before cuda] tpho=%d tb=%d tc=%d a[j]=%d %d %d idx=%d len=%d\n",tpho,tb,tc,poly_a[idx],poly_a[idx+len],poly_a[idx+len+len],idx,len);
            }*/

            poly_a[idx] = (poly_a[idx] + poly_a[idx + len] + poly_a[idx + len + len]);
            poly_a[idx + len] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
            poly_a[idx + len + len] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);

            /*if(idx == 50){
                printf("[after cuda] tpho=%d tb=%d tc=%d a[j]=%d %d %d idx=%d len=%d\n",tpho,tb,tc,poly_a[idx],poly_a[idx+len],poly_a[idx+len+len],idx,len);
            }*/
        }

        
        

        //4层基2NTT
        zeta_start = 160;

        for(len =42; len <= N_N653/4; len <<= 1){
            __syncthreads();
            zeta = zetas_inv_n653[zeta_start + (tid/len)];
            idx = ( tid / len) * len * 2 + tid % len;
            if(zeta == NUMBER_AKNCF){
                t=poly_a[idx];
                poly_a[idx]=(poly_a[idx + len] + t);
                poly_a[idx + len] = (poly_a[idx + len] - t);

            }
            else{
                t=poly_a[idx];
                poly_a[idx] = (t + poly_a[idx + len]);
                poly_a[idx + len] = (t - poly_a[idx + len]);
                poly_a[idx + len] = montgomery_reduce_n653_cuda((int64_t)zeta * poly_a[idx + len]);
            }
            

            zeta_start += N_N653 / (len * 2);

        }


        //最后一层基2NTT
        len = N_N653 / 2;
        __syncthreads();
        zeta = zetas_inv_n653[zeta_start + (tid / len)];
        idx = tid;
        t = poly_a[idx];
        poly_a[idx] = pseudomersenne_reduce_single_n653_cuda(poly_a[idx + len] + t);
        poly_a[idx + len] = pseudomersenne_reduce_single_n653_cuda(poly_a[idx + len] - t);

        //将数据写回全局内存
        __syncthreads();
        a[tid] = poly_a[tid];
        a[tid+N_N653/2]=poly_a[tid+N_N653/2];
    }
}

/*2024-4-19:
输入:array_a:设备端素阶数域上的多项式数组
    array_ntta:设备端伪梅森数缓缓上的多项式数组

输出:存储到array_a中

处理流程:将数据分块后调用poly_extract_n653

线程组织形式:<<<BATCH_SIZE,1>>>
*/
__global__ void poly_extract_n653_q1_batch(poly * array_a ,nttpoly_n653 * array_ntta){
    poly_extract_n653_q1(&(array_a[blockIdx.x]),&(array_ntta[blockIdx.x]));
}

/*2024-4-19:
输入:b:设备端素阶数域多项式
    a:设备端伪梅森环上的多项式

输出:存储到b中

处理流程:TODO:细看一下*/
__device__ void poly_extract_n653_q1(poly *b,nttpoly_n653 *a){
    b->coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (a->coeffs[0] + a->coeffs[FPTRU_N])));
    for (int i = 1; i <= FPTRU_N - 2; i++)
        b->coeffs[i] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (a->coeffs[i] + a->coeffs[i + FPTRU_N - 1] + a->coeffs[i + FPTRU_N])));
    b->coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (a->coeffs[FPTRU_N - 1] + a->coeffs[2 * FPTRU_N - 2])));
}

#define special_CT_a(i,j)  t = a[j];a[j]=(a[i] + t);a[i] = (a[i] - t);
#define special_CT_b(i,j)  t = b[j];b[j]=(b[i] + t);b[i] = (b[i] - t);

#define special_CT_a_pse(i,j)  t = a[i];a[i]=pseudomersenne_reduce_single_n653_cuda(a[j] + t);a[j] = pseudomersenne_reduce_single_n653_cuda(a[j] - t);

#define CT_a(i,j,zeta)  t = montgomery_reduce_n653_cuda((int64_t)zeta * a[j]); a[j] = (a[i] - t); a[i] = (a[i] + t);

#define CT_b(i,j,zeta)  t = montgomery_reduce_n653_cuda((int64_t)zeta * b[j]); b[j] = (b[i] - t); b[i] = (b[i] + t);

#define GS_a(i,j,zeta) t = a[i]; a[i] = (t + a[j]); a[j] = (t - a[j]); a[j] = montgomery_reduce_n653_cuda((int64_t)zeta * a[j]);

/*2024-6-17:
<<<batch_size,168>>>
*/
__global__ void poly_mul_653_batch_q1_v3(poly *array_c,poly * array_a,poly *array_b){
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    int32_t a[8] ={0};
    int32_t b[8] ={0};//注意数据类型和nttpoly_n653保持一致
    int idx = threadIdx.x;
    int32_t zeta;
    int32_t t;

    poly pa;
    poly pb;

    /*int c21 = idx / 21;
    int m21 = idx % 21;
    
    int c7 = idx / 7;
    int m7 = idx - c7 * 7;*/
    pa = array_a[blockIdx.x];
    pb = array_b[blockIdx.x];

    //不做extend，直接读8个系数
    a[0] = pa.coeffs[idx];//array_a[blockIdx.x].coeffs[idx];
    a[1] = pa.coeffs[idx + 168];//array_a[blockIdx.x].coeffs[idx + 168];
    a[2] = pa.coeffs[idx + 168 * 2];//array_a[blockIdx.x].coeffs[idx + 168 * 2];
    
    if(idx + 168 * 3 < FPTRU_N){
        a[3] = pa.coeffs[idx + 168 * 3];//array_a[blockIdx.x].coeffs[idx + 168 * 3];
    }
    
    //不做extend，直接读8个系数
    b[0] = pb.coeffs[idx];//array_b[blockIdx.x].coeffs[idx];
    b[1] = pb.coeffs[idx + 168];//array_b[blockIdx.x].coeffs[idx + 168];
    b[2] = pb.coeffs[idx + 168 * 2];//array_b[blockIdx.x].coeffs[idx + 168 * 2];
    
    if(idx + 168 * 3 < FPTRU_N){
        b[3] = pb.coeffs[idx + 168 * 3];//array_b[blockIdx.x].coeffs[idx + 168 * 3];
    }
    
    //一层基2
    //zeta = zetas_n653[1]; //旋转因子为-1
    special_CT_a(0,4);
    special_CT_a(1,5);
    special_CT_a(2,6);
    special_CT_a(3,7);

    special_CT_b(0,4);
    special_CT_b(1,5);
    special_CT_b(2,6);
    special_CT_b(3,7);

    

    //2 基2
    zeta = zetas_n653_v2[2];

    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    //zeta = zetas_n653[3]; //旋转因子为-1
    special_CT_a(4,6);
    special_CT_a(5,7);

    special_CT_b(4,6);
    special_CT_b(5,7);


    //3 基2
    zeta = zetas_n653_v2[4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n653_v2[5];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n653_v2[6];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    special_CT_a(6,7);
    special_CT_b(6,7);

    //放回共享内存
    ntta.coeffs[idx] = a[0];
    ntta.coeffs[idx + 168] = a[1];
    ntta.coeffs[idx + 168 * 2] = a[2];
    ntta.coeffs[idx + 168 * 3] = a[3];
    ntta.coeffs[idx + 168 * 4] = a[4];
    ntta.coeffs[idx + 168 * 5] = a[5];
    ntta.coeffs[idx + 168 * 6] = a[6];
    ntta.coeffs[idx + 168 * 7] = a[7];

    nttb.coeffs[idx] = b[0];
    nttb.coeffs[idx + 168] = b[1];
    nttb.coeffs[idx + 168 * 2] = b[2];
    nttb.coeffs[idx + 168 * 3] = b[3];
    nttb.coeffs[idx + 168 * 4] = b[4];
    nttb.coeffs[idx + 168 * 5] = b[5];
    nttb.coeffs[idx + 168 * 6] = b[6];
    nttb.coeffs[idx + 168 * 7] = b[7];

    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }*/

   //4 基2
    //int start = (idx & 7) * 168;
    int group_num = idx / 21;
    int group_idx = idx % 21;
    int start = group_num * 168;
    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 21];
    a[2] = ntta.coeffs[start + group_idx + 21 * 2];
    a[3] = ntta.coeffs[start + group_idx + 21 * 3];
    a[4] = ntta.coeffs[start + group_idx + 21 * 4];
    a[5] = ntta.coeffs[start + group_idx + 21 * 5];
    a[6] = ntta.coeffs[start + group_idx + 21 * 6];
    a[7] = ntta.coeffs[start + group_idx + 21 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 21];
    b[2] = nttb.coeffs[start + group_idx + 21 * 2];
    b[3] = nttb.coeffs[start + group_idx + 21 * 3];
    b[4] = nttb.coeffs[start + group_idx + 21 * 4];
    b[5] = nttb.coeffs[start + group_idx + 21 * 5];
    b[6] = nttb.coeffs[start + group_idx + 21 * 6];
    b[7] = nttb.coeffs[start + group_idx + 21 * 7];

    //printf("%d,%d\n",idx,start);
    zeta = zetas_n653_v2[8 + group_num];

    //if(idx == 0){ printf("%d,(%d,%d)\n",zeta,a[0],a[4]);}
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);
    //if(idx == 0){ printf("%d,(%d,%d)\n",zeta,a[0],a[4]);}
    //goto res;

    //5 基2
    zeta = zetas_n653_v2[16 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);


    zeta = zetas_n653_v2[16 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    

    //6 基3
    zeta = zetas_n653_v2[32 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);


    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 21] = a[1];
    ntta.coeffs[start + group_idx + 21 * 2] = a[2];
    ntta.coeffs[start + group_idx + 21 * 3] = a[3];
    ntta.coeffs[start + group_idx + 21 * 4] = a[4];
    ntta.coeffs[start + group_idx + 21 * 5] = a[5];
    ntta.coeffs[start + group_idx + 21 * 6] = a[6];
    ntta.coeffs[start + group_idx + 21 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 21] = b[1];
    nttb.coeffs[start + group_idx + 21 * 2] = b[2];
    nttb.coeffs[start + group_idx + 21 * 3] = b[3];
    nttb.coeffs[start + group_idx + 21 * 4] = b[4];
    nttb.coeffs[start + group_idx + 21 * 5] = b[5];
    nttb.coeffs[start + group_idx + 21 * 6] = b[6];
    nttb.coeffs[start + group_idx + 21 * 7] = b[7];


    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }*/

   //基础 3，共有64组,先做24组，再做24组，再做16组
   //3.1
   group_num = idx / 7;
   group_idx = idx % 7;
   start = group_num * 21;
   
   int32_t zeta1 = zetas_n653_v2[64 + group_num * 2];
   int32_t zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

   int32_t tb, tc, tpho;

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
    ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
    ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
    nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
    nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    __syncthreads();
    //goto res;
    //TODO:为什么这里会存在溢出呢？
    //3.2
    group_num = idx / 7 + 24;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_n653_v2[64 + group_num * 2];
    zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
    ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
    ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);


    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
    nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
    nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    __syncthreads();
    

    //3.3
    group_num = idx / 7 + 48;
    group_idx = idx % 7;
    start = group_num * 21;
    if(group_num < 64){
        zeta1 = zetas_n653_v2[64 + group_num * 2];
        zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

        tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
        tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

        ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
        ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
        ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);

        tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
        tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

        nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
        nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
        nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    }
    __syncthreads();
/*
res:
    if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }
*/
    //basemul
    group_num = idx / 3;
    group_idx = idx % 3;

    zeta = zetas_n653_v2[N_N653 / 21 + group_num * 2];
    zeta = montgomery_reduce_n653_cuda((int64_t)zeta * root3_n653[group_idx]);

    basemul_n653(nttc.coeffs + 21 * group_num + 7 * group_idx, ntta.coeffs + 21 * group_num + 7 * group_idx, nttb.coeffs + 21 * group_num + 7 * group_idx, zeta);

    group_num = idx / 3 + 56;
    if(group_num < 64){
        zeta = zetas_n653_v2[N_N653 / 21 + group_num * 2];
        zeta = montgomery_reduce_n653_cuda((int64_t)zeta * root3_n653[group_idx]);

        basemul_n653(nttc.coeffs + 21 * group_num + 7 * group_idx, ntta.coeffs + 21 * group_num + 7 * group_idx, nttb.coeffs + 21 * group_num + 7 * group_idx, zeta);
    }

    __syncthreads();
    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

   //invntt
   // 1 - radix 3
    group_num = idx / 7;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_inv_n653_v2[group_num * 2];
    zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
    tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
    tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

    nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
    nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
    nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);

    //2 - radix 3
    group_num = idx / 7 + 24;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_inv_n653_v2[group_num * 2];
    zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
    tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
    tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

    nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
    nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
    nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);

    //3 - radix 3
    group_num = idx / 7 + 48;
    group_idx = idx % 7;
    start = group_num * 21;
    if(group_num < 64){
        zeta1 = zetas_inv_n653_v2[group_num * 2];
        zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
        tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
        tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

        nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
        nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
        nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);
    }

    __syncthreads();
    
    //3层- radix 2

    //1
    group_num = idx / 21;
    group_idx = idx % 21;
    start = group_num * 168;
    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 21];
    a[2] = nttc.coeffs[start + group_idx + 21 * 2];
    a[3] = nttc.coeffs[start + group_idx + 21 * 3];
    a[4] = nttc.coeffs[start + group_idx + 21 * 4];
    a[5] = nttc.coeffs[start + group_idx + 21 * 5];
    a[6] = nttc.coeffs[start + group_idx + 21 * 6];
    a[7] = nttc.coeffs[start + group_idx + 21 * 7];

    zeta = zetas_inv_n653_v2[128 + group_num * 4];
    GS_a(0,1,zeta);
    
    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 3];
    GS_a(6,7,zeta);

    //2
    zeta = zetas_inv_n653_v2[160 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n653_v2[160 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //3
    zeta = zetas_inv_n653_v2[176 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 21] = a[1];
    nttc.coeffs[start + group_idx + 21 * 2] = a[2];
    nttc.coeffs[start + group_idx + 21 * 3] = a[3];
    nttc.coeffs[start + group_idx + 21 * 4] = a[4];
    nttc.coeffs[start + group_idx + 21 * 5] = a[5];
    nttc.coeffs[start + group_idx + 21 * 6] = a[6];
    nttc.coeffs[start + group_idx + 21 * 7] = a[7];

    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

    a[0] = nttc.coeffs[idx];
    a[1] = nttc.coeffs[idx + 168];
    a[2] = nttc.coeffs[idx + 168 * 2];
    a[3] = nttc.coeffs[idx + 168 * 3];
    a[4] = nttc.coeffs[idx + 168 * 4];
    a[5] = nttc.coeffs[idx + 168 * 5];
    a[6] = nttc.coeffs[idx + 168 * 6];
    a[7] = nttc.coeffs[idx + 168 * 7];
    
    //1
    zeta = zetas_inv_n653_v2[184];
    GS_a(0,1,zeta);
    
    zeta = zetas_inv_n653_v2[185];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n653_v2[186];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n653_v2[187];
    GS_a(6,7,zeta);

    //2
    zeta = zetas_inv_n653_v2[188];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n653_v2[189];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    
    //3
    special_CT_a_pse(0,4);
    special_CT_a_pse(1,5);
    special_CT_a_pse(2,6);
    special_CT_a_pse(3,7);

    nttc.coeffs[idx] = a[0];
    nttc.coeffs[idx + 168] = a[1];
    nttc.coeffs[idx + 168 * 2] = a[2];
    nttc.coeffs[idx + 168 * 3] = a[3];
    nttc.coeffs[idx + 168 * 4] = a[4];
    nttc.coeffs[idx + 168 * 5] = a[5];
    nttc.coeffs[idx + 168 * 6] = a[6];
    nttc.coeffs[idx + 168 * 7] = a[7];

    __syncthreads();
    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

    //只需要FPTRU_N个线程来完成 除去0和652
    array_c[blockIdx.x].coeffs[1 + idx * 3 ] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[1 + idx * 3 ] + nttc.coeffs[1 + idx * 3  + FPTRU_N - 1] + nttc.coeffs[1 + idx * 3  + FPTRU_N])));

    array_c[blockIdx.x].coeffs[2 + idx * 3  ] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[2 + idx * 3  ] + nttc.coeffs[2 + idx * 3   + FPTRU_N - 1] + nttc.coeffs[2 + idx * 3   + FPTRU_N])));

    array_c[blockIdx.x].coeffs[3 + idx * 3  ] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[3 + idx * 3  ] + nttc.coeffs[3 + idx * 3   + FPTRU_N - 1] + nttc.coeffs[3 + idx * 3   + FPTRU_N])));


    if(idx < 148){
        if(idx == 0){
            array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
            array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
        }
        else{
            array_c[blockIdx.x].coeffs[idx + 504] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx + 504] + nttc.coeffs[idx + 504 + FPTRU_N - 1] + nttc.coeffs[idx + 504 + FPTRU_N])));
        }
    }


}

/*2024-6-23:
设备端函数，处理一次多项式的乘法*/
__device__ void poly_mul_653_batch_q1_v3_device(poly *array_c,poly * array_a,poly *array_b){
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    int32_t a[8] ={0};
    int32_t b[8] ={0};//注意数据类型和nttpoly_n653保持一致
    int idx = threadIdx.x;
    int32_t zeta;
    int32_t t;

    poly pa;
    poly pb;

    /*int c21 = idx / 21;
    int m21 = idx % 21;
    
    int c7 = idx / 7;
    int m7 = idx - c7 * 7;*/
    pa = array_a[blockIdx.x];
    pb = array_b[blockIdx.x];

    //不做extend，直接读8个系数
    a[0] = pa.coeffs[idx];//array_a[blockIdx.x].coeffs[idx];
    a[1] = pa.coeffs[idx + 168];//array_a[blockIdx.x].coeffs[idx + 168];
    a[2] = pa.coeffs[idx + 168 * 2];//array_a[blockIdx.x].coeffs[idx + 168 * 2];
    
    if(idx + 168 * 3 < FPTRU_N){
        a[3] = pa.coeffs[idx + 168 * 3];//array_a[blockIdx.x].coeffs[idx + 168 * 3];
    }
    
    //不做extend，直接读8个系数
    b[0] = pb.coeffs[idx];//array_b[blockIdx.x].coeffs[idx];
    b[1] = pb.coeffs[idx + 168];//array_b[blockIdx.x].coeffs[idx + 168];
    b[2] = pb.coeffs[idx + 168 * 2];//array_b[blockIdx.x].coeffs[idx + 168 * 2];
    
    if(idx + 168 * 3 < FPTRU_N){
        b[3] = pb.coeffs[idx + 168 * 3];//array_b[blockIdx.x].coeffs[idx + 168 * 3];
    }
    
    //一层基2
    //zeta = zetas_n653[1]; //旋转因子为-1
    special_CT_a(0,4);
    special_CT_a(1,5);
    special_CT_a(2,6);
    special_CT_a(3,7);

    special_CT_b(0,4);
    special_CT_b(1,5);
    special_CT_b(2,6);
    special_CT_b(3,7);

    

    //2 基2
    zeta = zetas_n653_v2[2];

    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    //zeta = zetas_n653[3]; //旋转因子为-1
    special_CT_a(4,6);
    special_CT_a(5,7);

    special_CT_b(4,6);
    special_CT_b(5,7);


    //3 基2
    zeta = zetas_n653_v2[4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n653_v2[5];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n653_v2[6];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    special_CT_a(6,7);
    special_CT_b(6,7);

    //放回共享内存
    ntta.coeffs[idx] = a[0];
    ntta.coeffs[idx + 168] = a[1];
    ntta.coeffs[idx + 168 * 2] = a[2];
    ntta.coeffs[idx + 168 * 3] = a[3];
    ntta.coeffs[idx + 168 * 4] = a[4];
    ntta.coeffs[idx + 168 * 5] = a[5];
    ntta.coeffs[idx + 168 * 6] = a[6];
    ntta.coeffs[idx + 168 * 7] = a[7];

    nttb.coeffs[idx] = b[0];
    nttb.coeffs[idx + 168] = b[1];
    nttb.coeffs[idx + 168 * 2] = b[2];
    nttb.coeffs[idx + 168 * 3] = b[3];
    nttb.coeffs[idx + 168 * 4] = b[4];
    nttb.coeffs[idx + 168 * 5] = b[5];
    nttb.coeffs[idx + 168 * 6] = b[6];
    nttb.coeffs[idx + 168 * 7] = b[7];

    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }*/

   //4 基2
    //int start = (idx & 7) * 168;
    int group_num = idx / 21;
    int group_idx = idx % 21;
    int start = group_num * 168;
    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 21];
    a[2] = ntta.coeffs[start + group_idx + 21 * 2];
    a[3] = ntta.coeffs[start + group_idx + 21 * 3];
    a[4] = ntta.coeffs[start + group_idx + 21 * 4];
    a[5] = ntta.coeffs[start + group_idx + 21 * 5];
    a[6] = ntta.coeffs[start + group_idx + 21 * 6];
    a[7] = ntta.coeffs[start + group_idx + 21 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 21];
    b[2] = nttb.coeffs[start + group_idx + 21 * 2];
    b[3] = nttb.coeffs[start + group_idx + 21 * 3];
    b[4] = nttb.coeffs[start + group_idx + 21 * 4];
    b[5] = nttb.coeffs[start + group_idx + 21 * 5];
    b[6] = nttb.coeffs[start + group_idx + 21 * 6];
    b[7] = nttb.coeffs[start + group_idx + 21 * 7];

    //printf("%d,%d\n",idx,start);
    zeta = zetas_n653_v2[8 + group_num];

    //if(idx == 0){ printf("%d,(%d,%d)\n",zeta,a[0],a[4]);}
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);
    //if(idx == 0){ printf("%d,(%d,%d)\n",zeta,a[0],a[4]);}
    //goto res;

    //5 基2
    zeta = zetas_n653_v2[16 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);


    zeta = zetas_n653_v2[16 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    

    //6 基3
    zeta = zetas_n653_v2[32 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);


    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 21] = a[1];
    ntta.coeffs[start + group_idx + 21 * 2] = a[2];
    ntta.coeffs[start + group_idx + 21 * 3] = a[3];
    ntta.coeffs[start + group_idx + 21 * 4] = a[4];
    ntta.coeffs[start + group_idx + 21 * 5] = a[5];
    ntta.coeffs[start + group_idx + 21 * 6] = a[6];
    ntta.coeffs[start + group_idx + 21 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 21] = b[1];
    nttb.coeffs[start + group_idx + 21 * 2] = b[2];
    nttb.coeffs[start + group_idx + 21 * 3] = b[3];
    nttb.coeffs[start + group_idx + 21 * 4] = b[4];
    nttb.coeffs[start + group_idx + 21 * 5] = b[5];
    nttb.coeffs[start + group_idx + 21 * 6] = b[6];
    nttb.coeffs[start + group_idx + 21 * 7] = b[7];


    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }*/

   //基础 3，共有64组,先做24组，再做24组，再做16组
   //3.1
   group_num = idx / 7;
   group_idx = idx % 7;
   start = group_num * 21;
   
   int32_t zeta1 = zetas_n653_v2[64 + group_num * 2];
   int32_t zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

   int32_t tb, tc, tpho;

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
    ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
    ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
    nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
    nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    __syncthreads();
    //goto res;
    //TODO:为什么这里会存在溢出呢？
    //3.2
    group_num = idx / 7 + 24;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_n653_v2[64 + group_num * 2];
    zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
    ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
    ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);


    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
    nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
    nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    __syncthreads();
    

    //3.3
    group_num = idx / 7 + 48;
    group_idx = idx % 7;
    start = group_num * 21;
    if(group_num < 64){
        zeta1 = zetas_n653_v2[64 + group_num * 2];
        zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

        tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
        tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

        ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
        ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
        ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);

        tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
        tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

        nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
        nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
        nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    }
    __syncthreads();
/*
res:
    if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }
*/
    //basemul
    group_num = idx / 3;
    group_idx = idx % 3;

    zeta = zetas_n653_v2[N_N653 / 21 + group_num * 2];
    zeta = montgomery_reduce_n653_cuda((int64_t)zeta * root3_n653[group_idx]);

    basemul_n653(nttc.coeffs + 21 * group_num + 7 * group_idx, ntta.coeffs + 21 * group_num + 7 * group_idx, nttb.coeffs + 21 * group_num + 7 * group_idx, zeta);

    group_num = idx / 3 + 56;
    if(group_num < 64){
        zeta = zetas_n653_v2[N_N653 / 21 + group_num * 2];
        zeta = montgomery_reduce_n653_cuda((int64_t)zeta * root3_n653[group_idx]);

        basemul_n653(nttc.coeffs + 21 * group_num + 7 * group_idx, ntta.coeffs + 21 * group_num + 7 * group_idx, nttb.coeffs + 21 * group_num + 7 * group_idx, zeta);
    }

    __syncthreads();
    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

   //invntt
   // 1 - radix 3
    group_num = idx / 7;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_inv_n653_v2[group_num * 2];
    zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
    tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
    tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

    nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
    nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
    nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);

    //2 - radix 3
    group_num = idx / 7 + 24;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_inv_n653_v2[group_num * 2];
    zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
    tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
    tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

    nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
    nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
    nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);

    //3 - radix 3
    group_num = idx / 7 + 48;
    group_idx = idx % 7;
    start = group_num * 21;
    if(group_num < 64){
        zeta1 = zetas_inv_n653_v2[group_num * 2];
        zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
        tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
        tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

        nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
        nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
        nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);
    }

    __syncthreads();
    
    //3层- radix 2

    //1
    group_num = idx / 21;
    group_idx = idx % 21;
    start = group_num * 168;
    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 21];
    a[2] = nttc.coeffs[start + group_idx + 21 * 2];
    a[3] = nttc.coeffs[start + group_idx + 21 * 3];
    a[4] = nttc.coeffs[start + group_idx + 21 * 4];
    a[5] = nttc.coeffs[start + group_idx + 21 * 5];
    a[6] = nttc.coeffs[start + group_idx + 21 * 6];
    a[7] = nttc.coeffs[start + group_idx + 21 * 7];

    zeta = zetas_inv_n653_v2[128 + group_num * 4];
    GS_a(0,1,zeta);
    
    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 3];
    GS_a(6,7,zeta);

    //2
    zeta = zetas_inv_n653_v2[160 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n653_v2[160 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //3
    zeta = zetas_inv_n653_v2[176 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 21] = a[1];
    nttc.coeffs[start + group_idx + 21 * 2] = a[2];
    nttc.coeffs[start + group_idx + 21 * 3] = a[3];
    nttc.coeffs[start + group_idx + 21 * 4] = a[4];
    nttc.coeffs[start + group_idx + 21 * 5] = a[5];
    nttc.coeffs[start + group_idx + 21 * 6] = a[6];
    nttc.coeffs[start + group_idx + 21 * 7] = a[7];

    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

    a[0] = nttc.coeffs[idx];
    a[1] = nttc.coeffs[idx + 168];
    a[2] = nttc.coeffs[idx + 168 * 2];
    a[3] = nttc.coeffs[idx + 168 * 3];
    a[4] = nttc.coeffs[idx + 168 * 4];
    a[5] = nttc.coeffs[idx + 168 * 5];
    a[6] = nttc.coeffs[idx + 168 * 6];
    a[7] = nttc.coeffs[idx + 168 * 7];
    
    //1
    zeta = zetas_inv_n653_v2[184];
    GS_a(0,1,zeta);
    
    zeta = zetas_inv_n653_v2[185];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n653_v2[186];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n653_v2[187];
    GS_a(6,7,zeta);

    //2
    zeta = zetas_inv_n653_v2[188];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n653_v2[189];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    
    //3
    special_CT_a_pse(0,4);
    special_CT_a_pse(1,5);
    special_CT_a_pse(2,6);
    special_CT_a_pse(3,7);

    nttc.coeffs[idx] = a[0];
    nttc.coeffs[idx + 168] = a[1];
    nttc.coeffs[idx + 168 * 2] = a[2];
    nttc.coeffs[idx + 168 * 3] = a[3];
    nttc.coeffs[idx + 168 * 4] = a[4];
    nttc.coeffs[idx + 168 * 5] = a[5];
    nttc.coeffs[idx + 168 * 6] = a[6];
    nttc.coeffs[idx + 168 * 7] = a[7];

    __syncthreads();
    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

    //只需要FPTRU_N个线程来完成 除去0和652
    array_c[blockIdx.x].coeffs[1 + idx * 3 ] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[1 + idx * 3 ] + nttc.coeffs[1 + idx * 3  + FPTRU_N - 1] + nttc.coeffs[1 + idx * 3  + FPTRU_N])));

    array_c[blockIdx.x].coeffs[2 + idx * 3  ] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[2 + idx * 3  ] + nttc.coeffs[2 + idx * 3   + FPTRU_N - 1] + nttc.coeffs[2 + idx * 3   + FPTRU_N])));

    array_c[blockIdx.x].coeffs[3 + idx * 3  ] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[3 + idx * 3  ] + nttc.coeffs[3 + idx * 3   + FPTRU_N - 1] + nttc.coeffs[3 + idx * 3   + FPTRU_N])));


    if(idx < 148){
        if(idx == 0){
            array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
            array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
        }
        else{
            array_c[blockIdx.x].coeffs[idx + 504] = fq_freeze(montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx + 504] + nttc.coeffs[idx + 504 + FPTRU_N - 1] + nttc.coeffs[idx + 504 + FPTRU_N])));
        }
    }


}

/*__device__ void CT_ptx(int32_t* px,int32_t* py,int32_t zeta){
    int32_t x = *px;
    int32_t y = *py;

    asm(
        "{\n\t"
        ""

        }
    );

    *px = x;
    *py = y;
}*/



__global__ void poly_mul_653_batch_q2(poly *array_c,poly * array_a,poly *array_b){
    __shared__ nttpoly_n653 ntta;
    __shared__ nttpoly_n653 nttb;
    __shared__ nttpoly_n653 nttc;

    int32_t a[8] ={0};
    int32_t b[8] ={0};//注意数据类型和nttpoly_n653保持一致
    int idx = threadIdx.x;
    int32_t zeta;
    int32_t t;

    poly pa;
    poly pb;

    /*int c21 = idx / 21;
    int m21 = idx % 21;
    
    int c7 = idx / 7;
    int m7 = idx - c7 * 7;*/
    pa = array_a[blockIdx.x];
    pb = array_b[blockIdx.x];

    //不做extend，直接读8个系数
    a[0] = pa.coeffs[idx];//array_a[blockIdx.x].coeffs[idx];
    a[1] = pa.coeffs[idx + 168];//array_a[blockIdx.x].coeffs[idx + 168];
    a[2] = pa.coeffs[idx + 168 * 2];//array_a[blockIdx.x].coeffs[idx + 168 * 2];
    
    if(idx + 168 * 3 < FPTRU_N){
        a[3] = pa.coeffs[idx + 168 * 3];//array_a[blockIdx.x].coeffs[idx + 168 * 3];
    }
    
    //不做extend，直接读8个系数
    b[0] = pb.coeffs[idx];//array_b[blockIdx.x].coeffs[idx];
    b[1] = pb.coeffs[idx + 168];//array_b[blockIdx.x].coeffs[idx + 168];
    b[2] = pb.coeffs[idx + 168 * 2];//array_b[blockIdx.x].coeffs[idx + 168 * 2];
    
    if(idx + 168 * 3 < FPTRU_N){
        b[3] = pb.coeffs[idx + 168 * 3];//array_b[blockIdx.x].coeffs[idx + 168 * 3];
    }
    
    //一层基2
    //zeta = zetas_n653[1]; //旋转因子为-1
    special_CT_a(0,4);
    special_CT_a(1,5);
    special_CT_a(2,6);
    special_CT_a(3,7);

    special_CT_b(0,4);
    special_CT_b(1,5);
    special_CT_b(2,6);
    special_CT_b(3,7);

    

    //2 基2
    zeta = zetas_n653_v2[2];

    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    //zeta = zetas_n653[3]; //旋转因子为-1
    special_CT_a(4,6);
    special_CT_a(5,7);

    special_CT_b(4,6);
    special_CT_b(5,7);


    //3 基2
    zeta = zetas_n653_v2[4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n653_v2[5];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n653_v2[6];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    special_CT_a(6,7);
    special_CT_b(6,7);

    //放回共享内存
    ntta.coeffs[idx] = a[0];
    ntta.coeffs[idx + 168] = a[1];
    ntta.coeffs[idx + 168 * 2] = a[2];
    ntta.coeffs[idx + 168 * 3] = a[3];
    ntta.coeffs[idx + 168 * 4] = a[4];
    ntta.coeffs[idx + 168 * 5] = a[5];
    ntta.coeffs[idx + 168 * 6] = a[6];
    ntta.coeffs[idx + 168 * 7] = a[7];

    nttb.coeffs[idx] = b[0];
    nttb.coeffs[idx + 168] = b[1];
    nttb.coeffs[idx + 168 * 2] = b[2];
    nttb.coeffs[idx + 168 * 3] = b[3];
    nttb.coeffs[idx + 168 * 4] = b[4];
    nttb.coeffs[idx + 168 * 5] = b[5];
    nttb.coeffs[idx + 168 * 6] = b[6];
    nttb.coeffs[idx + 168 * 7] = b[7];

    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }*/

   //4 基2
    //int start = (idx & 7) * 168;
    int group_num = idx / 21;
    int group_idx = idx % 21;
    int start = group_num * 168;
    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 21];
    a[2] = ntta.coeffs[start + group_idx + 21 * 2];
    a[3] = ntta.coeffs[start + group_idx + 21 * 3];
    a[4] = ntta.coeffs[start + group_idx + 21 * 4];
    a[5] = ntta.coeffs[start + group_idx + 21 * 5];
    a[6] = ntta.coeffs[start + group_idx + 21 * 6];
    a[7] = ntta.coeffs[start + group_idx + 21 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 21];
    b[2] = nttb.coeffs[start + group_idx + 21 * 2];
    b[3] = nttb.coeffs[start + group_idx + 21 * 3];
    b[4] = nttb.coeffs[start + group_idx + 21 * 4];
    b[5] = nttb.coeffs[start + group_idx + 21 * 5];
    b[6] = nttb.coeffs[start + group_idx + 21 * 6];
    b[7] = nttb.coeffs[start + group_idx + 21 * 7];

    //printf("%d,%d\n",idx,start);
    zeta = zetas_n653_v2[8 + group_num];

    //if(idx == 0){ printf("%d,(%d,%d)\n",zeta,a[0],a[4]);}
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);
    //if(idx == 0){ printf("%d,(%d,%d)\n",zeta,a[0],a[4]);}
    //goto res;

    //5 基2
    zeta = zetas_n653_v2[16 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);


    zeta = zetas_n653_v2[16 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    

    //6 基3
    zeta = zetas_n653_v2[32 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n653_v2[32 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);


    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 21] = a[1];
    ntta.coeffs[start + group_idx + 21 * 2] = a[2];
    ntta.coeffs[start + group_idx + 21 * 3] = a[3];
    ntta.coeffs[start + group_idx + 21 * 4] = a[4];
    ntta.coeffs[start + group_idx + 21 * 5] = a[5];
    ntta.coeffs[start + group_idx + 21 * 6] = a[6];
    ntta.coeffs[start + group_idx + 21 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 21] = b[1];
    nttb.coeffs[start + group_idx + 21 * 2] = b[2];
    nttb.coeffs[start + group_idx + 21 * 3] = b[3];
    nttb.coeffs[start + group_idx + 21 * 4] = b[4];
    nttb.coeffs[start + group_idx + 21 * 5] = b[5];
    nttb.coeffs[start + group_idx + 21 * 6] = b[6];
    nttb.coeffs[start + group_idx + 21 * 7] = b[7];


    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }*/

   //基础 3，共有64组,先做24组，再做24组，再做16组
   //3.1
   group_num = idx / 7;
   group_idx = idx % 7;
   start = group_num * 21;
   
   int32_t zeta1 = zetas_n653_v2[64 + group_num * 2];
   int32_t zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

   int32_t tb, tc, tpho;

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
    ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
    ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
    nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
    nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    __syncthreads();
    //goto res;
    //TODO:为什么这里会存在溢出呢？
    //3.2
    group_num = idx / 7 + 24;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_n653_v2[64 + group_num * 2];
    zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
    ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
    ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);


    tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
    tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

    nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
    nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
    nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    __syncthreads();
    

    //3.3
    group_num = idx / 7 + 48;
    group_idx = idx % 7;
    start = group_num * 21;
    if(group_num < 64){
        zeta1 = zetas_n653_v2[64 + group_num * 2];
        zeta2 = zetas_n653_v2[64 + group_num * 2 + 1];

        tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * ntta.coeffs[start + group_idx + 7]);
        tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * ntta.coeffs[start + group_idx + 14]);
        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

        ntta.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tb - tpho);
        ntta.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] - tc + tpho);
        ntta.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(ntta.coeffs[start + group_idx] + tb + tc);

        tb = montgomery_reduce_n653_cuda((int64_t)zeta1 * nttb.coeffs[start + group_idx + 7]);
        tc = montgomery_reduce_n653_cuda((int64_t)zeta2 * nttb.coeffs[start + group_idx + 14]);
        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (tb - tc));

        nttb.coeffs[start + group_idx + 14] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tb - tpho);
        nttb.coeffs[start + group_idx + 7] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] - tc + tpho);
        nttb.coeffs[start + group_idx] = pseudomersenne_reduce_single_n653_cuda(nttb.coeffs[start + group_idx] + tb + tc);

    }
    __syncthreads();
/*
res:
    if(idx == 0){
        for(int i=0;i<N_N653;i++){
            printf("%d,",ntta.coeffs[i]);
        }
        printf("\n\n");

        for(int i=0;i<N_N653;i++){
            printf("%d,",nttb.coeffs[i]);
        }
        printf("\n\n");
    }
*/
    //basemul
    group_num = idx / 3;
    group_idx = idx % 3;

    zeta = zetas_n653_v2[N_N653 / 21 + group_num * 2];
    zeta = montgomery_reduce_n653_cuda((int64_t)zeta * root3_n653[group_idx]);

    basemul_n653(nttc.coeffs + 21 * group_num + 7 * group_idx, ntta.coeffs + 21 * group_num + 7 * group_idx, nttb.coeffs + 21 * group_num + 7 * group_idx, zeta);

    group_num = idx / 3 + 56;
    if(group_num < 64){
        zeta = zetas_n653_v2[N_N653 / 21 + group_num * 2];
        zeta = montgomery_reduce_n653_cuda((int64_t)zeta * root3_n653[group_idx]);

        basemul_n653(nttc.coeffs + 21 * group_num + 7 * group_idx, ntta.coeffs + 21 * group_num + 7 * group_idx, nttb.coeffs + 21 * group_num + 7 * group_idx, zeta);
    }

    __syncthreads();
    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

   //invntt
   // 1 - radix 3
    group_num = idx / 7;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_inv_n653_v2[group_num * 2];
    zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
    tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
    tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

    nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
    nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
    nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);

    //2 - radix 3
    group_num = idx / 7 + 24;
    group_idx = idx % 7;
    start = group_num * 21;

    zeta1 = zetas_inv_n653_v2[group_num * 2];
    zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

    tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
    tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
    tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

    nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
    nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
    nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);

    //3 - radix 3
    group_num = idx / 7 + 48;
    group_idx = idx % 7;
    start = group_num * 21;
    if(group_num < 64){
        zeta1 = zetas_inv_n653_v2[group_num * 2];
        zeta2 = zetas_inv_n653_v2[group_num * 2 + 1];

        tpho = montgomery_reduce_n653_cuda((int64_t)root3_n653[1] * (nttc.coeffs[start + group_idx + 7] - nttc.coeffs[start + group_idx + 14]));
        tb = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 7] - tpho);
        tc = (nttc.coeffs[start + group_idx] - nttc.coeffs[start + group_idx + 14] + tpho);

        nttc.coeffs[start + group_idx] = (nttc.coeffs[start + group_idx] + nttc.coeffs[start + group_idx + 7] + nttc.coeffs[start + group_idx + 14]);
        nttc.coeffs[start + group_idx + 7] = montgomery_reduce_n653_cuda((int64_t)zeta1 * tb);
        nttc.coeffs[start + group_idx + 14] = montgomery_reduce_n653_cuda((int64_t)zeta2 * tc);
    }

    __syncthreads();
    
    //3层- radix 2

    //1
    group_num = idx / 21;
    group_idx = idx % 21;
    start = group_num * 168;
    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 21];
    a[2] = nttc.coeffs[start + group_idx + 21 * 2];
    a[3] = nttc.coeffs[start + group_idx + 21 * 3];
    a[4] = nttc.coeffs[start + group_idx + 21 * 4];
    a[5] = nttc.coeffs[start + group_idx + 21 * 5];
    a[6] = nttc.coeffs[start + group_idx + 21 * 6];
    a[7] = nttc.coeffs[start + group_idx + 21 * 7];

    zeta = zetas_inv_n653_v2[128 + group_num * 4];
    GS_a(0,1,zeta);
    
    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n653_v2[128 + group_num * 4 + 3];
    GS_a(6,7,zeta);

    //2
    zeta = zetas_inv_n653_v2[160 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n653_v2[160 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //3
    zeta = zetas_inv_n653_v2[176 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 21] = a[1];
    nttc.coeffs[start + group_idx + 21 * 2] = a[2];
    nttc.coeffs[start + group_idx + 21 * 3] = a[3];
    nttc.coeffs[start + group_idx + 21 * 4] = a[4];
    nttc.coeffs[start + group_idx + 21 * 5] = a[5];
    nttc.coeffs[start + group_idx + 21 * 6] = a[6];
    nttc.coeffs[start + group_idx + 21 * 7] = a[7];

    __syncthreads();

    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

    a[0] = nttc.coeffs[idx];
    a[1] = nttc.coeffs[idx + 168];
    a[2] = nttc.coeffs[idx + 168 * 2];
    a[3] = nttc.coeffs[idx + 168 * 3];
    a[4] = nttc.coeffs[idx + 168 * 4];
    a[5] = nttc.coeffs[idx + 168 * 5];
    a[6] = nttc.coeffs[idx + 168 * 6];
    a[7] = nttc.coeffs[idx + 168 * 7];
    
    //1
    zeta = zetas_inv_n653_v2[184];
    GS_a(0,1,zeta);
    
    zeta = zetas_inv_n653_v2[185];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n653_v2[186];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n653_v2[187];
    GS_a(6,7,zeta);

    //2
    zeta = zetas_inv_n653_v2[188];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n653_v2[189];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    
    //3
    special_CT_a_pse(0,4);
    special_CT_a_pse(1,5);
    special_CT_a_pse(2,6);
    special_CT_a_pse(3,7);

    nttc.coeffs[idx] = a[0];
    nttc.coeffs[idx + 168] = a[1];
    nttc.coeffs[idx + 168 * 2] = a[2];
    nttc.coeffs[idx + 168 * 3] = a[3];
    nttc.coeffs[idx + 168 * 4] = a[4];
    nttc.coeffs[idx + 168 * 5] = a[5];
    nttc.coeffs[idx + 168 * 6] = a[6];
    nttc.coeffs[idx + 168 * 7] = a[7];

    __syncthreads();
    /*if(idx == 0){
        for(int i=0;i<N_N653;i++){
                printf("%d,",nttc.coeffs[i]);
            }
        printf("\n\n");
    }*/

    //只需要FPTRU_N个线程来完成 除去0和652
    array_c[blockIdx.x].coeffs[1 + idx * 3 ] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[1 + idx * 3 ] + nttc.coeffs[1 + idx * 3  + FPTRU_N - 1] + nttc.coeffs[1 + idx * 3  + FPTRU_N]))) & (FPTRU_Q2 - 1);
    array_c[blockIdx.x].coeffs[2 + idx * 3  ] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[2 + idx * 3 ] + nttc.coeffs[2 + idx * 3  + FPTRU_N - 1] + nttc.coeffs[2 + idx * 3  + FPTRU_N]))) & (FPTRU_Q2 - 1);

    array_c[blockIdx.x].coeffs[3 + idx * 3  ] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[3 + idx * 3 ] + nttc.coeffs[3 + idx * 3 + FPTRU_N - 1] + nttc.coeffs[3 + idx * 3  + FPTRU_N]))) & (FPTRU_Q2 - 1);


    if(idx < 148){
        if(idx == 0){
            array_c[blockIdx.x].coeffs[0] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N]))) & (FPTRU_Q2 - 1);
            array_c[blockIdx.x].coeffs[FPTRU_N - 1] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2]))) & (FPTRU_Q2 - 1);
        }
        else{
            array_c[blockIdx.x].coeffs[idx + 504] = (montgomery_reduce_n653_cuda((int64_t)FACTOR_NAKYFHAIRNF * (nttc.coeffs[idx + 504] + nttc.coeffs[idx + 504 + FPTRU_N - 1] + nttc.coeffs[idx + 504 + FPTRU_N]))) & (FPTRU_Q2 - 1);
        }
    }


}
