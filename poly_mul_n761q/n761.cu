#include "hip/hip_runtime.h"
#include "n761.h"

#include "../cbd.h"

/*2024-10-14
线程组织结构:
<<<BATCH_SIZE,192>>>*/

__device__ int32_t zetas_n761[N_N761 / 3] = {
    5592919, 33026177, 26790957, 33026177, 13499153, 21938565, 26790957, 33026177, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    32738467, 16657861, 28138468, 501067, 16015129, 21620241, 1529124, 28870577, 18285219, 3050639, 27906268, 21100145, 5343767, 15280732, 12498775, 17354707,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    9795452, 12096975, 33505735, 10865068, 16076627, 23564159, 31258325, 19464215, 24580770, 6067746, 18546326, 875322, 23054283, 15379670, 17941644, 2515533,
    17182522, 9742026, 11427911, 16642093, 24272961, 6983968, 22883012, 17558876, 10658623, 29624690, 1466133, 20016426, 18447657, 8460173, 19793647, 24333065,
    32738467, 16657861, 28138468, 501067, 16015129, 21620241, 1529124, 28870577, 18285219, 3050639, 27906268, 21100145, 5343767, 15280732, 12498775, 17354707,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    10420445, 32038148, 2264, 33265110, 25958325, 11902971, 25622482, 27761690, 28792776, 33361188, 24969419, 12394212, 33193998, 6229980, 8743221, 4457820,
    28382998, 8431289, 1312494, 4681025, 29392120, 295408, 2951574, 9444158, 16620023, 32579155, 31375729, 11964204, 33164555, 16089224, 25578586, 22029388,
    19233249, 12984689, 30813171, 15944111, 13539352, 27143752, 7098522, 30855744, 15029351, 2274807, 31244694, 16958068, 23199393, 17755942, 6583826, 30969403,
    1918505, 22360437, 26047052, 3762725, 10986757, 5860566, 28381447, 3454840, 31955344, 13111549, 10268342, 11767139, 6958801, 13085226, 6345790, 33276057,
    9795452, 12096975, 33505735, 10865068, 16076627, 23564159, 31258325, 19464215, 24580770, 6067746, 18546326, 875322, 23054283, 15379670, 17941644, 2515533,
    17182522, 9742026, 11427911, 16642093, 24272961, 6983968, 22883012, 17558876, 10658623, 29624690, 1466133, 20016426, 18447657, 8460173, 19793647, 24333065,
    32738467, 16657861, 28138468, 501067, 16015129, 21620241, 1529124, 28870577, 18285219, 3050639, 27906268, 21100145, 5343767, 15280732, 12498775, 17354707,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    5592919, 31075518, 21303665, 22682749, 11872764, 2432073, 19322199, 11292275, 23696636, 32659557, 23346277, 31556495, 21579819, 5882189, 6581391, 32328326,
    10696307, 16874833, 31719194, 6911566, 29285890, 23963075, 31256330, 9623777, 31413463, 32254537, 11613809, 24253081, 24356853, 24689701, 17056149, 15482269,
    10167038, 22751501, 18535303, 3968231, 1394219, 26379347, 16811031, 28313524, 16967280, 23729319, 31158593, 2331434, 549280, 18765598, 27900090, 4955113,
    1126043, 7409310, 26996082, 3493835, 5276286, 25397512, 30230071, 26868776, 1340079, 15120857, 24105137, 12241024, 12565260, 33504217, 7283953, 25114323,
    7865111, 32661754, 22892358, 6749048, 9488390, 10088648, 22153949, 5957087, 6543554, 17785943, 9563220, 4840264, 22490456, 6137845, 17318481, 17583519,
    24218356, 23955081, 26738484, 22876568, 3544406, 28057775, 18074286, 26082996, 32554080, 21141507, 15212341, 7347621, 25101513, 16054630, 18249116, 7050722,
    2250785, 7291436, 3750223, 26357349, 10321733, 17918503, 8792503, 33073869, 21274321, 10730693, 6218037, 32966155, 23321992, 20997298, 25154884, 8989918,
    5527986, 7143126, 31109078, 27760189, 9686871, 5400208, 13544313, 4038243, 9254376, 26381630, 18797971, 13377401, 350844, 19639872, 20608463, 28613855,
    10420445, 32038148, 2264, 33265110, 25958325, 11902971, 25622482, 27761690, 28792776, 33361188, 24969419, 12394212, 33193998, 6229980, 8743221, 4457820,
    28382998, 8431289, 1312494, 4681025, 29392120, 295408, 2951574, 9444158, 16620023, 32579155, 31375729, 11964204, 33164555, 16089224, 25578586, 22029388,
    19233249, 12984689, 30813171, 15944111, 13539352, 27143752, 7098522, 30855744, 15029351, 2274807, 31244694, 16958068, 23199393, 17755942, 6583826, 30969403,
    1918505, 22360437, 26047052, 3762725, 10986757, 5860566, 28381447, 3454840, 31955344, 13111549, 10268342, 11767139, 6958801, 13085226, 6345790, 33276057,
    9795452, 12096975, 33505735, 10865068, 16076627, 23564159, 31258325, 19464215, 24580770, 6067746, 18546326, 875322, 23054283, 15379670, 17941644, 2515533,
    17182522, 9742026, 11427911, 16642093, 24272961, 6983968, 22883012, 17558876, 10658623, 29624690, 1466133, 20016426, 18447657, 8460173, 19793647, 24333065,
    32738467, 16657861, 28138468, 501067, 16015129, 21620241, 1529124, 28870577, 18285219, 3050639, 27906268, 21100145, 5343767, 15280732, 12498775, 17354707,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177};

__device__ int32_t zetas_inv_n761[N_N761 / 3] = {
    4936482, 12941874, 13910465, 33199493, 20172936, 14752366, 7168707, 24295961, 29512094, 20006024, 28150129, 23863466, 5790148, 2441259, 26407211, 28022351,
    24560419, 8395453, 12553039, 10228345, 584182, 27332300, 22819644, 12276016, 476468, 24757834, 15631834, 23228604, 7192988, 29800114, 26258901, 31299552,
    26499615, 15301221, 17495707, 8448824, 26202716, 18337996, 12408830, 996257, 7467341, 15476051, 5492562, 30005931, 10673769, 6811853, 9595256, 9331981,
    15966818, 16231856, 27412492, 11059881, 28710073, 23987117, 15764394, 27006783, 27593250, 11396388, 23461689, 24061947, 26801289, 10657979, 888583, 25685226,
    8436014, 26266384, 46120, 20985077, 21309313, 9445200, 18429480, 32210258, 6681561, 3320266, 8152825, 28274051, 30056502, 6554255, 26141027, 32424294,
    28595224, 5650247, 14784739, 33001057, 31218903, 2391744, 9821018, 16583057, 5236813, 16739306, 7170990, 32156118, 29582106, 15015034, 10798836, 23383299,
    18068068, 16494188, 8860636, 9193484, 9297256, 21936528, 1295800, 2136874, 23926560, 2294007, 9587262, 4264447, 26638771, 1831143, 16675504, 22854030,
    1222011, 26968946, 27668148, 11970518, 1993842, 10204060, 890780, 9853701, 22258062, 14228138, 31118264, 21677573, 10867588, 12246672, 2474819, 27957418,
    274280, 27204547, 20465111, 26591536, 21783198, 23281995, 20438788, 1594993, 30095497, 5168890, 27689771, 22563580, 29787612, 7503285, 11189900, 31631832,
    2580934, 26966511, 15794395, 10350944, 16592269, 2305643, 31275530, 18520986, 2694593, 26451815, 6406585, 20010985, 17606226, 2737166, 20565648, 14317088,
    11520949, 7971751, 17461113, 385782, 21586133, 2174608, 971182, 16930314, 24106179, 30598763, 33254929, 4158217, 28869312, 32237843, 25119048, 5167339,
    29092517, 24807116, 27320357, 356339, 21156125, 8580918, 189149, 4757561, 5788647, 7927855, 21647366, 7592012, 285227, 33548073, 1512189, 23129892,
    9217272, 13756690, 25090164, 15102680, 13533911, 32084204, 3925647, 22891714, 15991461, 10667325, 26566369, 9277376, 16908244, 22122426, 23808311, 16367815,
    31034804, 15608693, 18170667, 10496054, 32675015, 15004011, 27482591, 8969567, 14086122, 2292012, 9986178, 17473710, 22685269, 44602, 21453362, 23754885,
    16195630, 21051562, 18269605, 28206570, 12450192, 5644069, 30499698, 15265118, 4679760, 32021213, 11930096, 17535208, 33049270, 5411869, 16892476, 811870,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    274280, 27204547, 20465111, 26591536, 21783198, 23281995, 20438788, 1594993, 30095497, 5168890, 27689771, 22563580, 29787612, 7503285, 11189900, 31631832,
    2580934, 26966511, 15794395, 10350944, 16592269, 2305643, 31275530, 18520986, 2694593, 26451815, 6406585, 20010985, 17606226, 2737166, 20565648, 14317088,
    11520949, 7971751, 17461113, 385782, 21586133, 2174608, 971182, 16930314, 24106179, 30598763, 33254929, 4158217, 28869312, 32237843, 25119048, 5167339,
    29092517, 24807116, 27320357, 356339, 21156125, 8580918, 189149, 4757561, 5788647, 7927855, 21647366, 7592012, 285227, 33548073, 1512189, 23129892,
    9217272, 13756690, 25090164, 15102680, 13533911, 32084204, 3925647, 22891714, 15991461, 10667325, 26566369, 9277376, 16908244, 22122426, 23808311, 16367815,
    31034804, 15608693, 18170667, 10496054, 32675015, 15004011, 27482591, 8969567, 14086122, 2292012, 9986178, 17473710, 22685269, 44602, 21453362, 23754885,
    16195630, 21051562, 18269605, 28206570, 12450192, 5644069, 30499698, 15265118, 4679760, 32021213, 11930096, 17535208, 33049270, 5411869, 16892476, 811870,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    9217272, 13756690, 25090164, 15102680, 13533911, 32084204, 3925647, 22891714, 15991461, 10667325, 26566369, 9277376, 16908244, 22122426, 23808311, 16367815,
    31034804, 15608693, 18170667, 10496054, 32675015, 15004011, 27482591, 8969567, 14086122, 2292012, 9986178, 17473710, 22685269, 44602, 21453362, 23754885,
    16195630, 21051562, 18269605, 28206570, 12450192, 5644069, 30499698, 15265118, 4679760, 32021213, 11930096, 17535208, 33049270, 5411869, 16892476, 811870,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    16195630, 21051562, 18269605, 28206570, 12450192, 5644069, 30499698, 15265118, 4679760, 32021213, 11930096, 17535208, 33049270, 5411869, 16892476, 811870,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177, 11611772, 20051184, 6759380, 33026177, 6759380, 33026177, 33026177};


#define special_CT_a(i,j)  t = a[j];a[j]=(a[i] + t);a[i] = (a[i] - t);
#define special_CT_b(i,j)  t = b[j];b[j]=(b[i] + t);b[i] = (b[i] - t);

#define pse_CT_a(i,j)  t = a[j];a[j]=(a[i] + t);a[i] = pseudomersenne_reduce_single_n761_cuda(a[i] - t);
#define pse_CT_b(i,j)  t = b[j];b[j]=(b[i] + t);b[i] = pseudomersenne_reduce_single_n761_cuda(b[i] - t);

#define CT_a(i,j,zeta)  t = montgomery_reduce_n761_cuda((int64_t)zeta * a[j]); a[j] = (a[i] - t); a[i] = (a[i] + t);

#define CT_b(i,j,zeta)  t = montgomery_reduce_n761_cuda((int64_t)zeta * b[j]); b[j] = (b[i] - t); b[i] = (b[i] + t);

#define GS_a(i,j,zeta) t = a[i]; a[i] = (t + a[j]); a[j] = (t - a[j]); a[j] = montgomery_reduce_n761_cuda((int64_t)zeta * a[j]);

#define KARA(a, b, x, y, d) ((montgomery_reduce_n761_cuda((int64_t)(a[x] + a[y]) * (b[x] + b[y])) - d[x] - d[y]))

__device__ int32_t montgomery_reduce_n761_cuda(int64_t a)
{
    int32_t t;
    t = (int32_t)a * NUMBER_QCMOSSH;
    t = (a - (int64_t)t * Q_N761) >> 32;
    return t;
}

__device__ int32_t pseudomersenne_reduce_single_n761_cuda(int32_t a)
{
    int32_t t0, t1;
    t0 = a & 0x1ffffff;
    t1 = a >> 25;
    t0 = (t1 << 12) + t0 - t1 - Q_N761;
    return t0;
}

__device__ void basemul_n761_cuda(int32_t c[3], const int32_t a[3], const int32_t b[3], const int32_t zeta)
{
    int i;
    int32_t d[3];
    int32_t cc[3];

    for (i = 0; i < 3; i++)
        d[i] = montgomery_reduce_n761_cuda((int64_t)a[i] * b[i]);

    cc[0] = (d[0] + montgomery_reduce_n761_cuda((int64_t)zeta * (KARA(a, b, 1, 2, d))));
    cc[1] = (KARA(a, b, 0, 1, d) + montgomery_reduce_n761_cuda((int64_t)zeta * (d[2])));
    cc[2] = (KARA(a, b, 0, 2, d) + d[1]);

    c[0] = cc[0],c[1]=cc[1],c[2]=cc[2];
}



__global__ void poly_mul_761_batch_q1(poly * array_c,poly * array_a,poly * array_b){
    __shared__ nttpoly_n761 ntta;
    __shared__ nttpoly_n761 nttb;
    __shared__ nttpoly_n761 nttc;

    int32_t a[8]={0};
    int32_t b[8]={0};

    int32_t zeta;
    int32_t t;

    a[0] = array_a[blockIdx.x].coeffs[threadIdx.x];
    a[1] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    a[2] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];

    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) a[3] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];


    b[0] = array_b[blockIdx.x].coeffs[threadIdx.x];
    b[1] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    b[2] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];

    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) b[3] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];


    //1 基2
    special_CT_a(0,4);
    special_CT_a(1,5);
    special_CT_a(2,6);
    special_CT_a(3,7);

    special_CT_b(0,4);
    special_CT_b(1,5);
    special_CT_b(2,6);
    special_CT_b(3,7);

    //1 基2
    zeta = zetas_n761[2];

    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    pse_CT_a(4,6);
    pse_CT_a(5,7);

    pse_CT_b(4,6);
    pse_CT_b(5,7);

    //3 基2
    zeta = zetas_n761[4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[5];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[6];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    pse_CT_a(6,7);
    pse_CT_b(6,7);

    //放回共享内存
    ntta.coeffs[threadIdx.x] = a[0];
    ntta.coeffs[threadIdx.x + 192] = a[1];
    ntta.coeffs[threadIdx.x + 192 * 2] = a[2];
    ntta.coeffs[threadIdx.x + 192 * 3] = a[3];
    ntta.coeffs[threadIdx.x + 192 * 4] = a[4];
    ntta.coeffs[threadIdx.x + 192 * 5] = a[5];
    ntta.coeffs[threadIdx.x + 192 * 6] = a[6];
    ntta.coeffs[threadIdx.x + 192 * 7] = a[7];

    nttb.coeffs[threadIdx.x] = b[0];
    nttb.coeffs[threadIdx.x + 192] = b[1];
    nttb.coeffs[threadIdx.x + 192 * 2] = b[2];
    nttb.coeffs[threadIdx.x + 192 * 3] = b[3];
    nttb.coeffs[threadIdx.x + 192 * 4] = b[4];
    nttb.coeffs[threadIdx.x + 192 * 5] = b[5];
    nttb.coeffs[threadIdx.x + 192 * 6] = b[6];
    nttb.coeffs[threadIdx.x + 192 * 7] = b[7];

    __syncthreads();

    int group_num = threadIdx.x / 24;
    int group_idx = threadIdx.x % 24;
    int start = group_num * 192;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 24];
    a[2] = ntta.coeffs[start + group_idx + 24 * 2];
    a[3] = ntta.coeffs[start + group_idx + 24 * 3];
    a[4] = ntta.coeffs[start + group_idx + 24 * 4];
    a[5] = ntta.coeffs[start + group_idx + 24 * 5];
    a[6] = ntta.coeffs[start + group_idx + 24 * 6];
    a[7] = ntta.coeffs[start + group_idx + 24 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 24];
    b[2] = nttb.coeffs[start + group_idx + 24 * 2];
    b[3] = nttb.coeffs[start + group_idx + 24 * 3];
    b[4] = nttb.coeffs[start + group_idx + 24 * 4];
    b[5] = nttb.coeffs[start + group_idx + 24 * 5];
    b[6] = nttb.coeffs[start + group_idx + 24 * 6];
    b[7] = nttb.coeffs[start + group_idx + 24 * 7];


    //4 基2
    zeta = zetas_n761[8 + group_num];
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);

    //5 基2
    zeta = zetas_n761[16 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n761[16 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    //6 基2
    zeta = zetas_n761[32 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 24] = a[1];
    ntta.coeffs[start + group_idx + 24 * 2] = a[2];
    ntta.coeffs[start + group_idx + 24 * 3] = a[3];
    ntta.coeffs[start + group_idx + 24 * 4] = a[4];
    ntta.coeffs[start + group_idx + 24 * 5] = a[5];
    ntta.coeffs[start + group_idx + 24 * 6] = a[6];
    ntta.coeffs[start + group_idx + 24 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 24] = b[1];
    nttb.coeffs[start + group_idx + 24 * 2] = b[2];
    nttb.coeffs[start + group_idx + 24 * 3] = b[3];
    nttb.coeffs[start + group_idx + 24 * 4] = b[4];
    nttb.coeffs[start + group_idx + 24 * 5] = b[5];
    nttb.coeffs[start + group_idx + 24 * 6] = b[6];
    nttb.coeffs[start + group_idx + 24 * 7] = b[7];

    __syncthreads();

    group_num = threadIdx.x / 3;
    group_idx = threadIdx.x % 3;
    start = group_num * 24;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 3];
    a[2] = ntta.coeffs[start + group_idx + 3 * 2];
    a[3] = ntta.coeffs[start + group_idx + 3 * 3];
    a[4] = ntta.coeffs[start + group_idx + 3 * 4];
    a[5] = ntta.coeffs[start + group_idx + 3 * 5];
    a[6] = ntta.coeffs[start + group_idx + 3 * 6];
    a[7] = ntta.coeffs[start + group_idx + 3 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 3];
    b[2] = nttb.coeffs[start + group_idx + 3 * 2];
    b[3] = nttb.coeffs[start + group_idx + 3 * 3];
    b[4] = nttb.coeffs[start + group_idx + 3 * 4];
    b[5] = nttb.coeffs[start + group_idx + 3 * 5];
    b[6] = nttb.coeffs[start + group_idx + 3 * 6];
    b[7] = nttb.coeffs[start + group_idx + 3 * 7];

    //7 基2
    zeta = zetas_n761[64 + group_num];
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);

    //8 基2
    zeta = zetas_n761[128 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n761[128 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    //9 基2
    zeta = zetas_n761[256 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 3] = a[1];
    ntta.coeffs[start + group_idx + 3 * 2] = a[2];
    ntta.coeffs[start + group_idx + 3 * 3] = a[3];
    ntta.coeffs[start + group_idx + 3 * 4] = a[4];
    ntta.coeffs[start + group_idx + 3 * 5] = a[5];
    ntta.coeffs[start + group_idx + 3 * 6] = a[6];
    ntta.coeffs[start + group_idx + 3 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 3] = b[1];
    nttb.coeffs[start + group_idx + 3 * 2] = b[2];
    nttb.coeffs[start + group_idx + 3 * 3] = b[3];
    nttb.coeffs[start + group_idx + 3 * 4] = b[4];
    nttb.coeffs[start + group_idx + 3 * 5] = b[5];
    nttb.coeffs[start + group_idx + 3 * 6] = b[6];
    nttb.coeffs[start + group_idx + 3 * 7] = b[7];
    
    __syncthreads();
    
    //512 个 basemul

    //0-191个
    zeta = zetas_n761[256 + threadIdx.x / 2];
    if(threadIdx.x & 1) zeta = -zeta;
    basemul_n761_cuda(nttc.coeffs + 3 * threadIdx.x, ntta.coeffs + 3 * threadIdx.x,nttb.coeffs + 3 * threadIdx.x,zeta);

    //192 - 383
    zeta = zetas_n761[256 + threadIdx.x / 2 + 96];
    if(threadIdx.x & 1) zeta = -zeta;
    basemul_n761_cuda(nttc.coeffs + 3 * threadIdx.x + 576, ntta.coeffs + 3 * threadIdx.x + 576, nttb.coeffs + 3 * threadIdx.x + 576, zeta);

    if(threadIdx.x < 128){
        zeta = zetas_n761[256 + threadIdx.x / 2 + 192];
        if(threadIdx.x & 1) zeta = -zeta;
        basemul_n761_cuda(nttc.coeffs + 3 * threadIdx.x + 1152, ntta.coeffs + 3 * threadIdx.x + 1152, nttb.coeffs + 3 * threadIdx.x + 1152, zeta);
    }

    __syncthreads();


    //inv_ntt
    group_num = threadIdx.x / 3;
    group_idx = threadIdx.x % 3;
    start = group_num * 24;

    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 3];
    a[2] = nttc.coeffs[start + group_idx + 3 * 2];
    a[3] = nttc.coeffs[start + group_idx + 3 * 3];
    a[4] = nttc.coeffs[start + group_idx + 3 * 4];
    a[5] = nttc.coeffs[start + group_idx + 3 * 5];
    a[6] = nttc.coeffs[start + group_idx + 3 * 6];
    a[7] = nttc.coeffs[start + group_idx + 3 * 7];


    //1 基2 256
    zeta = zetas_inv_n761[group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 3];
    GS_a(6,7,zeta);

    //2 基2 128
    zeta = zetas_inv_n761[256 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[256 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //3 基2 64
    zeta = zetas_inv_n761[384 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 3] = a[1];
    nttc.coeffs[start + group_idx + 3 * 2] = a[2];
    nttc.coeffs[start + group_idx + 3 * 3] = a[3];
    nttc.coeffs[start + group_idx + 3 * 4] = a[4];
    nttc.coeffs[start + group_idx + 3 * 5] = a[5];
    nttc.coeffs[start + group_idx + 3 * 6] = a[6];
    nttc.coeffs[start + group_idx + 3 * 7] = a[7];

    __syncthreads();

    group_num = threadIdx.x / 24;
    group_idx = threadIdx.x % 24;
    start = group_num * 192;

    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 24];
    a[2] = nttc.coeffs[start + group_idx + 24 * 2];
    a[3] = nttc.coeffs[start + group_idx + 24 * 3];
    a[4] = nttc.coeffs[start + group_idx + 24 * 4];
    a[5] = nttc.coeffs[start + group_idx + 24 * 5];
    a[6] = nttc.coeffs[start + group_idx + 24 * 6];
    a[7] = nttc.coeffs[start + group_idx + 24 * 7];

    //4 基2 32
    zeta = zetas_inv_n761[448 + group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 3];
    GS_a(6,7,zeta);

#pragma unroll
    for(int i=0;i<8;i++){
        pseudomersenne_reduce_single_n761_cuda(a[i]);
        pseudomersenne_reduce_single_n761_cuda(b[i]);
    }
    
    //5 基2 16
    zeta = zetas_inv_n761[480 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[480 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //6 基2 8
    zeta = zetas_inv_n761[496 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 24] = a[1];
    nttc.coeffs[start + group_idx + 24 * 2] = a[2];
    nttc.coeffs[start + group_idx + 24 * 3] = a[3];
    nttc.coeffs[start + group_idx + 24 * 4] = a[4];
    nttc.coeffs[start + group_idx + 24 * 5] = a[5];
    nttc.coeffs[start + group_idx + 24 * 6] = a[6];
    nttc.coeffs[start + group_idx + 24 * 7] = a[7];

    __syncthreads();
    

    a[0] = nttc.coeffs[threadIdx.x];
    a[1] = nttc.coeffs[threadIdx.x + 192];
    a[2] = nttc.coeffs[threadIdx.x + 192 * 2];
    a[3] = nttc.coeffs[threadIdx.x + 192 * 3];
    a[4] = nttc.coeffs[threadIdx.x + 192 * 4];
    a[5] = nttc.coeffs[threadIdx.x + 192 * 5];
    a[6] = nttc.coeffs[threadIdx.x + 192 * 6];
    a[7] = nttc.coeffs[threadIdx.x + 192 * 7];

    //7 基2 4
    zeta = zetas_inv_n761[504];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[504 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[504 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[504 + 3];
    GS_a(6,7,zeta);

    //8 基2 2
    zeta = zetas_inv_n761[508];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[508 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //9 基2 1
    zeta = zetas_inv_n761[510];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[threadIdx.x] = a[0];
    nttc.coeffs[threadIdx.x + 192] = a[1];
    nttc.coeffs[threadIdx.x + 192 * 2] = a[2];
    nttc.coeffs[threadIdx.x + 192 * 3] = a[3];
    nttc.coeffs[threadIdx.x + 192 * 4] = a[4];
    nttc.coeffs[threadIdx.x + 192 * 5] = a[5];
    nttc.coeffs[threadIdx.x + 192 * 6] = a[6];
    nttc.coeffs[threadIdx.x + 192 * 7] = a[7];

    __syncthreads();
    
    //1 - 192
    array_c[blockIdx.x].coeffs[threadIdx.x + 1] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[threadIdx.x + 1] + nttc.coeffs[threadIdx.x + 1 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 1 + FPTRU_N])));
    
    //193 - 384
    array_c[blockIdx.x].coeffs[threadIdx.x + 193] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[threadIdx.x + 193] + nttc.coeffs[threadIdx.x + 193 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 193 + FPTRU_N])));

    //385 - 576
    array_c[blockIdx.x].coeffs[threadIdx.x + 385] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[threadIdx.x + 385] + nttc.coeffs[threadIdx.x + 385 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 385 + FPTRU_N])));
    
    //0,760, 577-759
    if(threadIdx.x <= 183){
        if(threadIdx.x == 183){
            array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
            array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
        }
        else{
            array_c[blockIdx.x].coeffs[threadIdx.x + 577] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[threadIdx.x + 577] + nttc.coeffs[threadIdx.x + 577 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 577 + FPTRU_N])));
        }
    }

}


__global__ void poly_mul_761_batch_q2(poly * array_c,poly * array_a,poly * array_b){
    __shared__ nttpoly_n761 ntta;
    __shared__ nttpoly_n761 nttb;
    __shared__ nttpoly_n761 nttc;

    int32_t a[8]={0};
    int32_t b[8]={0};

    int32_t zeta;
    int32_t t;

    a[0] = array_a[blockIdx.x].coeffs[threadIdx.x];
    a[1] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    a[2] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];

    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) a[3] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];
    // else a[3] = 0;
    // a[4] = 0;
    // a[5] = 0;
    // a[6] = 0;
    // a[7] = 0;



    b[0] = array_b[blockIdx.x].coeffs[threadIdx.x];
    b[1] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    b[2] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];

    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) b[3] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];
    // else b[3] = 0;
    // b[4] = 0;
    // b[5] = 0;
    // b[6] = 0;
    // b[7] = 0;


    //1 基2
    special_CT_a(0,4);
    special_CT_a(1,5);
    special_CT_a(2,6);
    special_CT_a(3,7);

    special_CT_b(0,4);
    special_CT_b(1,5);
    special_CT_b(2,6);
    special_CT_b(3,7);

    //1 基2
    zeta = zetas_n761[2];

    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    pse_CT_a(4,6);
    pse_CT_a(5,7);

    pse_CT_b(4,6);
    pse_CT_b(5,7);

    //3 基2
    zeta = zetas_n761[4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[5];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[6];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    pse_CT_a(6,7);
    pse_CT_b(6,7);

    //放回共享内存
    ntta.coeffs[threadIdx.x] = a[0];
    ntta.coeffs[threadIdx.x + 192] = a[1];
    ntta.coeffs[threadIdx.x + 192 * 2] = a[2];
    ntta.coeffs[threadIdx.x + 192 * 3] = a[3];
    ntta.coeffs[threadIdx.x + 192 * 4] = a[4];
    ntta.coeffs[threadIdx.x + 192 * 5] = a[5];
    ntta.coeffs[threadIdx.x + 192 * 6] = a[6];
    ntta.coeffs[threadIdx.x + 192 * 7] = a[7];

    nttb.coeffs[threadIdx.x] = b[0];
    nttb.coeffs[threadIdx.x + 192] = b[1];
    nttb.coeffs[threadIdx.x + 192 * 2] = b[2];
    nttb.coeffs[threadIdx.x + 192 * 3] = b[3];
    nttb.coeffs[threadIdx.x + 192 * 4] = b[4];
    nttb.coeffs[threadIdx.x + 192 * 5] = b[5];
    nttb.coeffs[threadIdx.x + 192 * 6] = b[6];
    nttb.coeffs[threadIdx.x + 192 * 7] = b[7];

    __syncthreads();

    int group_num = threadIdx.x / 24;
    int group_idx = threadIdx.x % 24;
    int start = group_num * 192;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 24];
    a[2] = ntta.coeffs[start + group_idx + 24 * 2];
    a[3] = ntta.coeffs[start + group_idx + 24 * 3];
    a[4] = ntta.coeffs[start + group_idx + 24 * 4];
    a[5] = ntta.coeffs[start + group_idx + 24 * 5];
    a[6] = ntta.coeffs[start + group_idx + 24 * 6];
    a[7] = ntta.coeffs[start + group_idx + 24 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 24];
    b[2] = nttb.coeffs[start + group_idx + 24 * 2];
    b[3] = nttb.coeffs[start + group_idx + 24 * 3];
    b[4] = nttb.coeffs[start + group_idx + 24 * 4];
    b[5] = nttb.coeffs[start + group_idx + 24 * 5];
    b[6] = nttb.coeffs[start + group_idx + 24 * 6];
    b[7] = nttb.coeffs[start + group_idx + 24 * 7];


    //4 基2
    zeta = zetas_n761[8 + group_num];
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);

    //5 基2
    zeta = zetas_n761[16 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n761[16 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    //6 基2
    zeta = zetas_n761[32 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 24] = a[1];
    ntta.coeffs[start + group_idx + 24 * 2] = a[2];
    ntta.coeffs[start + group_idx + 24 * 3] = a[3];
    ntta.coeffs[start + group_idx + 24 * 4] = a[4];
    ntta.coeffs[start + group_idx + 24 * 5] = a[5];
    ntta.coeffs[start + group_idx + 24 * 6] = a[6];
    ntta.coeffs[start + group_idx + 24 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 24] = b[1];
    nttb.coeffs[start + group_idx + 24 * 2] = b[2];
    nttb.coeffs[start + group_idx + 24 * 3] = b[3];
    nttb.coeffs[start + group_idx + 24 * 4] = b[4];
    nttb.coeffs[start + group_idx + 24 * 5] = b[5];
    nttb.coeffs[start + group_idx + 24 * 6] = b[6];
    nttb.coeffs[start + group_idx + 24 * 7] = b[7];

    __syncthreads();

    group_num = threadIdx.x / 3;
    group_idx = threadIdx.x % 3;
    start = group_num * 24;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 3];
    a[2] = ntta.coeffs[start + group_idx + 3 * 2];
    a[3] = ntta.coeffs[start + group_idx + 3 * 3];
    a[4] = ntta.coeffs[start + group_idx + 3 * 4];
    a[5] = ntta.coeffs[start + group_idx + 3 * 5];
    a[6] = ntta.coeffs[start + group_idx + 3 * 6];
    a[7] = ntta.coeffs[start + group_idx + 3 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 3];
    b[2] = nttb.coeffs[start + group_idx + 3 * 2];
    b[3] = nttb.coeffs[start + group_idx + 3 * 3];
    b[4] = nttb.coeffs[start + group_idx + 3 * 4];
    b[5] = nttb.coeffs[start + group_idx + 3 * 5];
    b[6] = nttb.coeffs[start + group_idx + 3 * 6];
    b[7] = nttb.coeffs[start + group_idx + 3 * 7];

    //7 基2
    zeta = zetas_n761[64 + group_num];
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);

    //8 基2
    zeta = zetas_n761[128 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n761[128 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    //9 基2
    zeta = zetas_n761[256 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 3] = a[1];
    ntta.coeffs[start + group_idx + 3 * 2] = a[2];
    ntta.coeffs[start + group_idx + 3 * 3] = a[3];
    ntta.coeffs[start + group_idx + 3 * 4] = a[4];
    ntta.coeffs[start + group_idx + 3 * 5] = a[5];
    ntta.coeffs[start + group_idx + 3 * 6] = a[6];
    ntta.coeffs[start + group_idx + 3 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 3] = b[1];
    nttb.coeffs[start + group_idx + 3 * 2] = b[2];
    nttb.coeffs[start + group_idx + 3 * 3] = b[3];
    nttb.coeffs[start + group_idx + 3 * 4] = b[4];
    nttb.coeffs[start + group_idx + 3 * 5] = b[5];
    nttb.coeffs[start + group_idx + 3 * 6] = b[6];
    nttb.coeffs[start + group_idx + 3 * 7] = b[7];
    
    __syncthreads();
    // printf("NTTA result\n");
    // if(threadIdx.x == 0 && blockIdx.x == 0){
        //printf("NTTA result\n");
        //int i = 1 + 1;不行还是会出错 zhc4.txt
        // for(int i=0;i<N_N761;i++){
        //     printf("%d,",ntta.coeffs[i]);
        // }
        // printf("\n\n");
        // printf("NTTB result\n");
        // for(int i=0;i<N_N761;i++){
        //     printf("%d,",nttb.coeffs[i]);
        // }
        // printf("\n\n");
    //}
    //__syncthreads();
    //512 个 basemul

    //0-191个
    zeta = zetas_n761[256 + threadIdx.x / 2];
    if(threadIdx.x & 1) zeta = -zeta;
    basemul_n761_cuda(nttc.coeffs + 3 * threadIdx.x, ntta.coeffs + 3 * threadIdx.x,nttb.coeffs + 3 * threadIdx.x,zeta);

    //192 - 383
    zeta = zetas_n761[256 + threadIdx.x / 2 + 96];
    if(threadIdx.x & 1) zeta = -zeta;
    basemul_n761_cuda(nttc.coeffs + 3 * threadIdx.x + 576, ntta.coeffs + 3 * threadIdx.x + 576, nttb.coeffs + 3 * threadIdx.x + 576, zeta);

    if(threadIdx.x < 128){
        zeta = zetas_n761[256 + threadIdx.x / 2 + 192];
        if(threadIdx.x & 1) zeta = -zeta;
        basemul_n761_cuda(nttc.coeffs + 3 * threadIdx.x + 1152, ntta.coeffs + 3 * threadIdx.x + 1152, nttb.coeffs + 3 * threadIdx.x + 1152, zeta);
    }

    __syncthreads();


    //inv_ntt
    group_num = threadIdx.x / 3;
    group_idx = threadIdx.x % 3;
    start = group_num * 24;

    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 3];
    a[2] = nttc.coeffs[start + group_idx + 3 * 2];
    a[3] = nttc.coeffs[start + group_idx + 3 * 3];
    a[4] = nttc.coeffs[start + group_idx + 3 * 4];
    a[5] = nttc.coeffs[start + group_idx + 3 * 5];
    a[6] = nttc.coeffs[start + group_idx + 3 * 6];
    a[7] = nttc.coeffs[start + group_idx + 3 * 7];


    //1 基2 256
    zeta = zetas_inv_n761[group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 3];
    GS_a(6,7,zeta);

    //2 基2 128
    zeta = zetas_inv_n761[256 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[256 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //3 基2 64
    zeta = zetas_inv_n761[384 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 3] = a[1];
    nttc.coeffs[start + group_idx + 3 * 2] = a[2];
    nttc.coeffs[start + group_idx + 3 * 3] = a[3];
    nttc.coeffs[start + group_idx + 3 * 4] = a[4];
    nttc.coeffs[start + group_idx + 3 * 5] = a[5];
    nttc.coeffs[start + group_idx + 3 * 6] = a[6];
    nttc.coeffs[start + group_idx + 3 * 7] = a[7];

    __syncthreads();

    group_num = threadIdx.x / 24;
    group_idx = threadIdx.x % 24;
    start = group_num * 192;

    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 24];
    a[2] = nttc.coeffs[start + group_idx + 24 * 2];
    a[3] = nttc.coeffs[start + group_idx + 24 * 3];
    a[4] = nttc.coeffs[start + group_idx + 24 * 4];
    a[5] = nttc.coeffs[start + group_idx + 24 * 5];
    a[6] = nttc.coeffs[start + group_idx + 24 * 6];
    a[7] = nttc.coeffs[start + group_idx + 24 * 7];

    //4 基2 32
    zeta = zetas_inv_n761[448 + group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 3];
    GS_a(6,7,zeta);

#pragma unroll
    for(int i=0;i<8;i++){
        pseudomersenne_reduce_single_n761_cuda(a[i]);
        pseudomersenne_reduce_single_n761_cuda(b[i]);
    }
    
    //5 基2 16
    zeta = zetas_inv_n761[480 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[480 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //6 基2 8
    zeta = zetas_inv_n761[496 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 24] = a[1];
    nttc.coeffs[start + group_idx + 24 * 2] = a[2];
    nttc.coeffs[start + group_idx + 24 * 3] = a[3];
    nttc.coeffs[start + group_idx + 24 * 4] = a[4];
    nttc.coeffs[start + group_idx + 24 * 5] = a[5];
    nttc.coeffs[start + group_idx + 24 * 6] = a[6];
    nttc.coeffs[start + group_idx + 24 * 7] = a[7];

    __syncthreads();
    

    a[0] = nttc.coeffs[threadIdx.x];
    a[1] = nttc.coeffs[threadIdx.x + 192];
    a[2] = nttc.coeffs[threadIdx.x + 192 * 2];
    a[3] = nttc.coeffs[threadIdx.x + 192 * 3];
    a[4] = nttc.coeffs[threadIdx.x + 192 * 4];
    a[5] = nttc.coeffs[threadIdx.x + 192 * 5];
    a[6] = nttc.coeffs[threadIdx.x + 192 * 6];
    a[7] = nttc.coeffs[threadIdx.x + 192 * 7];

    //7 基2 4
    zeta = zetas_inv_n761[504];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[504 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[504 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[504 + 3];
    GS_a(6,7,zeta);

    //8 基2 2
    zeta = zetas_inv_n761[508];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[508 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //9 基2 1
    zeta = zetas_inv_n761[510];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[threadIdx.x] = a[0];
    nttc.coeffs[threadIdx.x + 192] = a[1];
    nttc.coeffs[threadIdx.x + 192 * 2] = a[2];
    nttc.coeffs[threadIdx.x + 192 * 3] = a[3];
    nttc.coeffs[threadIdx.x + 192 * 4] = a[4];
    nttc.coeffs[threadIdx.x + 192 * 5] = a[5];
    nttc.coeffs[threadIdx.x + 192 * 6] = a[6];
    nttc.coeffs[threadIdx.x + 192 * 7] = a[7];

    __syncthreads();
    
    //1 - 192
    array_c[blockIdx.x].coeffs[threadIdx.x + 1] = (montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[threadIdx.x + 1] + nttc.coeffs[threadIdx.x + 1 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 1 + FPTRU_N]))) & (FPTRU_Q2 - 1);
    
    //193 - 384
    array_c[blockIdx.x].coeffs[threadIdx.x + 193] = (montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[threadIdx.x + 193] + nttc.coeffs[threadIdx.x + 193 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 193 + FPTRU_N]))) & (FPTRU_Q2 - 1);

    //385 - 576
    array_c[blockIdx.x].coeffs[threadIdx.x + 385] = (montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[threadIdx.x + 385] + nttc.coeffs[threadIdx.x + 385 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 385 + FPTRU_N]))) & (FPTRU_Q2 - 1);;
    
    //0,760, 577-759
    if(threadIdx.x <= 183){
        if(threadIdx.x == 183){
            array_c[blockIdx.x].coeffs[0] = (montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N]))) & (FPTRU_Q2 - 1);
            array_c[blockIdx.x].coeffs[FPTRU_N - 1] = (montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2]))) & (FPTRU_Q2 - 1);
        }
        else{
            array_c[blockIdx.x].coeffs[threadIdx.x + 577] = (montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (nttc.coeffs[threadIdx.x + 577] + nttc.coeffs[threadIdx.x + 577 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 577 + FPTRU_N]))) & (FPTRU_Q2 - 1);;
        }
    }

}




__device__  void d_poly_mul_q1_761(nttpoly_n761 & ntta, nttpoly_n761 & nttb, poly * array_a,poly * array_c){
    int32_t a[8]={0};
    int32_t b[8]={0};

    int32_t zeta;
    int32_t t;

    


    a[0] = array_a[blockIdx.x].coeffs[threadIdx.x];
    a[1] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    a[2] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];

    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) a[3] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];

    b[0] = nttb.coeffs[threadIdx.x];
    b[1] = nttb.coeffs[threadIdx.x + blockDim.x];
    b[2] = nttb.coeffs[threadIdx.x + blockDim.x * 2];
    b[3] = nttb.coeffs[threadIdx.x + blockDim.x * 3];
    b[4] = nttb.coeffs[threadIdx.x + blockDim.x * 4];
    b[5] = nttb.coeffs[threadIdx.x + blockDim.x * 5];
    b[6] = nttb.coeffs[threadIdx.x + blockDim.x * 6];
    b[7] = nttb.coeffs[threadIdx.x + blockDim.x * 7];

    //1 基2
    special_CT_a(0,4);
    special_CT_a(1,5);
    special_CT_a(2,6);
    special_CT_a(3,7);

    special_CT_b(0,4);
    special_CT_b(1,5);
    special_CT_b(2,6);
    special_CT_b(3,7);

    //1 基2
    zeta = zetas_n761[2];

    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    pse_CT_a(4,6);
    pse_CT_a(5,7);

    pse_CT_b(4,6);
    pse_CT_b(5,7);

    //3 基2
    zeta = zetas_n761[4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[5];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[6];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    pse_CT_a(6,7);
    pse_CT_b(6,7);

    //放回共享内存
    ntta.coeffs[threadIdx.x] = a[0];
    ntta.coeffs[threadIdx.x + 192] = a[1];
    ntta.coeffs[threadIdx.x + 192 * 2] = a[2];
    ntta.coeffs[threadIdx.x + 192 * 3] = a[3];
    ntta.coeffs[threadIdx.x + 192 * 4] = a[4];
    ntta.coeffs[threadIdx.x + 192 * 5] = a[5];
    ntta.coeffs[threadIdx.x + 192 * 6] = a[6];
    ntta.coeffs[threadIdx.x + 192 * 7] = a[7];

    nttb.coeffs[threadIdx.x] = b[0];
    nttb.coeffs[threadIdx.x + 192] = b[1];
    nttb.coeffs[threadIdx.x + 192 * 2] = b[2];
    nttb.coeffs[threadIdx.x + 192 * 3] = b[3];
    nttb.coeffs[threadIdx.x + 192 * 4] = b[4];
    nttb.coeffs[threadIdx.x + 192 * 5] = b[5];
    nttb.coeffs[threadIdx.x + 192 * 6] = b[6];
    nttb.coeffs[threadIdx.x + 192 * 7] = b[7];

    __syncthreads();

    int group_num = threadIdx.x / 24;
    int group_idx = threadIdx.x % 24;
    int start = group_num * 192;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 24];
    a[2] = ntta.coeffs[start + group_idx + 24 * 2];
    a[3] = ntta.coeffs[start + group_idx + 24 * 3];
    a[4] = ntta.coeffs[start + group_idx + 24 * 4];
    a[5] = ntta.coeffs[start + group_idx + 24 * 5];
    a[6] = ntta.coeffs[start + group_idx + 24 * 6];
    a[7] = ntta.coeffs[start + group_idx + 24 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 24];
    b[2] = nttb.coeffs[start + group_idx + 24 * 2];
    b[3] = nttb.coeffs[start + group_idx + 24 * 3];
    b[4] = nttb.coeffs[start + group_idx + 24 * 4];
    b[5] = nttb.coeffs[start + group_idx + 24 * 5];
    b[6] = nttb.coeffs[start + group_idx + 24 * 6];
    b[7] = nttb.coeffs[start + group_idx + 24 * 7];


    //4 基2
    zeta = zetas_n761[8 + group_num];
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);

    //5 基2
    zeta = zetas_n761[16 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n761[16 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    //6 基2
    zeta = zetas_n761[32 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n761[32 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 24] = a[1];
    ntta.coeffs[start + group_idx + 24 * 2] = a[2];
    ntta.coeffs[start + group_idx + 24 * 3] = a[3];
    ntta.coeffs[start + group_idx + 24 * 4] = a[4];
    ntta.coeffs[start + group_idx + 24 * 5] = a[5];
    ntta.coeffs[start + group_idx + 24 * 6] = a[6];
    ntta.coeffs[start + group_idx + 24 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 24] = b[1];
    nttb.coeffs[start + group_idx + 24 * 2] = b[2];
    nttb.coeffs[start + group_idx + 24 * 3] = b[3];
    nttb.coeffs[start + group_idx + 24 * 4] = b[4];
    nttb.coeffs[start + group_idx + 24 * 5] = b[5];
    nttb.coeffs[start + group_idx + 24 * 6] = b[6];
    nttb.coeffs[start + group_idx + 24 * 7] = b[7];

    __syncthreads();

    group_num = threadIdx.x / 3;
    group_idx = threadIdx.x % 3;
    start = group_num * 24;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 3];
    a[2] = ntta.coeffs[start + group_idx + 3 * 2];
    a[3] = ntta.coeffs[start + group_idx + 3 * 3];
    a[4] = ntta.coeffs[start + group_idx + 3 * 4];
    a[5] = ntta.coeffs[start + group_idx + 3 * 5];
    a[6] = ntta.coeffs[start + group_idx + 3 * 6];
    a[7] = ntta.coeffs[start + group_idx + 3 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 3];
    b[2] = nttb.coeffs[start + group_idx + 3 * 2];
    b[3] = nttb.coeffs[start + group_idx + 3 * 3];
    b[4] = nttb.coeffs[start + group_idx + 3 * 4];
    b[5] = nttb.coeffs[start + group_idx + 3 * 5];
    b[6] = nttb.coeffs[start + group_idx + 3 * 6];
    b[7] = nttb.coeffs[start + group_idx + 3 * 7];

    //7 基2
    zeta = zetas_n761[64 + group_num];
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);

    //8 基2
    zeta = zetas_n761[128 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n761[128 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    //9 基2
    zeta = zetas_n761[256 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n761[256 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 3] = a[1];
    ntta.coeffs[start + group_idx + 3 * 2] = a[2];
    ntta.coeffs[start + group_idx + 3 * 3] = a[3];
    ntta.coeffs[start + group_idx + 3 * 4] = a[4];
    ntta.coeffs[start + group_idx + 3 * 5] = a[5];
    ntta.coeffs[start + group_idx + 3 * 6] = a[6];
    ntta.coeffs[start + group_idx + 3 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 3] = b[1];
    nttb.coeffs[start + group_idx + 3 * 2] = b[2];
    nttb.coeffs[start + group_idx + 3 * 3] = b[3];
    nttb.coeffs[start + group_idx + 3 * 4] = b[4];
    nttb.coeffs[start + group_idx + 3 * 5] = b[5];
    nttb.coeffs[start + group_idx + 3 * 6] = b[6];
    nttb.coeffs[start + group_idx + 3 * 7] = b[7];
    
    __syncthreads();
    
    //512 个 basemul

    //0-191个
    zeta = zetas_n761[256 + threadIdx.x / 2];
    if(threadIdx.x & 1) zeta = -zeta;
    basemul_n761_cuda(ntta.coeffs + 3 * threadIdx.x, ntta.coeffs + 3 * threadIdx.x,nttb.coeffs + 3 * threadIdx.x,zeta);

    //192 - 383
    zeta = zetas_n761[256 + threadIdx.x / 2 + 96];
    if(threadIdx.x & 1) zeta = -zeta;
    basemul_n761_cuda(ntta.coeffs + 3 * threadIdx.x + 576, ntta.coeffs + 3 * threadIdx.x + 576, nttb.coeffs + 3 * threadIdx.x + 576, zeta);

    if(threadIdx.x < 128){
        zeta = zetas_n761[256 + threadIdx.x / 2 + 192];
        if(threadIdx.x & 1) zeta = -zeta;
        basemul_n761_cuda(ntta.coeffs + 3 * threadIdx.x + 1152, ntta.coeffs + 3 * threadIdx.x + 1152, nttb.coeffs + 3 * threadIdx.x + 1152, zeta);
    }

    __syncthreads();


    //inv_ntt
    group_num = threadIdx.x / 3;
    group_idx = threadIdx.x % 3;
    start = group_num * 24;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 3];
    a[2] = ntta.coeffs[start + group_idx + 3 * 2];
    a[3] = ntta.coeffs[start + group_idx + 3 * 3];
    a[4] = ntta.coeffs[start + group_idx + 3 * 4];
    a[5] = ntta.coeffs[start + group_idx + 3 * 5];
    a[6] = ntta.coeffs[start + group_idx + 3 * 6];
    a[7] = ntta.coeffs[start + group_idx + 3 * 7];


    //1 基2 256
    zeta = zetas_inv_n761[group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[group_num * 4 + 3];
    GS_a(6,7,zeta);

    //2 基2 128
    zeta = zetas_inv_n761[256 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[256 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //3 基2 64
    zeta = zetas_inv_n761[384 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 3] = a[1];
    ntta.coeffs[start + group_idx + 3 * 2] = a[2];
    ntta.coeffs[start + group_idx + 3 * 3] = a[3];
    ntta.coeffs[start + group_idx + 3 * 4] = a[4];
    ntta.coeffs[start + group_idx + 3 * 5] = a[5];
    ntta.coeffs[start + group_idx + 3 * 6] = a[6];
    ntta.coeffs[start + group_idx + 3 * 7] = a[7];

    __syncthreads();

    group_num = threadIdx.x / 24;
    group_idx = threadIdx.x % 24;
    start = group_num * 192;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 24];
    a[2] = ntta.coeffs[start + group_idx + 24 * 2];
    a[3] = ntta.coeffs[start + group_idx + 24 * 3];
    a[4] = ntta.coeffs[start + group_idx + 24 * 4];
    a[5] = ntta.coeffs[start + group_idx + 24 * 5];
    a[6] = ntta.coeffs[start + group_idx + 24 * 6];
    a[7] = ntta.coeffs[start + group_idx + 24 * 7];

    //4 基2 32
    zeta = zetas_inv_n761[448 + group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[448 + group_num * 4 + 3];
    GS_a(6,7,zeta);

#pragma unroll
    for(int i=0;i<8;i++){
        pseudomersenne_reduce_single_n761_cuda(a[i]);
        pseudomersenne_reduce_single_n761_cuda(b[i]);
    }
    
    //5 基2 16
    zeta = zetas_inv_n761[480 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[480 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //6 基2 8
    zeta = zetas_inv_n761[496 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 24] = a[1];
    ntta.coeffs[start + group_idx + 24 * 2] = a[2];
    ntta.coeffs[start + group_idx + 24 * 3] = a[3];
    ntta.coeffs[start + group_idx + 24 * 4] = a[4];
    ntta.coeffs[start + group_idx + 24 * 5] = a[5];
    ntta.coeffs[start + group_idx + 24 * 6] = a[6];
    ntta.coeffs[start + group_idx + 24 * 7] = a[7];

    __syncthreads();
    

    a[0] = ntta.coeffs[threadIdx.x];
    a[1] = ntta.coeffs[threadIdx.x + 192];
    a[2] = ntta.coeffs[threadIdx.x + 192 * 2];
    a[3] = ntta.coeffs[threadIdx.x + 192 * 3];
    a[4] = ntta.coeffs[threadIdx.x + 192 * 4];
    a[5] = ntta.coeffs[threadIdx.x + 192 * 5];
    a[6] = ntta.coeffs[threadIdx.x + 192 * 6];
    a[7] = ntta.coeffs[threadIdx.x + 192 * 7];

    //7 基2 4
    zeta = zetas_inv_n761[504];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n761[504 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n761[504 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n761[504 + 3];
    GS_a(6,7,zeta);

    //8 基2 2
    zeta = zetas_inv_n761[508];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n761[508 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //9 基2 1
    zeta = zetas_inv_n761[510];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    ntta.coeffs[threadIdx.x] = a[0];
    ntta.coeffs[threadIdx.x + 192] = a[1];
    ntta.coeffs[threadIdx.x + 192 * 2] = a[2];
    ntta.coeffs[threadIdx.x + 192 * 3] = a[3];
    ntta.coeffs[threadIdx.x + 192 * 4] = a[4];
    ntta.coeffs[threadIdx.x + 192 * 5] = a[5];
    ntta.coeffs[threadIdx.x + 192 * 6] = a[6];
    ntta.coeffs[threadIdx.x + 192 * 7] = a[7];

    __syncthreads();
    
    //1 - 192
    array_c[blockIdx.x].coeffs[threadIdx.x + 1] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (ntta.coeffs[threadIdx.x + 1] + ntta.coeffs[threadIdx.x + 1 + FPTRU_N - 1] + ntta.coeffs[threadIdx.x + 1 + FPTRU_N])));
    
    //193 - 384
    array_c[blockIdx.x].coeffs[threadIdx.x + 193] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (ntta.coeffs[threadIdx.x + 193] + ntta.coeffs[threadIdx.x + 193 + FPTRU_N - 1] + ntta.coeffs[threadIdx.x + 193 + FPTRU_N])));

    //385 - 576
    array_c[blockIdx.x].coeffs[threadIdx.x + 385] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (ntta.coeffs[threadIdx.x + 385] + ntta.coeffs[threadIdx.x + 385 + FPTRU_N - 1] + ntta.coeffs[threadIdx.x + 385 + FPTRU_N])));
    
    //0,760, 577-759
    if(threadIdx.x <= 183){
        if(threadIdx.x == 183){
            array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (ntta.coeffs[0] + ntta.coeffs[FPTRU_N])));
            array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (ntta.coeffs[FPTRU_N - 1] + ntta.coeffs[2 * FPTRU_N - 2])));
        }
        else{
            array_c[blockIdx.x].coeffs[threadIdx.x + 577] = fq_freeze(montgomery_reduce_n761_cuda((int64_t)FACTOR_JAOHADAH * (ntta.coeffs[threadIdx.x + 577] + ntta.coeffs[threadIdx.x + 577 + FPTRU_N - 1] + ntta.coeffs[threadIdx.x + 577 + FPTRU_N])));
        }
    }
}

__device__ void d_poly_sample_and_double(nttpoly_n761 & nttb,const uint8_t buf[FPTRU_COIN_BYTES / 2],uint16_t is_double,uint16_t is_add1){
    unsigned int i, j;
    uint32_t t, d;
    int16_t a, b;

    j = threadIdx.x % 8;

#pragma unroll
    for(i = threadIdx.x / 8; i < 72; i+=24){
        t = load32_littleendian(buf + 4 * i);
        d = t & 0x55555555;
        d += (t >> 1) & 0x55555555;

        a = (d >> (4 * j + 0)) & 0x3;
        b = (d >> (4 * j + 2)) & 0x3;
        
        a = a - b +  is_double * (a - b) + is_add1 * (!(8 * i + j));
        nttb.coeffs[8 * i + j] = a;
    }

    if(threadIdx.x < 185){//761 = 192 * 3 + 185
        if(threadIdx.x == 184){
            i = 95;
            t = buf[4 * i];
            a = 0;
            b = 0;
            //直接基于t生成b
            for (j = 0; j < 2; j++){
                a += (t >> j) & 0x1;
            }
            for (j = 2; j < 4; j++){
                b += (t >> j) & 0x1;
            }
            a = (int16_t)(a - b) + is_double * (int16_t)(a - b) + is_add1 * (!(8 * i + 0));
            nttb.coeffs[8 * i + 0] = a;
        }
        else{
            t = load32_littleendian(buf + 4 * i);
            d = t & 0x55555555;
            d += (t >> 1) & 0x55555555;

            a = (d >> (4 * j + 0)) & 0x3;
            b = (d >> (4 * j + 2)) & 0x3;
            a = (int16_t)(a - b +  is_double * (a - b) + is_add1 * (!(8 * i + j)));//int16->int32
            nttb.coeffs[8 * i + j] = a;
        }
    }
}

__global__ void poly_poly_sample_and_double_mul_761_q1(unsigned char * array_coins,uint16_t is_double,uint16_t is_add1,int interval,poly * array_c,poly * array_a){
    __shared__ nttpoly_n761 ntta;
    __shared__ nttpoly_n761 nttb;


#pragma unroll
    for(int i=0;i<8;i++){ //init is important
        ntta.coeffs[i * 192 + threadIdx.x] = 0;
        nttb.coeffs[i * 192 + threadIdx.x] = 0;
    }
    __syncthreads();
    d_poly_sample_and_double(nttb,&array_coins[blockIdx.x * interval],is_double,is_add1);
    __syncthreads();
    d_poly_mul_q1_761(ntta, nttb, array_a, array_c);


}