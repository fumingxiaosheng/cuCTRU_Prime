#include "hip/hip_runtime.h"
#include "kernel.h"
#include "params.h"
#include "poly.h"
#include "pack.h"

#include <iostream>

#define RM_LEN 1277
#if (USING_PK_ENCODE == 1)
/* Based on the reference implementation of NTRU Prime (NIST 3rd round submission)
 * by Daniel J. Bernstein, Chitchanok Chuengsatiansup, Tanja Lange, Christine van Vredendaal.
 * It can be used for q \in {4091, 4621, 4591, 7879}.
 * */

void Encode(unsigned char *out, const uint16_t *R, const uint16_t *M, const long long len)
{
    if (len == 1)
    {
        uint16_t r = R[0];
        uint16_t m = M[0];
        while (m > 1)
        {
            *out++ = r;
            r >>= 8;
            m = (m + 255) >> 8;
        }
    }

    if (len > 1)
    {
        uint16_t R2[(len + 1) / 2];
        uint16_t M2[(len + 1) / 2];
        long long i;

        for (i = 0; i < len - 1; i += 2)
        {
            uint32_t m0 = M[i];
            uint32_t r = R[i] + R[i + 1] * m0;
            uint32_t m = M[i + 1] * m0;

            while (m >= 16384)
            {
                *out++ = r; //拼接操作限制了多个线程的并发
                r >>= 8;
                m = (m + 255) >> 8;
            }
            R2[i / 2] = r;
            M2[i / 2] = m;
        }

        if (i < len)
        {
            R2[i / 2] = R[i];
            M2[i / 2] = M[i];
        }

        Encode(out, R2, M2, (len + 1) / 2);
    }
}

void pack_pk(unsigned char *r, poly *a)
{
    uint16_t R[FPTRU_N], M[FPTRU_N];

    for (int i = 0; i < FPTRU_N; ++i)
        R[i] = (uint16_t)a->coeffs[i];

    for (int i = 0; i < FPTRU_N; ++i)
        M[i] = FPTRU_Q;

    Encode(r, R, M, FPTRU_N);
}

__device__ void Encode_2_gpu(unsigned char *out,uint16_t *R, uint16_t *M, long long len){
    uint16_t R2[RM_LEN];
    uint16_t M2[RM_LEN];
    long long i;//注意是否需要放到循环内部
    for(;len!=1;len=(len + 1) / 2){
        for(i=0;i<len-1;i+=2){
            uint32_t m0=M[i];
            uint32_t r=R[i]+R[i+1]*m0;
            uint32_t m=M[i+1]*m0;
            while (m >= 16384)
            {
                *out++ = r;//TODO:这里应该如何实现为线程的同步:1.最外层接->不行
                r >>= 8;
                m = (m + 255) >> 8;
            }
            R2[i / 2] = r;
            M2[i / 2] = m;
        }

        if (i < len){
            R2[i / 2] = R[i];//直接把奇数的值赋值过去
            M2[i / 2] = M[i];
        }

        //递归调用的过程，本质上是转化了R和R2的位置
        for(int j=0;j<RM_LEN;j++){
            R[j]=R2[j];
            M[j]=M2[j];
        }
    }

    if (len == 1)
    {
        uint16_t r = R[0];
        uint16_t m = M[0];
        while (m > 1)
        {
            *out++ = r;
            r >>= 8;
            m = (m + 255) >> 8;
        }
    }

}
__device__ void pack_pk_gpu(unsigned char *r, const poly *a)
{   
    //printf("[in pack_pk] start\n");
    //uint16_t R[FPTRU_N], M[FPTRU_N];
    uint16_t R[RM_LEN], M[RM_LEN];

    for (int i = 0; i < FPTRU_N; ++i)
        R[i] = (uint16_t)a->coeffs[i];

    for (int i = 0; i < FPTRU_N; ++i)
        M[i] = FPTRU_Q;
    
    
    Encode_2_gpu(r, R, M, FPTRU_N);
}

/*2024-4-24:
输入:array_r:待处理的公钥数组
    array_a:待转化的多项式数组

处理流程:数据分组，分给每一个线程进行处理

线程组织:<<<BATCH_SIZE,1>>>
*/
__global__ void pack_pk_batch(unsigned char *array_r,poly * array_a){
    //printf("[%d %d] start\n",blockIdx.x,threadIdx.x);
    pack_pk_gpu(&array_r[FPTRU_KEM_PUBLICKEYBYTES * (blockIdx.x)], &(array_a[blockIdx.x]));
    //printf("[%d %d] end\n",blockIdx.x,threadIdx.x);
}
#endif



#if (USING_PK_ENCODE == 1)
void Decode(uint16_t *out, const unsigned char *S, const uint16_t *M, const long long len)
{
    if (len == 1)
    {
        if (M[0] == 1)
            *out = 0;
        else if (M[0] <= 256)
            *out = uint32_mod_uint14_cpu(S[0], M[0]);
        else
            *out = uint32_mod_uint14_cpu(S[0] + (((uint16_t)S[1]) << 8), M[0]);
    }

    if (len > 1)
    {
        uint16_t R2[(len + 1) / 2];
        uint16_t M2[(len + 1) / 2];
        uint16_t bottomr[len / 2];
        uint32_t bottomt[len / 2];


        // uint16_t R2[1277];//uint16_t R2[(len + 1) / 2];
        // uint16_t M2[1277];//uint16_t M2[(len + 1) / 2];
        // uint16_t bottomr[1277];//uint16_t bottomr[len / 2];
        // uint32_t bottomt[1277];//uint32_t bottomt[len / 2];
        long long i;
        for (i = 0; i < len - 1; i += 2)
        {
            uint32_t m = M[i] * (uint32_t)M[i + 1];
            if (m > 256 * 16383)
            {
                bottomt[i / 2] = 256 * 256;
                bottomr[i / 2] = S[0] + 256 * S[1];
                S += 2;
                M2[i / 2] = (((m + 255) >> 8) + 255) >> 8;
            }
            else if (m >= 16384)
            {
                bottomt[i / 2] = 256;
                bottomr[i / 2] = S[0];
                S += 1;
                M2[i / 2] = (m + 255) >> 8;
            }
            else
            {
                bottomt[i / 2] = 1;
                bottomr[i / 2] = 0;
                M2[i / 2] = m;
            }
        }
        if (i < len)
            M2[i / 2] = M[i];

        Decode(R2, S, M2, (len + 1) / 2);

        for (i = 0; i < len - 1; i += 2)
        {
            uint32_t r = bottomr[i / 2];//和当时自己的临时状态有关系
            uint32_t r1;
            uint16_t r0;
            r += bottomt[i / 2] * R2[i / 2]; //当前的输入和上一级的输出有关系
            uint32_divmod_uint14_cpu(&r1, &r0, r, M[i]);
            r1 = uint32_mod_uint14_cpu(r1, M[i + 1]);
            *out++ = r0;
            *out++ = r1;
        }
        if (i < len)
            *out++ = R2[i / 2];
    }
}

void unpack_pk(poly *r, const unsigned char *a)
{
    uint16_t R[FPTRU_N], M[FPTRU_N];

    for (int i = 0; i < FPTRU_N; ++i)
        M[i] = FPTRU_Q;

    Decode(R, a, M, FPTRU_N);

    for (int i = 0; i < FPTRU_N; ++i)
        r->coeffs[i] = ((int16_t)R[i]);
}

/*__global__ void unpack_pk_batch(){

}*/
#endif



void pack_pk_simple(unsigned char *r, const poly *a)
{
    unsigned int i = 0 ;
    for (i = 0; i < FPTRU_N/8; i ++) //95
    {
        r[i *13] = a->coeffs[i*8];
        r[i *13+ 1] = (a->coeffs[i*8] >> 8)| ((int16_t)a->coeffs[i*8+1] << 5);
        r[i *13+ 2] = a->coeffs[i*8+1] >> 3;
        r[i *13+ 3] = (a->coeffs[i*8+1] >> 11 )| ((int16_t)a->coeffs[i*8+2] << 2);
        r[i *13+ 4] = (a->coeffs[i*8+2] >> 6  )| ((int16_t)a->coeffs[i*8+3] << 7);
        r[i *13+ 5] = a->coeffs[i*8+3] >> 1;
        r[i *13+ 6] = (a->coeffs[i*8+3] >> 9  )| ((int16_t)a->coeffs[i*8+4] << 4);
        r[i *13+ 7] = a->coeffs[i*8+4] >> 4;
        r[i *13+ 8] = (a->coeffs[i*8+4] >> 12 )| ((int16_t)a->coeffs[i*8+5] << 1);
        r[i *13+ 9] = (a->coeffs[i*8+5] >> 7  )| ((int16_t)a->coeffs[i*8+6] << 6);
        r[i *13+ 10] = a->coeffs[i*8+6] >> 2;
        r[i *13+ 11] = (a->coeffs[i*8+6] >> 10) | ((int16_t)a->coeffs[i*8+7] << 3);
        r[i *13+ 12] = a->coeffs[i*8+7] >> 5;
    }
    r[i*13] = a->coeffs[i*8];
    r[i*13+ 1] = (a->coeffs[i*8] >> 8) & 0x1F;
}

__device__ void pack_pk_simple_gpu(unsigned char *r, const poly *a){
    int i = threadIdx.x;
    if(i == FPTRU_N/8){
        r[i*13] = a->coeffs[i*8];
        r[i*13+ 1] = (a->coeffs[i*8] >> 8) & 0x1F;
    }
    else{
        r[i *13] = a->coeffs[i*8];
        r[i *13+ 1] = (a->coeffs[i*8] >> 8)| ((int16_t)a->coeffs[i*8+1] << 5);
        r[i *13+ 2] = a->coeffs[i*8+1] >> 3;
        r[i *13+ 3] = (a->coeffs[i*8+1] >> 11 )| ((int16_t)a->coeffs[i*8+2] << 2);
        r[i *13+ 4] = (a->coeffs[i*8+2] >> 6  )| ((int16_t)a->coeffs[i*8+3] << 7);
        r[i *13+ 5] = a->coeffs[i*8+3] >> 1;
        r[i *13+ 6] = (a->coeffs[i*8+3] >> 9  )| ((int16_t)a->coeffs[i*8+4] << 4);
        r[i *13+ 7] = a->coeffs[i*8+4] >> 4;
        r[i *13+ 8] = (a->coeffs[i*8+4] >> 12 )| ((int16_t)a->coeffs[i*8+5] << 1);
        r[i *13+ 9] = (a->coeffs[i*8+5] >> 7  )| ((int16_t)a->coeffs[i*8+6] << 6);
        r[i *13+ 10] = a->coeffs[i*8+6] >> 2;
        r[i *13+ 11] = (a->coeffs[i*8+6] >> 10) | ((int16_t)a->coeffs[i*8+7] << 3);
        r[i *13+ 12] = a->coeffs[i*8+7] >> 5;
    }

}
__global__ void pack_pk_simple_batch(unsigned char *r, const poly *a){
    pack_pk_simple_gpu(&r[FPTRU_KEM_PUBLICKEYBYTES * blockIdx.x], &a[blockIdx.x]);
}



void unpack_pk_simple(poly *r, const unsigned char *a) {

    unsigned int i = 0;
    for (i = 0; i < FPTRU_N/8; i++) {
        r->coeffs[i*8] = a[i*13] | (uint16_t)a[i*13 + 1] << 8;
        r->coeffs[i*8] &= 0x1FFF;
        r->coeffs[i*8+1] = (a[i*13 + 1] >> 5) | (uint16_t)(a[i*13 + 2]  << 3) | (uint16_t)(a[i*13 + 3]  << 11) ;
        r->coeffs[i*8+1] &= 0x1FFF;
        r->coeffs[i*8+2] = (a[i*13 + 3] >> 2) | (uint16_t)(a[i*13 + 4]  << 6);
        r->coeffs[i*8+2] &= 0x1FFF;
        r->coeffs[i*8+3] = (a[i*13 + 4] >> 7) | (uint16_t)(a[i*13 + 5]  << 1)| (uint16_t)(a[i*13 + 6]   << 9);
        r->coeffs[i*8+3] &= 0x1FFF;
        r->coeffs[i*8+4] = (a[i*13 + 6] >> 4) | (uint16_t)(a[i*13 + 7]  << 4) | (uint16_t)(a[i*13 + 8]  << 12);
        r->coeffs[i*8+4] &= 0x1FFF;
        r->coeffs[i*8+5] = (a[i*13 + 8] >> 1) | (uint16_t)(a[i*13 + 9]  << 7) ;
        r->coeffs[i*8+5] &= 0x1FFF;
        r->coeffs[i*8+6] = (a[i*13 + 9] >> 6) | (uint16_t)(a[i*13 + 10] << 2) | (uint16_t)(a[i*13 + 11] << 10);
        r->coeffs[i*8+6] &= 0x1FFF;
        r->coeffs[i*8+7] = (a[i*13 + 11] >> 3) | (uint16_t)(a[i*13 + 12] << 5) ;
        r->coeffs[i*8+7] &= 0x1FFF;
    }
    r->coeffs[i*8] = a[i*13] | (uint16_t)a[i*13 + 1] << 8;
    r->coeffs[i*8] &= 0x1FFF;
}

__device__ void unpack_pk_simple_gpu(poly *r, const unsigned char *a){
    int i = threadIdx.x;
    if(i == FPTRU_N/8){
        r->coeffs[i*8] = a[i*13] | (uint16_t)a[i*13 + 1] << 8;
        r->coeffs[i*8] &= 0x1FFF;
    }
    else{
        r->coeffs[i*8] = a[i*13] | (uint16_t)a[i*13 + 1] << 8;
        r->coeffs[i*8] &= 0x1FFF;
        r->coeffs[i*8+1] = (a[i*13 + 1] >> 5) | (uint16_t)(a[i*13 + 2]  << 3) | (uint16_t)(a[i*13 + 3]  << 11) ;
        r->coeffs[i*8+1] &= 0x1FFF;
        r->coeffs[i*8+2] = (a[i*13 + 3] >> 2) | (uint16_t)(a[i*13 + 4]  << 6);
        r->coeffs[i*8+2] &= 0x1FFF;
        r->coeffs[i*8+3] = (a[i*13 + 4] >> 7) | (uint16_t)(a[i*13 + 5]  << 1)| (uint16_t)(a[i*13 + 6]   << 9);
        r->coeffs[i*8+3] &= 0x1FFF;
        r->coeffs[i*8+4] = (a[i*13 + 6] >> 4) | (uint16_t)(a[i*13 + 7]  << 4) | (uint16_t)(a[i*13 + 8]  << 12);
        r->coeffs[i*8+4] &= 0x1FFF;
        r->coeffs[i*8+5] = (a[i*13 + 8] >> 1) | (uint16_t)(a[i*13 + 9]  << 7) ;
        r->coeffs[i*8+5] &= 0x1FFF;
        r->coeffs[i*8+6] = (a[i*13 + 9] >> 6) | (uint16_t)(a[i*13 + 10] << 2) | (uint16_t)(a[i*13 + 11] << 10);
        r->coeffs[i*8+6] &= 0x1FFF;
        r->coeffs[i*8+7] = (a[i*13 + 11] >> 3) | (uint16_t)(a[i*13 + 12] << 5) ;
        r->coeffs[i*8+7] &= 0x1FFF;
    }
}
__global__ void unpack_pk_simple_batch(poly *r, const unsigned char *a, int interval){
    unpack_pk_simple_gpu(&r[blockIdx.x], &a[blockIdx.x * interval]);
}


__device__ void pack_sk_gpu(unsigned char *r, const poly *a)
{
    unsigned int i;
    uint8_t t[8];
#if ((FPTRU_N == 653) || (FPTRU_N == 1277))
    for (i = 0; i < FPTRU_N / 8; i++)
    {
        t[0] = FPTRU_BOUND - a->coeffs[8 * i + 0];
        t[1] = FPTRU_BOUND - a->coeffs[8 * i + 1];
        t[2] = FPTRU_BOUND - a->coeffs[8 * i + 2];
        t[3] = FPTRU_BOUND - a->coeffs[8 * i + 3];
        t[4] = FPTRU_BOUND - a->coeffs[8 * i + 4];
        t[5] = FPTRU_BOUND - a->coeffs[8 * i + 5];
        t[6] = FPTRU_BOUND - a->coeffs[8 * i + 6];
        t[7] = FPTRU_BOUND - a->coeffs[8 * i + 7];

        r[4 * i + 0] = (t[0] >> 0) | (t[1] << 4);
        r[4 * i + 1] = (t[2] >> 0) | (t[3] << 4);
        r[4 * i + 2] = (t[4] >> 0) | (t[5] << 4);
        r[4 * i + 3] = (t[6] << 0) | (t[7] << 4);
    }
    t[0] = FPTRU_BOUND - a->coeffs[8 * i + 0];
    t[1] = FPTRU_BOUND - a->coeffs[8 * i + 1];
    t[2] = FPTRU_BOUND - a->coeffs[8 * i + 2];
    t[3] = FPTRU_BOUND - a->coeffs[8 * i + 3];
    t[4] = FPTRU_BOUND - a->coeffs[8 * i + 4];
    r[4 * i + 0] = (t[0] >> 0) | (t[1] << 4);
    r[4 * i + 1] = (t[2] >> 0) | (t[3] << 4);
    r[4 * i + 2] = (t[4] >> 0) & 0xf;
#elif (FPTRU_N == 761)
    for (i = 0; i < FPTRU_N / 8; i++)
    {
        t[0] = FPTRU_BOUND - a->coeffs[8 * i + 0];
        t[1] = FPTRU_BOUND - a->coeffs[8 * i + 1];
        t[2] = FPTRU_BOUND - a->coeffs[8 * i + 2];
        t[3] = FPTRU_BOUND - a->coeffs[8 * i + 3];
        t[4] = FPTRU_BOUND - a->coeffs[8 * i + 4];
        t[5] = FPTRU_BOUND - a->coeffs[8 * i + 5];
        t[6] = FPTRU_BOUND - a->coeffs[8 * i + 6];
        t[7] = FPTRU_BOUND - a->coeffs[8 * i + 7];

        r[4 * i + 0] = (t[0] >> 0) | (t[1] << 4);
        r[4 * i + 1] = (t[2] >> 0) | (t[3] << 4);
        r[4 * i + 2] = (t[4] >> 0) | (t[5] << 4);
        r[4 * i + 3] = (t[6] << 0) | (t[7] << 4);
    }
    t[0] = FPTRU_BOUND - a->coeffs[8 * i + 0];
    r[4 * i + 0] = (t[0] >> 0) & 0xf;
#endif
}

void pack_sk(unsigned char *r,poly *a)
{
    unsigned int i;
    uint8_t t[8];
#if ((FPTRU_N == 653) || (FPTRU_N == 1277))
    for (i = 0; i < FPTRU_N / 8; i++)
    {
        t[0] = FPTRU_BOUND - a->coeffs[8 * i + 0];
        t[1] = FPTRU_BOUND - a->coeffs[8 * i + 1];
        t[2] = FPTRU_BOUND - a->coeffs[8 * i + 2];
        t[3] = FPTRU_BOUND - a->coeffs[8 * i + 3];
        t[4] = FPTRU_BOUND - a->coeffs[8 * i + 4];
        t[5] = FPTRU_BOUND - a->coeffs[8 * i + 5];
        t[6] = FPTRU_BOUND - a->coeffs[8 * i + 6];
        t[7] = FPTRU_BOUND - a->coeffs[8 * i + 7];

        r[4 * i + 0] = (t[0] >> 0) | (t[1] << 4);
        r[4 * i + 1] = (t[2] >> 0) | (t[3] << 4);
        r[4 * i + 2] = (t[4] >> 0) | (t[5] << 4);
        r[4 * i + 3] = (t[6] << 0) | (t[7] << 4);
    }
    t[0] = FPTRU_BOUND - a->coeffs[8 * i + 0];
    t[1] = FPTRU_BOUND - a->coeffs[8 * i + 1];
    t[2] = FPTRU_BOUND - a->coeffs[8 * i + 2];
    t[3] = FPTRU_BOUND - a->coeffs[8 * i + 3];
    t[4] = FPTRU_BOUND - a->coeffs[8 * i + 4];
    r[4 * i + 0] = (t[0] >> 0) | (t[1] << 4);
    r[4 * i + 1] = (t[2] >> 0) | (t[3] << 4);
    r[4 * i + 2] = (t[4] >> 0) & 0xf;
#elif (FPTRU_N == 761)
    for (i = 0; i < FPTRU_N / 8; i++)
    {
        t[0] = FPTRU_BOUND - a->coeffs[8 * i + 0];
        t[1] = FPTRU_BOUND - a->coeffs[8 * i + 1];
        t[2] = FPTRU_BOUND - a->coeffs[8 * i + 2];
        t[3] = FPTRU_BOUND - a->coeffs[8 * i + 3];
        t[4] = FPTRU_BOUND - a->coeffs[8 * i + 4];
        t[5] = FPTRU_BOUND - a->coeffs[8 * i + 5];
        t[6] = FPTRU_BOUND - a->coeffs[8 * i + 6];
        t[7] = FPTRU_BOUND - a->coeffs[8 * i + 7];

        r[4 * i + 0] = (t[0] >> 0) | (t[1] << 4);
        r[4 * i + 1] = (t[2] >> 0) | (t[3] << 4);
        r[4 * i + 2] = (t[4] >> 0) | (t[5] << 4);
        r[4 * i + 3] = (t[6] << 0) | (t[7] << 4);
    }
    t[0] = FPTRU_BOUND - a->coeffs[8 * i + 0];
    r[4 * i + 0] = (t[0] >> 0) & 0xf;
#endif
}


/*2024-4-29:
输入:array_r:输出数组，用于存储每一个私钥多项式对应的输出
    array_a:输入的多项式数组，存储每一个私钥
    
输出:存储到array_r中

处理流程:对数据进行分块处理,然后再调用pack_sk_v2

线程组织形式:<<<BATCH_SIZE,FPTRU_N/8 + 1>>>*/
__global__ void pack_sk_batch_v2(unsigned char *array_r, poly *array_a){

    pack_sk_v2(&array_r[FPTRU_PKE_SECRETKEYBYTES * (blockIdx.x)], &(array_a[blockIdx.x]));
}

/*2024-4-29:
输入:r:全局内存，用于存储输出的结果
    a:全局内存，代表了待pack的私钥多项式
    
输出:存储到r中

线程组织形式:<<<BATCH_SIZE,FPTRU_N/8 + 1>>>*/
__device__ void pack_sk_v2(unsigned char * r,const poly *a){
    uint8_t t[8];//每一个线程的寄存器内存

#if ((FPTRU_N == 653) || (FPTRU_N == 1277))
    if(threadIdx.x < FPTRU_N / 8){
        t[0] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 0];
        t[1] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 1];
        t[2] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 2];
        t[3] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 3];
        t[4] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 4];
        t[5] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 5];
        t[6] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 6];
        t[7] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 7];

        r[4 * threadIdx.x + 0] = (t[0] >> 0) | (t[1] << 4);
        r[4 * threadIdx.x + 1] = (t[2] >> 0) | (t[3] << 4);
        r[4 * threadIdx.x + 2] = (t[4] >> 0) | (t[5] << 4);
        r[4 * threadIdx.x + 3] = (t[6] << 0) | (t[7] << 4);
    }
    else{
        t[0] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 0];
        t[1] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 1];
        t[2] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 2];
        t[3] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 3];
        t[4] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 4];
        r[4 * threadIdx.x + 0] = (t[0] >> 0) | (t[1] << 4);
        r[4 * threadIdx.x + 1] = (t[2] >> 0) | (t[3] << 4);
        r[4 * threadIdx.x + 2] = (t[4] >> 0) & 0xf;
    }
#elif(FPTRU_N == 761)
    if(threadIdx.x < FPTRU_N / 8){
        t[0] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 0];
        t[1] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 1];
        t[2] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 2];
        t[3] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 3];
        t[4] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 4];
        t[5] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 5];
        t[6] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 6];
        t[7] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 7];

        r[4 * threadIdx.x + 0] = (t[0] >> 0) | (t[1] << 4);
        r[4 * threadIdx.x + 1] = (t[2] >> 0) | (t[3] << 4);
        r[4 * threadIdx.x + 2] = (t[4] >> 0) | (t[5] << 4);
        r[4 * threadIdx.x + 3] = (t[6] << 0) | (t[7] << 4);
    }
    else{
        t[0] = FPTRU_BOUND - a->coeffs[8 * threadIdx.x + 0];
        r[4 * threadIdx.x + 0] = (t[0] >> 0) & 0xf;
    }
#endif
}
/*2024-4-24:
输入:array_r:待压缩的公钥存储数组
    array_a:待压缩的多项式数组

处理流程:数据分组，分给每一个线程进行处理

线程组织:<<<BATCH_SIZE,1>>>
*/
__global__ void pack_sk_batch(unsigned char *array_r, poly *array_a){

    pack_sk_gpu(&array_r[FPTRU_PKE_SECRETKEYBYTES * (blockIdx.x)], &(array_a[blockIdx.x]));
}

void unpack_sk(poly *r, const unsigned char *a)
{
    unsigned int i;
#if ((FPTRU_N == 653) || (FPTRU_N == 1277))
    for (i = 0; i < FPTRU_N / 8; ++i)
    {
        r->coeffs[8 * i + 0] = (a[4 * i + 0] >> 0) & 15;
        r->coeffs[8 * i + 1] = (a[4 * i + 0] >> 4) & 15;
        r->coeffs[8 * i + 2] = (a[4 * i + 1] >> 0) & 15;
        r->coeffs[8 * i + 3] = (a[4 * i + 1] >> 4) & 15;
        r->coeffs[8 * i + 4] = (a[4 * i + 2] >> 0) & 15;
        r->coeffs[8 * i + 5] = (a[4 * i + 2] >> 4) & 15;
        r->coeffs[8 * i + 6] = (a[4 * i + 3] >> 0) & 15;
        r->coeffs[8 * i + 7] = (a[4 * i + 3] >> 4) & 15;

        r->coeffs[8 * i + 0] = FPTRU_BOUND - r->coeffs[8 * i + 0];
        r->coeffs[8 * i + 1] = FPTRU_BOUND - r->coeffs[8 * i + 1];
        r->coeffs[8 * i + 2] = FPTRU_BOUND - r->coeffs[8 * i + 2];
        r->coeffs[8 * i + 3] = FPTRU_BOUND - r->coeffs[8 * i + 3];
        r->coeffs[8 * i + 4] = FPTRU_BOUND - r->coeffs[8 * i + 4];
        r->coeffs[8 * i + 5] = FPTRU_BOUND - r->coeffs[8 * i + 5];
        r->coeffs[8 * i + 6] = FPTRU_BOUND - r->coeffs[8 * i + 6];
        r->coeffs[8 * i + 7] = FPTRU_BOUND - r->coeffs[8 * i + 7];
    }
    r->coeffs[8 * i + 0] = (a[4 * i + 0] >> 0) & 15;
    r->coeffs[8 * i + 1] = (a[4 * i + 0] >> 4) & 15;
    r->coeffs[8 * i + 2] = (a[4 * i + 1] >> 0) & 15;
    r->coeffs[8 * i + 3] = (a[4 * i + 1] >> 4) & 15;
    r->coeffs[8 * i + 4] = (a[4 * i + 2] >> 0) & 15;
    r->coeffs[8 * i + 0] = FPTRU_BOUND - r->coeffs[8 * i + 0];
    r->coeffs[8 * i + 1] = FPTRU_BOUND - r->coeffs[8 * i + 1];
    r->coeffs[8 * i + 2] = FPTRU_BOUND - r->coeffs[8 * i + 2];
    r->coeffs[8 * i + 3] = FPTRU_BOUND - r->coeffs[8 * i + 3];
    r->coeffs[8 * i + 4] = FPTRU_BOUND - r->coeffs[8 * i + 4];
#elif (FPTRU_N == 761)
    for (i = 0; i < FPTRU_N / 8; ++i)
    {
        r->coeffs[8 * i + 0] = (a[4 * i + 0] >> 0) & 15;
        r->coeffs[8 * i + 1] = (a[4 * i + 0] >> 4) & 15;
        r->coeffs[8 * i + 2] = (a[4 * i + 1] >> 0) & 15;
        r->coeffs[8 * i + 3] = (a[4 * i + 1] >> 4) & 15;
        r->coeffs[8 * i + 4] = (a[4 * i + 2] >> 0) & 15;
        r->coeffs[8 * i + 5] = (a[4 * i + 2] >> 4) & 15;
        r->coeffs[8 * i + 6] = (a[4 * i + 3] >> 0) & 15;
        r->coeffs[8 * i + 7] = (a[4 * i + 3] >> 4) & 15;

        r->coeffs[8 * i + 0] = FPTRU_BOUND - r->coeffs[8 * i + 0];
        r->coeffs[8 * i + 1] = FPTRU_BOUND - r->coeffs[8 * i + 1];
        r->coeffs[8 * i + 2] = FPTRU_BOUND - r->coeffs[8 * i + 2];
        r->coeffs[8 * i + 3] = FPTRU_BOUND - r->coeffs[8 * i + 3];
        r->coeffs[8 * i + 4] = FPTRU_BOUND - r->coeffs[8 * i + 4];
        r->coeffs[8 * i + 5] = FPTRU_BOUND - r->coeffs[8 * i + 5];
        r->coeffs[8 * i + 6] = FPTRU_BOUND - r->coeffs[8 * i + 6];
        r->coeffs[8 * i + 7] = FPTRU_BOUND - r->coeffs[8 * i + 7];
    }
    r->coeffs[8 * i + 0] = (a[4 * i + 0] >> 0) & 15;
    r->coeffs[8 * i + 0] = FPTRU_BOUND - r->coeffs[8 * i + 0];
#endif
}

void pack_ct(unsigned char *r, const poly *a)
{
    unsigned int i;
#if (FPTRU_Q2 == 1024)
    for (i = 0; i < FPTRU_N / 4; ++i)
    {
        r[5 * i + 0] = (a->coeffs[4 * i + 0] >> 0);
        r[5 * i + 1] = (a->coeffs[4 * i + 0] >> 8) | ((int16_t)a->coeffs[4 * i + 1] << 2);
        r[5 * i + 2] = (a->coeffs[4 * i + 1] >> 6) | ((int16_t)a->coeffs[4 * i + 2] << 4);
        r[5 * i + 3] = (a->coeffs[4 * i + 2] >> 4) | ((int16_t)a->coeffs[4 * i + 3] << 6);
        r[5 * i + 4] = (a->coeffs[4 * i + 3] >> 2);
    }
    r[5 * i + 0] = (a->coeffs[4 * i + 0] >> 0);
    r[5 * i + 1] = (a->coeffs[4 * i + 0] >> 8) & 0x3;
#elif (FPTRU_Q2 == 2048)
    for (i = 0; i < FPTRU_N / 8; ++i)
    {
        r[11 * i + 0] = a->coeffs[8 * i + 0] >> 0;
        r[11 * i + 1] = a->coeffs[8 * i + 0] >> 8 | ((int16_t)a->coeffs[8 * i + 1] << 3);
        r[11 * i + 2] = a->coeffs[8 * i + 1] >> 5 | ((int16_t)a->coeffs[8 * i + 2] << 6);
        r[11 * i + 3] = a->coeffs[8 * i + 2] >> 2;
        r[11 * i + 4] = a->coeffs[8 * i + 2] >> 10 | ((int16_t)a->coeffs[8 * i + 3] << 1);
        r[11 * i + 5] = a->coeffs[8 * i + 3] >> 7 | ((int16_t)a->coeffs[8 * i + 4] << 4);
        r[11 * i + 6] = a->coeffs[8 * i + 4] >> 4 | ((int16_t)a->coeffs[8 * i + 5] << 7);
        r[11 * i + 7] = a->coeffs[8 * i + 5] >> 1;
        r[11 * i + 8] = a->coeffs[8 * i + 5] >> 9 | ((int16_t)a->coeffs[8 * i + 6] << 2);
        r[11 * i + 9] = a->coeffs[8 * i + 6] >> 6 | ((int16_t)a->coeffs[8 * i + 7] << 5);
        r[11 * i + 10] = a->coeffs[8 * i + 7] >> 3;
    }
    r[11 * i + 0] = a->coeffs[8 * i + 0] >> 0;
    r[11 * i + 1] = a->coeffs[8 * i + 0] >> 8 | ((int16_t)a->coeffs[8 * i + 1] << 3);
    r[11 * i + 2] = a->coeffs[8 * i + 1] >> 5 | ((int16_t)a->coeffs[8 * i + 2] << 6);
    r[11 * i + 3] = a->coeffs[8 * i + 2] >> 2;
    r[11 * i + 4] = a->coeffs[8 * i + 2] >> 10 | ((int16_t)a->coeffs[8 * i + 3] << 1);
    r[11 * i + 5] = a->coeffs[8 * i + 3] >> 7 | ((int16_t)a->coeffs[8 * i + 4] << 4);
    r[11 * i + 6] = (a->coeffs[8 * i + 4] >> 4) & 0x7f;
#elif (FPTRU_Q2 == 4096)
    for (i = 0; i < FPTRU_N / 2; i++)
    {
        r[3 * i + 0] = (a->coeffs[2 * i] >> 0);
        r[3 * i + 1] = (a->coeffs[2 * i] >> 8) | (a->coeffs[2 * i + 1] << 4);
        r[3 * i + 2] = (a->coeffs[2 * i + 1] >> 4);
    }
    r[3 * i + 0] = (a->coeffs[2 * i] >> 0);
    r[3 * i + 1] = (a->coeffs[2 * i] >> 8) & 0xf;
#endif
}

void unpack_ct(poly *r, const unsigned char *a)
{
    unsigned int i;
#if (FPTRU_Q2 == 1024)
    for (i = 0; i < FPTRU_N / 4; ++i)
    {
        r->coeffs[4 * i + 0] = ((a[5 * i + 0] >> 0) | ((uint16_t)a[5 * i + 1] << 8)) & 0x3FF;
        r->coeffs[4 * i + 1] = ((a[5 * i + 1] >> 2) | ((uint16_t)a[5 * i + 2] << 6)) & 0x3FF;
        r->coeffs[4 * i + 2] = ((a[5 * i + 2] >> 4) | ((uint16_t)a[5 * i + 3] << 4)) & 0x3FF;
        r->coeffs[4 * i + 3] = ((a[5 * i + 3] >> 6) | ((uint16_t)a[5 * i + 4] << 2)) & 0x3FF;
    }
    r->coeffs[4 * i + 0] = ((a[5 * i + 0] >> 0) | ((uint16_t)a[5 * i + 1] << 8)) & 0x3FF;
#elif (FPTRU_Q2 == 2048)
    for (i = 0; i < FPTRU_N / 8; ++i)
    {
        r->coeffs[8 * i + 0] = ((a[11 * i + 0] >> 0) | ((uint16_t)a[11 * i + 1] << 8)) & 0x7FF;
        r->coeffs[8 * i + 1] = ((a[11 * i + 1] >> 3) | ((uint16_t)a[11 * i + 2] << 5)) & 0x7FF;
        r->coeffs[8 * i + 2] = ((a[11 * i + 2] >> 6) | ((uint16_t)a[11 * i + 3] << 2) | ((uint16_t)a[11 * i + 4] << 10)) & 0x7FF;
        r->coeffs[8 * i + 3] = ((a[11 * i + 4] >> 1) | ((uint16_t)a[11 * i + 5] << 7)) & 0x7FF;
        r->coeffs[8 * i + 4] = ((a[11 * i + 5] >> 4) | ((uint16_t)a[11 * i + 6] << 4)) & 0x7FF;
        r->coeffs[8 * i + 5] = ((a[11 * i + 6] >> 7) | ((uint16_t)a[11 * i + 7] << 1) | ((uint16_t)a[11 * i + 8] << 9)) & 0x7FF;
        r->coeffs[8 * i + 6] = ((a[11 * i + 8] >> 2) | ((uint16_t)a[11 * i + 9] << 6)) & 0x7FF;
        r->coeffs[8 * i + 7] = ((a[11 * i + 9] >> 5) | ((uint16_t)a[11 * i + 10] << 3)) & 0x7FF;
    }
    r->coeffs[8 * i + 0] = ((a[11 * i + 0] >> 0) | ((uint16_t)a[11 * i + 1] << 8)) & 0x7FF;
    r->coeffs[8 * i + 1] = ((a[11 * i + 1] >> 3) | ((uint16_t)a[11 * i + 2] << 5)) & 0x7FF;
    r->coeffs[8 * i + 2] = ((a[11 * i + 2] >> 6) | ((uint16_t)a[11 * i + 3] << 2) | ((uint16_t)a[11 * i + 4] << 10)) & 0x7FF;
    r->coeffs[8 * i + 3] = ((a[11 * i + 4] >> 1) | ((uint16_t)a[11 * i + 5] << 7)) & 0x7FF;
    r->coeffs[8 * i + 4] = ((a[11 * i + 5] >> 4) | ((uint16_t)a[11 * i + 6] << 4)) & 0x7FF;
#elif (FPTRU_Q2 == 4096)
    for (i = 0; i < FPTRU_N / 2; i++)
    {
        r->coeffs[2 * i] = ((a[3 * i + 0] >> 0) | ((uint16_t)a[3 * i + 1] << 8)) & 0xFFF;
        r->coeffs[2 * i + 1] = ((a[3 * i + 1] >> 4) | ((uint16_t)a[3 * i + 2] << 4)) & 0xFFF;
    }
    r->coeffs[2 * i] = ((a[3 * i + 0] >> 0) | (((uint16_t)a[3 * i + 1] & 0xF) << 8)) & 0xFFF;
#endif
}

/*公式的验证结果如下
r->coeffs[0] = ((a[11 * idx_h + 0] >> 0) | ((uint16_t)a[11 * idx_h + 1] << 8) | (((uint16_t)a[11 * idx_h + 4] << 10) & 0 )) & 0x7FF
r->coeffs[1] = ((a[11 * idx_h + 1] >> 3) | ((uint16_t)a[11 * idx_h + 2] << 5) | (((uint16_t)a[11 * idx_h + 4] << 10) & 0 )) & 0x7FF
r->coeffs[2] = ((a[11 * idx_h + 2] >> 6) | ((uint16_t)a[11 * idx_h + 3] << 2) | (((uint16_t)a[11 * idx_h + 4] << 10) & 65535 )) & 0x7FF
r->coeffs[3] = ((a[11 * idx_h + 4] >> 1) | ((uint16_t)a[11 * idx_h + 5] << 7) | (((uint16_t)a[11 * idx_h + 8] << 9) & 0 )) & 0x7FF
r->coeffs[4] = ((a[11 * idx_h + 5] >> 4) | ((uint16_t)a[11 * idx_h + 6] << 4) | (((uint16_t)a[11 * idx_h + 8] << 9) & 0 )) & 0x7FF
r->coeffs[5] = ((a[11 * idx_h + 6] >> 7) | ((uint16_t)a[11 * idx_h + 7] << 1) | (((uint16_t)a[11 * idx_h + 8] << 9) & 65535 )) & 0x7FF
r->coeffs[6] = ((a[11 * idx_h + 8] >> 2) | ((uint16_t)a[11 * idx_h + 9] << 6) | (((uint16_t)a[11 * idx_h + 12] << 8) & 0 )) & 0x7FF
r->coeffs[7] = ((a[11 * idx_h + 9] >> 5) | ((uint16_t)a[11 * idx_h + 10] << 3) | (((uint16_t)a[11 * idx_h + 12] << 8) & 0 )) & 0x7FF
存在的问题:对于全局内存的访问占据了太多的时间
*/
__device__ void unpack_ct_cuda(poly * r,const unsigned char *a){
    uint16_t idx_l = threadIdx.x & 7;
    uint16_t idx_h = threadIdx.x >> 3;

    uint8_t l_3_l = idx_l % 3;
    uint8_t l_3_h = idx_l / 3;

    //unsigned char atmp[3];

    /*放到寄存器里没效果
    atmp[0]= a[11 * idx_h + (4 * l_3_h + l_3_l)];
    atmp[1]= a[11 * idx_h + (4 * l_3_h + l_3_l + 1)];
    atmp[2]= a[11 * idx_h + ((l_3_h +1) *4)];
    r->coeffs[threadIdx.x] = ((atmp[0] >> (l_3_h + l_3_l*3)) | ((uint16_t)atmp[1] << (8 - l_3_h - l_3_l*3)) | (((uint16_t)atmp[2] << (10 - l_3_h)) & ((!((idx_l + 1) % 3)) * 0xffff) )) & 0x7FF;*/

    r->coeffs[threadIdx.x]=((a[11 * idx_h + (4 * l_3_h + l_3_l)] >> (l_3_h + l_3_l*3)) | ((uint16_t)a[11 * idx_h + (4 * l_3_h + l_3_l + 1)] << (8 - l_3_h - l_3_l*3)) | (((uint16_t)a[11 * idx_h + ((l_3_h +1) *4)] << (10 - l_3_h)) & ((!((idx_l + 1) % 3)) * 0xffff) )) & 0x7FF;

}
__global__ void unpack_ct_batch(poly * r,const unsigned char *a){
    unpack_ct_cuda(&r[blockIdx.x],&a[blockIdx.x * FPTRU_KEM_CIPHERTEXTBYTES]);
}


/*2024-7-8*/
__device__ void pack_ct_1(unsigned char *r, const poly *a){
    int tid = threadIdx.x;
#if (FPTRU_Q2 == 1024)
    if(tid == FPTRU_N / 4){
        r[5 * tid + 0] = (a->coeffs[4 * tid + 0] >> 0);
        r[5 * tid + 1] = (a->coeffs[4 * tid + 0] >> 8) & 0x3;
    }
    else{
        r[5 * tid + 0] = (a->coeffs[4 * tid + 0] >> 0);
        r[5 * tid + 1] = (a->coeffs[4 * tid + 0] >> 8) | ((int16_t)a->coeffs[4 * tid + 1] << 2);
        r[5 * tid + 2] = (a->coeffs[4 * tid + 1] >> 6) | ((int16_t)a->coeffs[4 * tid + 2] << 4);
        r[5 * tid + 3] = (a->coeffs[4 * tid + 2] >> 4) | ((int16_t)a->coeffs[4 * tid + 3] << 6);
        r[5 * tid + 4] = (a->coeffs[4 * tid + 3] >> 2);
    }
#elif (FPTRU_Q2 == 2048)
    if(tid == FPTRU_N/8){
        r[11 * tid + 0] = a->coeffs[8 * tid + 0] >> 0;
        r[11 * tid + 1] = a->coeffs[8 * tid + 0] >> 8 | ((int16_t)a->coeffs[8 * tid + 1] << 3);
        r[11 * tid + 2] = a->coeffs[8 * tid + 1] >> 5 | ((int16_t)a->coeffs[8 * tid + 2] << 6);
        r[11 * tid + 3] = a->coeffs[8 * tid + 2] >> 2;
        r[11 * tid + 4] = a->coeffs[8 * tid + 2] >> 10 | ((int16_t)a->coeffs[8 * tid + 3] << 1);
        r[11 * tid + 5] = a->coeffs[8 * tid + 3] >> 7 | ((int16_t)a->coeffs[8 * tid + 4] << 4);
        r[11 * tid + 6] = (a->coeffs[8 * tid + 4] >> 4) & 0x7f;
    }
    else{
        r[11 * tid + 0] = a->coeffs[8 * tid + 0] >> 0;
        r[11 * tid + 1] = a->coeffs[8 * tid + 0] >> 8 | ((int16_t)a->coeffs[8 * tid + 1] << 3);
        r[11 * tid + 2] = a->coeffs[8 * tid + 1] >> 5 | ((int16_t)a->coeffs[8 * tid + 2] << 6);
        r[11 * tid + 3] = a->coeffs[8 * tid + 2] >> 2;
        r[11 * tid + 4] = a->coeffs[8 * tid + 2] >> 10 | ((int16_t)a->coeffs[8 * tid + 3] << 1);
        r[11 * tid + 5] = a->coeffs[8 * tid + 3] >> 7 | ((int16_t)a->coeffs[8 * tid + 4] << 4);
        r[11 * tid + 6] = a->coeffs[8 * tid + 4] >> 4 | ((int16_t)a->coeffs[8 * tid + 5] << 7);
        r[11 * tid + 7] = a->coeffs[8 * tid + 5] >> 1;
        r[11 * tid + 8] = a->coeffs[8 * tid + 5] >> 9 | ((int16_t)a->coeffs[8 * tid + 6] << 2);
        r[11 * tid + 9] = a->coeffs[8 * tid + 6] >> 6 | ((int16_t)a->coeffs[8 * tid + 7] << 5);
        r[11 * tid + 10] = a->coeffs[8 * tid + 7] >> 3;
    }

#elif (FPTRU_Q2 == 4096)
    if(tid == FPTRU_N / 2){
        r[3 * tid + 0] = (a->coeffs[2 * tid] >> 0);
        r[3 * tid + 1] = (a->coeffs[2 * tid] >> 8) & 0xf;
    }
    else{
        r[3 * tid + 0] = (a->coeffs[2 * tid] >> 0);
        r[3 * tid + 1] = (a->coeffs[2 * tid] >> 8) | (a->coeffs[2 * tid + 1] << 4);
        r[3 * tid + 2] = (a->coeffs[2 * tid + 1] >> 4);
    }

#endif
}
/*2024-7-8*/
__global__ void pack_ct_batch(unsigned char *r, const poly *a){
    //调试一下

    /*if(threadIdx.x==0 && blockIdx.x ==0){
       printf("in thread look sigma 0\n");
        for(int i=0;i<FPTRU_N;i++){
            printf("%d,",a[blockIdx.x].coeffs[i]);
        }
        printf("\n");


        printf("in thread look sigma 1\n");
        for(int i=0;i<FPTRU_N;i++){
            printf("%d,",a[blockIdx.x + 1].coeffs[i]);
        }
        printf("\n");

        printf("look sigma\n");
        for(int j=0;j<BATCH_SIZE;j++){
            for(int i=0;i<FPTRU_N;i++){
                printf("%d,",a[j].coeffs[i]);
            }
            printf("\n");
        }

    }
    __syncthreads();*/

    pack_ct_1(&r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x],&a[blockIdx.x]);
}


__device__ void Decode_gpu(uint16_t *out, unsigned char *S, uint16_t *M, const long long len)
{


    if (len == 1)
    {
        // if (M[0] == 1)
        //     *out = 0;
        // else if (M[0] <= 256)
        //     *out = uint32_mod_uint14(S[0], M[0]);
        // else
        //     *out = uint32_mod_uint14(S[0] + (((uint16_t)S[1]) << 8), M[0]);
    }

    if (len > 1)
    {
        uint16_t R2[(FPTRU_N + 1) / 2];
        uint16_t M2[(FPTRU_N + 1) / 2];
        uint16_t bottomr[FPTRU_N / 2];
        uint32_t bottomt[FPTRU_N / 2];
        long long i;
        // for (i = 0; i < len - 1; i += 2)
        // {
        //     uint32_t m = M[i] * (uint32_t)M[i + 1];
        //     if (m > 256 * 16383)
        //     {
        //         bottomt[i / 2] = 256 * 256;
        //         bottomr[i / 2] = S[0] + 256 * S[1];
        //         S += 2;
        //         M2[i / 2] = (((m + 255) >> 8) + 255) >> 8;
        //     }
        //     else if (m >= 16384)
        //     {
        //         bottomt[i / 2] = 256;
        //         bottomr[i / 2] = S[0];
        //         S += 1;
        //         M2[i / 2] = (m + 255) >> 8;
        //     }
        //     else
        //     {
        //         bottomt[i / 2] = 1;
        //         bottomr[i / 2] = 0;
        //         M2[i / 2] = m;
        //     }
        // }
        // if (i < len)
        //     M2[i / 2] = M[i];

        Decode_gpu(R2, S, M2, (len + 1) / 2);

        // for (i = 0; i < len - 1; i += 2)
        // {
        //     uint32_t r = bottomr[i / 2];
        //     uint32_t r1;
        //     uint16_t r0;
        //     r += bottomt[i / 2] * R2[i / 2];
        //     uint32_divmod_uint14(&r1, &r0, r, M[i]);
        //     r1 = uint32_mod_uint14(r1, M[i + 1]);
        //     *out++ = r0;
        //     *out++ = r1;
        // }
        // if (i < len)
        //     *out++ = R2[i / 2];
    }
}

__device__ void unpack_pk_gpu(poly *r, unsigned char *a)
{
    uint16_t R[FPTRU_N], M[FPTRU_N];

    for (int i = 0; i < FPTRU_N; ++i)
        M[i] = FPTRU_Q;

    Decode_gpu(R, a, M, FPTRU_N);

    for (int i = 0; i < FPTRU_N; ++i)
        r->coeffs[i] = ((int16_t)R[i]);
}

__global__ void unpack_pk_batch(poly *r, unsigned char *a){
    unpack_pk_gpu(&r[blockIdx.x],&a[blockIdx.x*FPTRU_KEM_PUBLICKEYBYTES]);
}
/*2024-7-28:
线程组织形式
<<<BATCH_SIZE,FPTRU_N/8 + 1(82),0,stream>>>*/
#include "coding.h"
#include "reduce.h"
__global__ void poly_fqcsubq_encode_compress_batch_pack_ct_653(unsigned char *r, poly *array_sigma, unsigned char *arrar_msg, int interval){
    unsigned char * msg = &arrar_msg[interval * blockIdx.x];
    //__shared__ poly sigma;

    //如果读一次很耗时的话，那么不如直接自己读sigma 
    int16_t coes[8];
    coes[0] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8];
    coes[1] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8 + 1];
    coes[2] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8 + 2];
    coes[3] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8 + 3];
    coes[4] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8 + 4];

    //TODO:设计一下，使得idx为FPTRU_N/8时,读取的值少一点->1.测试一下和使用分支的区别 2.能否形成合并内存访问，会不会造成浪费
    coes[5] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 5) * (threadIdx.x != FPTRU_N/8)];
    coes[6] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 6) * (threadIdx.x != FPTRU_N/8)];
    coes[7] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 7) * (threadIdx.x != FPTRU_N/8)];

    int k = threadIdx.x >> 1;

    uint8_t tmp,mh;
    int16_t mask,t;

    tmp = (threadIdx.x >= FPTRU_MSGBYTES * 2) ? 0 : (msg[k] >> ((threadIdx.x %2 ) << 2)) & 0xF; //这里的判断语句和一般的判断语句有什么区别呢
    /*if(threadIdx.x >= FPTRU_MSGBYTES * 2){
        tmp = 0;
    }
    else{
        tmp = (msg[k] >> ((threadIdx.x %2 ) << 2)) & 0xF;
    }*/
    mh = encode_e8(tmp);

    //后面可以一直使用mh

    mask = -(int16_t)((mh >> 0) & 1);
    t = (int32_t)((fqcsubq(coes[0]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[0] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 1) & 1);
    t = (int32_t)((fqcsubq(coes[1]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[1] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 2) & 1);
    t = (int32_t)((fqcsubq(coes[2]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[2] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 3) & 1);
    t = (int32_t)((fqcsubq(coes[3]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[3] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 4) & 1);
    t = (int32_t)((fqcsubq(coes[4]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[4] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 5) & 1);
    t = (int32_t)((fqcsubq(coes[5]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[5] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 6) & 1);
    t = (int32_t)((fqcsubq(coes[6]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[6] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 7) & 1);
    t = (int32_t)((fqcsubq(coes[7]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[7] = t & (FPTRU_Q2 - 1);

    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 0] = coes[0] >> 0;
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 1] = coes[0] >> 8 | ((int16_t)coes[1] << 3);
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 2] = coes[1] >> 5 | ((int16_t)coes[2] << 6);
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 3] = coes[2] >> 2;
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 4] = coes[2] >> 10 | ((int16_t)coes[3] << 1);
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 5] = coes[3] >> 7 | ((int16_t)coes[4] << 4);
    if(threadIdx.x < FPTRU_N/8){
        //做额外的赋值操作
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 6] = coes[4] >> 4 | ((int16_t)coes[5] << 7);
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 7] = coes[5] >> 1;
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 8] = coes[5] >> 9 | ((int16_t)coes[6] << 2);
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 9] = coes[6] >> 6 | ((int16_t)coes[7] << 5);
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 10] = coes[7] >> 3;
    }
    else{
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 11 * threadIdx.x + 6] = (coes[4] >> 4) & 0x7f;
    }
}


__global__ void poly_fqcsubq_encode_compress_batch_pack_ct_761(unsigned char *r, poly *array_sigma, unsigned char *arrar_msg, int interval){
    unsigned char * msg = &arrar_msg[interval * blockIdx.x];
    //__shared__ poly sigma;

    //如果读一次很耗时的话，那么不如直接自己读sigma 
    int16_t coes[8];
    coes[0] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8];
    coes[1] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 1) * (threadIdx.x != FPTRU_N/8)];
    coes[2] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 2) * (threadIdx.x != FPTRU_N/8)];
    coes[3] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 3) * (threadIdx.x != FPTRU_N/8)];
    coes[4] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 4) * (threadIdx.x != FPTRU_N/8)];
    coes[5] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 5) * (threadIdx.x != FPTRU_N/8)];
    coes[6] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 6) * (threadIdx.x != FPTRU_N/8)];
    coes[7] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 7) * (threadIdx.x != FPTRU_N/8)];

    int k = threadIdx.x >> 1;

    uint8_t tmp,mh;
    int16_t mask,t;

    tmp = (threadIdx.x >= FPTRU_MSGBYTES * 2) ? 0 : (msg[k] >> ((threadIdx.x %2 ) << 2)) & 0xF; //这里的判断语句和一般的判断语句有什么区别呢
    /*if(threadIdx.x >= FPTRU_MSGBYTES * 2){
        tmp = 0;
    }
    else{
        tmp = (msg[k] >> ((threadIdx.x %2 ) << 2)) & 0xF;
    }*/
    mh = encode_e8(tmp);

    //后面可以一直使用mh

    mask = -(int16_t)((mh >> 0) & 1);
    t = (int32_t)((fqcsubq(coes[0]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[0] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 1) & 1);
    t = (int32_t)((fqcsubq(coes[1]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[1] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 2) & 1);
    t = (int32_t)((fqcsubq(coes[2]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[2] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 3) & 1);
    t = (int32_t)((fqcsubq(coes[3]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[3] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 4) & 1);
    t = (int32_t)((fqcsubq(coes[4]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[4] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 5) & 1);
    t = (int32_t)((fqcsubq(coes[5]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[5] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 6) & 1);
    t = (int32_t)((fqcsubq(coes[6]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[6] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 7) & 1);
    t = (int32_t)((fqcsubq(coes[7]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[7] = t & (FPTRU_Q2 - 1);

    
    if(threadIdx.x == FPTRU_N/8){//95
        //做额外的赋值操作
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 0] = coes[0] >> 0;
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 1] = (coes[0] >> 8) & 0x3;
    }
    else{//0-94
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 0] = coes[0] >> 0;
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 0] = coes[4] >> 0;

        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 1] = (coes[0] >> 8) | ((int16_t)coes[1] << 2);
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 1] = (coes[4] >> 8) | ((int16_t)coes[5] << 2);

        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 2] = (coes[1] >> 6) | ((int16_t)coes[2] << 4);
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 2] = (coes[5] >> 6) | ((int16_t)coes[6] << 4);

        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 3] = (coes[2] >> 4) | ((int16_t)coes[3] << 6);
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 3] = (coes[6] >> 4) | ((int16_t)coes[7] << 6);

        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 4] = coes[3] >> 2;;
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 4] = coes[7] >> 2;
    }
}


__global__ void poly_fqcsubq_encode_compress_batch_pack_ct_1277(unsigned char *r, poly *array_sigma, unsigned char *arrar_msg, int interval){
    unsigned char * msg = &arrar_msg[interval * blockIdx.x];
    //__shared__ poly sigma;

    //如果读一次很耗时的话，那么不如直接自己读sigma 
    int16_t coes[8];
    coes[0] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8];
    coes[1] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8 + 1];
    coes[2] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8 + 2];
    coes[3] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8 + 3];
    coes[4] = array_sigma[blockIdx.x].coeffs[threadIdx.x * 8 + 4];
    coes[5] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 5) * (threadIdx.x != FPTRU_N/8)];
    coes[6] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 6) * (threadIdx.x != FPTRU_N/8)];
    coes[7] = array_sigma[blockIdx.x].coeffs[(threadIdx.x * 8 + 7) * (threadIdx.x != FPTRU_N/8)];

    int k = threadIdx.x >> 1;

    uint8_t tmp,mh;
    int16_t mask,t;

    tmp = (threadIdx.x >= FPTRU_MSGBYTES * 2) ? 0 : (msg[k] >> ((threadIdx.x %2 ) << 2)) & 0xF; //这里的判断语句和一般的判断语句有什么区别呢
    /*if(threadIdx.x >= FPTRU_MSGBYTES * 2){
        tmp = 0;
    }
    else{
        tmp = (msg[k] >> ((threadIdx.x %2 ) << 2)) & 0xF;
    }*/
    mh = encode_e8(tmp);

    //后面可以一直使用mh

    mask = -(int16_t)((mh >> 0) & 1);
    t = (int32_t)((fqcsubq(coes[0]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[0] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 1) & 1);
    t = (int32_t)((fqcsubq(coes[1]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[1] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 2) & 1);
    t = (int32_t)((fqcsubq(coes[2]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[2] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 3) & 1);
    t = (int32_t)((fqcsubq(coes[3]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[3] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 4) & 1);
    t = (int32_t)((fqcsubq(coes[4]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[4] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 5) & 1);
    t = (int32_t)((fqcsubq(coes[5]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[5] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 6) & 1);
    t = (int32_t)((fqcsubq(coes[6]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[6] = t & (FPTRU_Q2 - 1);

    mask = -(int16_t)((mh >> 7) & 1);
    t = (int32_t)((fqcsubq(coes[7]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
    t = t + (mask & (FPTRU_Q2 >> 1));
    coes[7] = t & (FPTRU_Q2 - 1);

    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 0] = coes[0] >> 0;
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 1] = (coes[0] >> 8) | ((int16_t)coes[1] << 2);
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 2] = (coes[1] >> 6) | ((int16_t)coes[2] << 4);
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 3] = (coes[2] >> 4) | ((int16_t)coes[3] << 6);
    r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2) + 4] = coes[3] >> 2;;
    
    if(threadIdx.x == FPTRU_N/8){ //159
        //做额外的赋值操作
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 0] = coes[4] >> 0;
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 1] = (coes[4] >> 8) & 0x3;
    }
    else{ //0-158
        
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 0] = coes[4] >> 0;

        
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 1] = (coes[4] >> 8) | ((int16_t)coes[5] << 2);

        
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 2] = (coes[5] >> 6) | ((int16_t)coes[6] << 4);

        
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 3] = (coes[6] >> 4) | ((int16_t)coes[7] << 6);

        
        r[FPTRU_PKE_CIPHERTEXTBYTES*blockIdx.x + 5 * (threadIdx.x * 2 + 1) + 4] = coes[7] >> 2;
    }
}