#include <iostream>
#include <hip/hip_runtime.h>
__device__ void dd(int x,uint16_t * a,uint16_t *b, uint16_t *c, uint16_t *d){
    uint16_t R2[1277];//uint16_t R2[(len + 1) / 2];
    uint16_t M2[1277];//uint16_t M2[(len + 1) / 2];
    uint16_t bottomr[1277];//uint16_t bottomr[len / 2];
    uint16_t bottomt[1277];//uint32_t bottomt[len / 2];
    a[0] = 10;
    if(x>0){
        printf("%d\n",x);
        dd(x-1,R2,M2,bottomr,bottomt);
    }
}
__global__ void digui(){
    uint16_t R2[1277];//uint16_t R2[(len + 1) / 2];
    uint16_t M2[1277];//uint16_t M2[(len + 1) / 2];
    uint16_t bottomr[1277];//uint16_t bottomr[len / 2];
    uint16_t bottomt[1277];//uint32_t bottomt[len / 2];
    printf("hi\n");
    dd(12,R2,M2,bottomr,bottomt);
}
int main(){
    digui<<<1,1>>>();
    hipDeviceSynchronize();
}