#include "hip/hip_runtime.h"

#include "n1277.h"

__device__ int32_t zetas_n1277[N_N1277 / 5] = {
    5592919, 33026177, 26790957, 33026177, 13499153, 21938565, 26790957, 33026177, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    32738467, 16657861, 28138468, 501067, 16015129, 21620241, 1529124, 28870577, 18285219, 3050639, 27906268, 21100145, 5343767, 15280732, 12498775, 17354707,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    9795452, 12096975, 33505735, 10865068, 16076627, 23564159, 31258325, 19464215, 24580770, 6067746, 18546326, 875322, 23054283, 15379670, 17941644, 2515533,
    17182522, 9742026, 11427911, 16642093, 24272961, 6983968, 22883012, 17558876, 10658623, 29624690, 1466133, 20016426, 18447657, 8460173, 19793647, 24333065,
    32738467, 16657861, 28138468, 501067, 16015129, 21620241, 1529124, 28870577, 18285219, 3050639, 27906268, 21100145, 5343767, 15280732, 12498775, 17354707,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    10420445, 32038148, 2264, 33265110, 25958325, 11902971, 25622482, 27761690, 28792776, 33361188, 24969419, 12394212, 33193998, 6229980, 8743221, 4457820,
    28382998, 8431289, 1312494, 4681025, 29392120, 295408, 2951574, 9444158, 16620023, 32579155, 31375729, 11964204, 33164555, 16089224, 25578586, 22029388,
    19233249, 12984689, 30813171, 15944111, 13539352, 27143752, 7098522, 30855744, 15029351, 2274807, 31244694, 16958068, 23199393, 17755942, 6583826, 30969403,
    1918505, 22360437, 26047052, 3762725, 10986757, 5860566, 28381447, 3454840, 31955344, 13111549, 10268342, 11767139, 6958801, 13085226, 6345790, 33276057,
    9795452, 12096975, 33505735, 10865068, 16076627, 23564159, 31258325, 19464215, 24580770, 6067746, 18546326, 875322, 23054283, 15379670, 17941644, 2515533,
    17182522, 9742026, 11427911, 16642093, 24272961, 6983968, 22883012, 17558876, 10658623, 29624690, 1466133, 20016426, 18447657, 8460173, 19793647, 24333065,
    32738467, 16657861, 28138468, 501067, 16015129, 21620241, 1529124, 28870577, 18285219, 3050639, 27906268, 21100145, 5343767, 15280732, 12498775, 17354707,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177,
    5592919, 31075518, 21303665, 22682749, 11872764, 2432073, 19322199, 11292275, 23696636, 32659557, 23346277, 31556495, 21579819, 5882189, 6581391, 32328326,
    10696307, 16874833, 31719194, 6911566, 29285890, 23963075, 31256330, 9623777, 31413463, 32254537, 11613809, 24253081, 24356853, 24689701, 17056149, 15482269,
    10167038, 22751501, 18535303, 3968231, 1394219, 26379347, 16811031, 28313524, 16967280, 23729319, 31158593, 2331434, 549280, 18765598, 27900090, 4955113,
    1126043, 7409310, 26996082, 3493835, 5276286, 25397512, 30230071, 26868776, 1340079, 15120857, 24105137, 12241024, 12565260, 33504217, 7283953, 25114323,
    7865111, 32661754, 22892358, 6749048, 9488390, 10088648, 22153949, 5957087, 6543554, 17785943, 9563220, 4840264, 22490456, 6137845, 17318481, 17583519,
    24218356, 23955081, 26738484, 22876568, 3544406, 28057775, 18074286, 26082996, 32554080, 21141507, 15212341, 7347621, 25101513, 16054630, 18249116, 7050722,
    2250785, 7291436, 3750223, 26357349, 10321733, 17918503, 8792503, 33073869, 21274321, 10730693, 6218037, 32966155, 23321992, 20997298, 25154884, 8989918,
    5527986, 7143126, 31109078, 27760189, 9686871, 5400208, 13544313, 4038243, 9254376, 26381630, 18797971, 13377401, 350844, 19639872, 20608463, 28613855,
    10420445, 32038148, 2264, 33265110, 25958325, 11902971, 25622482, 27761690, 28792776, 33361188, 24969419, 12394212, 33193998, 6229980, 8743221, 4457820,
    28382998, 8431289, 1312494, 4681025, 29392120, 295408, 2951574, 9444158, 16620023, 32579155, 31375729, 11964204, 33164555, 16089224, 25578586, 22029388,
    19233249, 12984689, 30813171, 15944111, 13539352, 27143752, 7098522, 30855744, 15029351, 2274807, 31244694, 16958068, 23199393, 17755942, 6583826, 30969403,
    1918505, 22360437, 26047052, 3762725, 10986757, 5860566, 28381447, 3454840, 31955344, 13111549, 10268342, 11767139, 6958801, 13085226, 6345790, 33276057,
    9795452, 12096975, 33505735, 10865068, 16076627, 23564159, 31258325, 19464215, 24580770, 6067746, 18546326, 875322, 23054283, 15379670, 17941644, 2515533,
    17182522, 9742026, 11427911, 16642093, 24272961, 6983968, 22883012, 17558876, 10658623, 29624690, 1466133, 20016426, 18447657, 8460173, 19793647, 24333065,
    32738467, 16657861, 28138468, 501067, 16015129, 21620241, 1529124, 28870577, 18285219, 3050639, 27906268, 21100145, 5343767, 15280732, 12498775, 17354707,
    31350138, 24124141, 10149685, 3198144, 32900457, 1140056, 24000204, 27372525, 10764447, 19493511, 9265027, 5611999, 13499153, 21938565, 26790957, 33026177};

__device__ int32_t zetas_inv_n1277[N_N1277 / 5] = {
    4936482, 12941874, 13910465, 33199493, 20172936, 14752366, 7168707, 24295961, 29512094, 20006024, 28150129, 23863466, 5790148, 2441259, 26407211, 28022351,
    24560419, 8395453, 12553039, 10228345, 584182, 27332300, 22819644, 12276016, 476468, 24757834, 15631834, 23228604, 7192988, 29800114, 26258901, 31299552,
    26499615, 15301221, 17495707, 8448824, 26202716, 18337996, 12408830, 996257, 7467341, 15476051, 5492562, 30005931, 10673769, 6811853, 9595256, 9331981,
    15966818, 16231856, 27412492, 11059881, 28710073, 23987117, 15764394, 27006783, 27593250, 11396388, 23461689, 24061947, 26801289, 10657979, 888583, 25685226,
    8436014, 26266384, 46120, 20985077, 21309313, 9445200, 18429480, 32210258, 6681561, 3320266, 8152825, 28274051, 30056502, 6554255, 26141027, 32424294,
    28595224, 5650247, 14784739, 33001057, 31218903, 2391744, 9821018, 16583057, 5236813, 16739306, 7170990, 32156118, 29582106, 15015034, 10798836, 23383299,
    18068068, 16494188, 8860636, 9193484, 9297256, 21936528, 1295800, 2136874, 23926560, 2294007, 9587262, 4264447, 26638771, 1831143, 16675504, 22854030,
    1222011, 26968946, 27668148, 11970518, 1993842, 10204060, 890780, 9853701, 22258062, 14228138, 31118264, 21677573, 10867588, 12246672, 2474819, 27957418,
    274280, 27204547, 20465111, 26591536, 21783198, 23281995, 20438788, 1594993, 30095497, 5168890, 27689771, 22563580, 29787612, 7503285, 11189900, 31631832,
    2580934, 26966511, 15794395, 10350944, 16592269, 2305643, 31275530, 18520986, 2694593, 26451815, 6406585, 20010985, 17606226, 2737166, 20565648, 14317088,
    11520949, 7971751, 17461113, 385782, 21586133, 2174608, 971182, 16930314, 24106179, 30598763, 33254929, 4158217, 28869312, 32237843, 25119048, 5167339,
    29092517, 24807116, 27320357, 356339, 21156125, 8580918, 189149, 4757561, 5788647, 7927855, 21647366, 7592012, 285227, 33548073, 1512189, 23129892,
    9217272, 13756690, 25090164, 15102680, 13533911, 32084204, 3925647, 22891714, 15991461, 10667325, 26566369, 9277376, 16908244, 22122426, 23808311, 16367815,
    31034804, 15608693, 18170667, 10496054, 32675015, 15004011, 27482591, 8969567, 14086122, 2292012, 9986178, 17473710, 22685269, 44602, 21453362, 23754885,
    16195630, 21051562, 18269605, 28206570, 12450192, 5644069, 30499698, 15265118, 4679760, 32021213, 11930096, 17535208, 33049270, 5411869, 16892476, 811870,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    274280, 27204547, 20465111, 26591536, 21783198, 23281995, 20438788, 1594993, 30095497, 5168890, 27689771, 22563580, 29787612, 7503285, 11189900, 31631832,
    2580934, 26966511, 15794395, 10350944, 16592269, 2305643, 31275530, 18520986, 2694593, 26451815, 6406585, 20010985, 17606226, 2737166, 20565648, 14317088,
    11520949, 7971751, 17461113, 385782, 21586133, 2174608, 971182, 16930314, 24106179, 30598763, 33254929, 4158217, 28869312, 32237843, 25119048, 5167339,
    29092517, 24807116, 27320357, 356339, 21156125, 8580918, 189149, 4757561, 5788647, 7927855, 21647366, 7592012, 285227, 33548073, 1512189, 23129892,
    9217272, 13756690, 25090164, 15102680, 13533911, 32084204, 3925647, 22891714, 15991461, 10667325, 26566369, 9277376, 16908244, 22122426, 23808311, 16367815,
    31034804, 15608693, 18170667, 10496054, 32675015, 15004011, 27482591, 8969567, 14086122, 2292012, 9986178, 17473710, 22685269, 44602, 21453362, 23754885,
    16195630, 21051562, 18269605, 28206570, 12450192, 5644069, 30499698, 15265118, 4679760, 32021213, 11930096, 17535208, 33049270, 5411869, 16892476, 811870,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    9217272, 13756690, 25090164, 15102680, 13533911, 32084204, 3925647, 22891714, 15991461, 10667325, 26566369, 9277376, 16908244, 22122426, 23808311, 16367815,
    31034804, 15608693, 18170667, 10496054, 32675015, 15004011, 27482591, 8969567, 14086122, 2292012, 9986178, 17473710, 22685269, 44602, 21453362, 23754885,
    16195630, 21051562, 18269605, 28206570, 12450192, 5644069, 30499698, 15265118, 4679760, 32021213, 11930096, 17535208, 33049270, 5411869, 16892476, 811870,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    16195630, 21051562, 18269605, 28206570, 12450192, 5644069, 30499698, 15265118, 4679760, 32021213, 11930096, 17535208, 33049270, 5411869, 16892476, 811870,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    6177812, 9550133, 32410281, 649880, 30352193, 23400652, 9426196, 2200199, 27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177,
    27938338, 24285310, 14056826, 22785890, 11611772, 20051184, 6759380, 33026177, 11611772, 20051184, 6759380, 33026177, 6759380, 33026177, 33026177};


__device__ int32_t montgomery_reduce_n1277_cuda(int64_t a)
{
    int32_t t;
    t = (int32_t)a * NUMBER_SCOMOJ;
    t = (a - (int64_t)t * Q_N1277) >> 32;
    return t;
}

__device__ int32_t pseudomersenne_reduce_single_n1277_cuda(int32_t a)
{
    int32_t t0, t1;

    t0 = a & 0x1ffffff;
    t1 = a >> 25;
    t0 = (t1 << 12) + t0 - t1 - Q_N1277;

    return t0;
}



#define special_CT_a(i,j)  t = a[j];a[j]=(a[i] + t);a[i] = (a[i] - t);
#define special_CT_b(i,j)  t = b[j];b[j]=(b[i] + t);b[i] = (b[i] - t);

#define pse_CT_a(i,j)  t = a[j];a[j]=(a[i] + t);a[i] = pseudomersenne_reduce_single_n1277_cuda(a[i] - t);
#define pse_CT_b(i,j)  t = b[j];b[j]=(b[i] + t);b[i] = pseudomersenne_reduce_single_n1277_cuda(b[i] - t);

#define sCT_b1(i,j) t = FACTOR_CSONFIDC * b[j];b[j] = (b[i] - t);b[i] = (b[i] + t);
#define sCT_b2(i,j) t = b[j];b[j]=(b[i] + t);b[i] = b[i] - t;
        
#define CT_a(i,j,zeta)  t = montgomery_reduce_n1277_cuda((int64_t)zeta * a[j]); a[j] = (a[i] - t); a[i] = (a[i] + t);

#define CT_b(i,j,zeta)  t = montgomery_reduce_n1277_cuda((int64_t)zeta * b[j]); b[j] = (b[i] - t); b[i] = (b[i] + t);

#define GS_a(i,j,zeta) t = a[i]; a[i] = (t + a[j]); a[j] = (t - a[j]); a[j] = montgomery_reduce_n1277_cuda((int64_t)zeta * a[j]);

#define pse_GS_a(i,j) t = a[i]; a[i] = pseudomersenne_reduce_single_n1277_cuda(t + a[j]); a[j] = pseudomersenne_reduce_single_n1277_cuda(a[j]-t);

#define special_GS_a(i,j) t = a[i];a[i] = (a[j] + t);a[j] = (a[j] - t);

#define KARA(a, b, x, y, d) ((montgomery_reduce_n1277_cuda((int64_t)(a[x] + a[y]) * (b[x] + b[y])) - d[x] - d[y]))


__device__ void basemul_n1277_cuda(int32_t c[5], const int32_t a[5], const int32_t b[5], const int32_t zeta)
{
    int i;
    int32_t d[5];

    for (i = 0; i < 5; i++)
        d[i] = montgomery_reduce_n1277_cuda((int64_t)a[i] * b[i]);

    c[0] = (d[0] + montgomery_reduce_n1277_cuda((int64_t)zeta * (KARA(a, b, 1, 4, d) + KARA(a, b, 2, 3, d))));
    c[1] = (KARA(a, b, 0, 1, d) + montgomery_reduce_n1277_cuda((int64_t)zeta * (KARA(a, b, 2, 4, d) + d[3])));
    c[2] = (KARA(a, b, 0, 2, d) + d[1] + montgomery_reduce_n1277_cuda((int64_t)zeta * (KARA(a, b, 3, 4, d))));
    c[3] = (KARA(a, b, 0, 3, d) + KARA(a, b, 1, 2, d) + montgomery_reduce_n1277_cuda((int64_t)zeta * (d[4])));
    c[4] = (KARA(a, b, 0, 4, d) + KARA(a, b, 1, 3, d) + d[2]);
}


__global__ void poly_mul_1277_batch_q1(poly * array_c,poly * array_a,poly * array_b){
    __shared__ nttpoly_n1277 ntta;
    __shared__ nttpoly_n1277 nttb;
    __shared__ nttpoly_n1277 nttc;

    int32_t a[8]={0};
    int32_t b[8]={0};

    int32_t zeta;
    int32_t t;

    a[0] = array_a[blockIdx.x].coeffs[threadIdx.x];
    a[1] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    a[2] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];
    

    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) a[3] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];


    b[0] = array_b[blockIdx.x].coeffs[threadIdx.x];
    b[1] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    b[2] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];
    
    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) b[3] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];


    //1 基2
    special_CT_a(0,4);
    special_CT_a(1,5);
    special_CT_a(2,6);
    special_CT_a(3,7);

    special_CT_b(0,4);
    special_CT_b(1,5);
    special_CT_b(2,6);
    special_CT_b(3,7);

    //2 基2
    zeta = zetas_n1277[2];

    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    // CT_b(0,2,zeta);
    // CT_b(1,3,zeta);
    sCT_b1(0,2);
    sCT_b1(1,3);

    pse_CT_a(4,6);
    pse_CT_a(5,7);

    // pse_CT_b(4,6);
    // pse_CT_b(5,7);
    sCT_b2(4,6);
    sCT_b2(5,7);


    //3 基2
    zeta = zetas_n1277[4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n1277[5];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n1277[6];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    pse_CT_a(6,7);
    pse_CT_b(6,7);

    //放回共享内存
    ntta.coeffs[threadIdx.x] = a[0];
    ntta.coeffs[threadIdx.x + blockDim.x] = a[1];
    ntta.coeffs[threadIdx.x + blockDim.x * 2] = a[2];
    ntta.coeffs[threadIdx.x + blockDim.x * 3] = a[3];
    ntta.coeffs[threadIdx.x + blockDim.x * 4] = a[4];
    ntta.coeffs[threadIdx.x + blockDim.x * 5] = a[5];
    ntta.coeffs[threadIdx.x + blockDim.x * 6] = a[6];
    ntta.coeffs[threadIdx.x + blockDim.x * 7] = a[7];

    nttb.coeffs[threadIdx.x] = b[0];
    nttb.coeffs[threadIdx.x + blockDim.x] = b[1];
    nttb.coeffs[threadIdx.x + blockDim.x * 2] = b[2];
    nttb.coeffs[threadIdx.x + blockDim.x * 3] = b[3];
    nttb.coeffs[threadIdx.x + blockDim.x * 4] = b[4];
    nttb.coeffs[threadIdx.x + blockDim.x * 5] = b[5];
    nttb.coeffs[threadIdx.x + blockDim.x * 6] = b[6];
    nttb.coeffs[threadIdx.x + blockDim.x * 7] = b[7];

    __syncthreads();


    int group_num = threadIdx.x / 40;
    int group_idx = threadIdx.x % 40;
    int start = group_num * 320;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 40];
    a[2] = ntta.coeffs[start + group_idx + 40 * 2];
    a[3] = ntta.coeffs[start + group_idx + 40 * 3];
    a[4] = ntta.coeffs[start + group_idx + 40 * 4];
    a[5] = ntta.coeffs[start + group_idx + 40 * 5];
    a[6] = ntta.coeffs[start + group_idx + 40 * 6];
    a[7] = ntta.coeffs[start + group_idx + 40 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 40];
    b[2] = nttb.coeffs[start + group_idx + 40 * 2];
    b[3] = nttb.coeffs[start + group_idx + 40 * 3];
    b[4] = nttb.coeffs[start + group_idx + 40 * 4];
    b[5] = nttb.coeffs[start + group_idx + 40 * 5];
    b[6] = nttb.coeffs[start + group_idx + 40 * 6];
    b[7] = nttb.coeffs[start + group_idx + 40 * 7];


    //4 基2
    zeta = zetas_n1277[8 + group_num];

    if (zeta == NUMBER_CSNNH){
        pse_CT_a(0,4);
        pse_CT_a(1,5);
        pse_CT_a(2,6);
        pse_CT_a(3,7);

        pse_CT_b(0,4);
        pse_CT_b(1,5);
        pse_CT_b(2,6);
        pse_CT_b(3,7);
    }

    else{
        CT_a(0,4,zeta);
        CT_a(1,5,zeta);
        CT_a(2,6,zeta);
        CT_a(3,7,zeta);

        CT_b(0,4,zeta);
        CT_b(1,5,zeta);
        CT_b(2,6,zeta);
        CT_b(3,7,zeta);
    }
    

    //5 基2
    zeta = zetas_n1277[16 + group_num * 2];
    
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n1277[16 + group_num * 2 + 1];

    if (zeta == NUMBER_CSNNH){
        pse_CT_a(4,6);
        pse_CT_a(5,7);

        pse_CT_b(4,6);
        pse_CT_b(5,7);
    }
    else{
        CT_a(4,6,zeta);
        CT_a(5,7,zeta);

        CT_b(4,6,zeta);
        CT_b(5,7,zeta);
    }
    

    //6 基2
    zeta = zetas_n1277[32 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n1277[32 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n1277[32 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    // 7
    zeta = zetas_n1277[32 + group_num * 4 + 3];

    if(zeta == NUMBER_CSNNH){
        pse_CT_a(6,7);
        pse_CT_b(6,7);
    }
    else{
        CT_a(6,7,zeta);
        CT_b(6,7,zeta);
    }
    

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 40] = a[1];
    ntta.coeffs[start + group_idx + 40 * 2] = a[2];
    ntta.coeffs[start + group_idx + 40 * 3] = a[3];
    ntta.coeffs[start + group_idx + 40 * 4] = a[4];
    ntta.coeffs[start + group_idx + 40 * 5] = a[5];
    ntta.coeffs[start + group_idx + 40 * 6] = a[6];
    ntta.coeffs[start + group_idx + 40 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 40] = b[1];
    nttb.coeffs[start + group_idx + 40 * 2] = b[2];
    nttb.coeffs[start + group_idx + 40 * 3] = b[3];
    nttb.coeffs[start + group_idx + 40 * 4] = b[4];
    nttb.coeffs[start + group_idx + 40 * 5] = b[5];
    nttb.coeffs[start + group_idx + 40 * 6] = b[6];
    nttb.coeffs[start + group_idx + 40 * 7] = b[7];

    __syncthreads();


    group_num = threadIdx.x / 5;
    group_idx = threadIdx.x % 5;
    start = group_num * 40;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 5];
    a[2] = ntta.coeffs[start + group_idx + 5 * 2];
    a[3] = ntta.coeffs[start + group_idx + 5 * 3];
    a[4] = ntta.coeffs[start + group_idx + 5 * 4];
    a[5] = ntta.coeffs[start + group_idx + 5 * 5];
    a[6] = ntta.coeffs[start + group_idx + 5 * 6];
    a[7] = ntta.coeffs[start + group_idx + 5 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 5];
    b[2] = nttb.coeffs[start + group_idx + 5 * 2];
    b[3] = nttb.coeffs[start + group_idx + 5 * 3];
    b[4] = nttb.coeffs[start + group_idx + 5 * 4];
    b[5] = nttb.coeffs[start + group_idx + 5 * 5];
    b[6] = nttb.coeffs[start + group_idx + 5 * 6];
    b[7] = nttb.coeffs[start + group_idx + 5 * 7];

    //7 基2
    zeta = zetas_n1277[64 + group_num];
    if (zeta == NUMBER_CSNNH){
        pse_CT_a(0,4);
        pse_CT_a(1,5);
        pse_CT_a(2,6);
        pse_CT_a(3,7);

        pse_CT_b(0,4);
        pse_CT_b(1,5);
        pse_CT_b(2,6);
        pse_CT_b(3,7);
    }

    else{
        CT_a(0,4,zeta);
        CT_a(1,5,zeta);
        CT_a(2,6,zeta);
        CT_a(3,7,zeta);

        CT_b(0,4,zeta);
        CT_b(1,5,zeta);
        CT_b(2,6,zeta);
        CT_b(3,7,zeta);
    }

    //8 基2
    zeta = zetas_n1277[128 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n1277[128 + group_num * 2 + 1];
    if (zeta == NUMBER_CSNNH){
        pse_CT_a(4,6);
        pse_CT_a(5,7);

        pse_CT_b(4,6);
        pse_CT_b(5,7);
    }
    else{
        CT_a(4,6,zeta);
        CT_a(5,7,zeta);

        CT_b(4,6,zeta);
        CT_b(5,7,zeta);
    }

    //9 基2
    zeta = zetas_n1277[256 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n1277[256 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n1277[256 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n1277[256 + group_num * 4 + 3];
    if(zeta == NUMBER_CSNNH){
        pse_CT_a(6,7);
        pse_CT_b(6,7);
    }
    else{
        CT_a(6,7,zeta);
        CT_b(6,7,zeta);
    }

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 5] = a[1];
    ntta.coeffs[start + group_idx + 5 * 2] = a[2];
    ntta.coeffs[start + group_idx + 5 * 3] = a[3];
    ntta.coeffs[start + group_idx + 5 * 4] = a[4];
    ntta.coeffs[start + group_idx + 5 * 5] = a[5];
    ntta.coeffs[start + group_idx + 5 * 6] = a[6];
    ntta.coeffs[start + group_idx + 5 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 5] = b[1];
    nttb.coeffs[start + group_idx + 5 * 2] = b[2];
    nttb.coeffs[start + group_idx + 5 * 3] = b[3];
    nttb.coeffs[start + group_idx + 5 * 4] = b[4];
    nttb.coeffs[start + group_idx + 5 * 5] = b[5];
    nttb.coeffs[start + group_idx + 5 * 6] = b[6];
    nttb.coeffs[start + group_idx + 5 * 7] = b[7];
    
    __syncthreads();

    // if(threadIdx.x ==0){
    //     printf("look ntta\n");
    //     for(int i=0;i<N_N1277;i++){
    //         printf("%d,",ntta.coeffs[i]);
    //     }
    //     printf("\n");

    //     printf("look nttb\n");
    //     for(int i=0;i<N_N1277;i++){
    //         printf("%d,",nttb.coeffs[i]);
    //     }
    //     printf("\n");
    // }
    // __syncthreads();
    
    //512 个 basemul

    //0-319个
    zeta = zetas_n1277[256 + threadIdx.x / 2];
    if(threadIdx.x & 1) zeta = -zeta;
    basemul_n1277_cuda(nttc.coeffs + 5 * threadIdx.x, ntta.coeffs + 5 * threadIdx.x,nttb.coeffs + 5 * threadIdx.x,zeta);


    if(threadIdx.x < 192){
        zeta = zetas_n1277[256 + threadIdx.x / 2 + 160];
        if(threadIdx.x & 1) zeta = -zeta;
        basemul_n1277_cuda(nttc.coeffs + 5 * threadIdx.x + 1600, ntta.coeffs + 5 * threadIdx.x + 1600, nttb.coeffs + 5 * threadIdx.x + 1600, zeta);
    }

    __syncthreads();

    //  if(threadIdx.x ==0){
    //     printf("look nttc\n");
    //     for(int i=0;i<N_N1277;i++){
    //         printf("%d,",nttc.coeffs[i]);
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    //inv_ntt
    group_num = threadIdx.x / 5;
    group_idx = threadIdx.x % 5;
    start = group_num * 40;

    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 5];
    a[2] = nttc.coeffs[start + group_idx + 5 * 2];
    a[3] = nttc.coeffs[start + group_idx + 5 * 3];
    a[4] = nttc.coeffs[start + group_idx + 5 * 4];
    a[5] = nttc.coeffs[start + group_idx + 5 * 5];
    a[6] = nttc.coeffs[start + group_idx + 5 * 6];
    a[7] = nttc.coeffs[start + group_idx + 5 * 7];


    //1 基2 256
    zeta = zetas_inv_n1277[group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n1277[group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n1277[group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n1277[group_num * 4 + 3];
    if(zeta == NUMBER_CSNNH){
        special_GS_a(6,7);
    }
    else{
        GS_a(6,7,zeta);
    }
    

    //2 基2 128
    zeta = zetas_inv_n1277[256 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n1277[256 + group_num * 2 + 1];
    if(zeta == NUMBER_CSNNH){
        special_GS_a(4,6);
        special_GS_a(5,7);
    }
    else{
        GS_a(4,6,zeta);
        GS_a(5,7,zeta);
    }
    

    //3 基2 64
    zeta = zetas_inv_n1277[384 + group_num];
    if(zeta == NUMBER_CSNNH){
        special_GS_a(0,4);
        special_GS_a(1,5);
        special_GS_a(2,6);
        special_GS_a(3,7);
    }
    else{
        GS_a(0,4,zeta);
        GS_a(1,5,zeta);
        GS_a(2,6,zeta);
        GS_a(3,7,zeta);
    }
    

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 5] = a[1];
    nttc.coeffs[start + group_idx + 5 * 2] = a[2];
    nttc.coeffs[start + group_idx + 5 * 3] = a[3];
    nttc.coeffs[start + group_idx + 5 * 4] = a[4];
    nttc.coeffs[start + group_idx + 5 * 5] = a[5];
    nttc.coeffs[start + group_idx + 5 * 6] = a[6];
    nttc.coeffs[start + group_idx + 5 * 7] = a[7];

    __syncthreads();

    

    group_num = threadIdx.x / 40;
    group_idx = threadIdx.x % 40;
    start = group_num * 320;

    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 40];
    a[2] = nttc.coeffs[start + group_idx + 40 * 2];
    a[3] = nttc.coeffs[start + group_idx + 40 * 3];
    a[4] = nttc.coeffs[start + group_idx + 40 * 4];
    a[5] = nttc.coeffs[start + group_idx + 40 * 5];
    a[6] = nttc.coeffs[start + group_idx + 40 * 6];
    a[7] = nttc.coeffs[start + group_idx + 40 * 7];
    
#pragma unroll
    for(int i=0;i<8;i++){
        a[i] = pseudomersenne_reduce_single_n1277_cuda(a[i]);
    }

    //4 基2 32
    zeta = zetas_inv_n1277[448 + group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n1277[448 + group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n1277[448 + group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n1277[448 + group_num * 4 + 3];
    if(zeta == NUMBER_CSNNH){
        special_GS_a(6,7);
    }
    else{
        GS_a(6,7,zeta);
    }
    

    
    //5 基2 16
    zeta = zetas_inv_n1277[480 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n1277[480 + group_num * 2 + 1];
    if(zeta == NUMBER_CSNNH){
        special_GS_a(4,6);
        special_GS_a(5,7);
    }
    else{
        GS_a(4,6,zeta);
        GS_a(5,7,zeta);
    }

    //6 基2 8
    zeta = zetas_inv_n1277[496 + group_num];
    if(zeta == NUMBER_CSNNH){
        special_GS_a(0,4);
        special_GS_a(1,5);
        special_GS_a(2,6);
        special_GS_a(3,7);
    }
    else{
        GS_a(0,4,zeta);
        GS_a(1,5,zeta);
        GS_a(2,6,zeta);
        GS_a(3,7,zeta);
    }

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 40] = a[1];
    nttc.coeffs[start + group_idx + 40 * 2] = a[2];
    nttc.coeffs[start + group_idx + 40 * 3] = a[3];
    nttc.coeffs[start + group_idx + 40 * 4] = a[4];
    nttc.coeffs[start + group_idx + 40 * 5] = a[5];
    nttc.coeffs[start + group_idx + 40 * 6] = a[6];
    nttc.coeffs[start + group_idx + 40 * 7] = a[7];

    __syncthreads();
    

    a[0] = nttc.coeffs[threadIdx.x];
    a[1] = nttc.coeffs[threadIdx.x + 320];
    a[2] = nttc.coeffs[threadIdx.x + 320 * 2];
    a[3] = nttc.coeffs[threadIdx.x + 320 * 3];
    a[4] = nttc.coeffs[threadIdx.x + 320 * 4];
    a[5] = nttc.coeffs[threadIdx.x + 320 * 5];
    a[6] = nttc.coeffs[threadIdx.x + 320 * 6];
    a[7] = nttc.coeffs[threadIdx.x + 320 * 7];

    //7 基2 4
    zeta = zetas_inv_n1277[504];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n1277[504 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n1277[504 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n1277[504 + 3];
    if(zeta == NUMBER_CSNNH){
        special_GS_a(6,7);
    }
    else{
        GS_a(6,7,zeta);
    }

    //8 基2 2
    zeta = zetas_inv_n1277[508];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n1277[508 + 1];
    if(zeta == NUMBER_CSNNH){
        special_GS_a(4,6);
        special_GS_a(5,7);
    }
    else{
        GS_a(4,6,zeta);
        GS_a(5,7,zeta);
    }

    //9 基2 1
    //zeta = zetas_inv_n1277[510];
    pse_GS_a(0,4);
    pse_GS_a(1,5);
    pse_GS_a(2,6);
    pse_GS_a(3,7);

    nttc.coeffs[threadIdx.x] = a[0];
    nttc.coeffs[threadIdx.x + 320] = a[1];
    nttc.coeffs[threadIdx.x + 320 * 2] = a[2];
    nttc.coeffs[threadIdx.x + 320 * 3] = a[3];
    nttc.coeffs[threadIdx.x + 320 * 4] = a[4];
    nttc.coeffs[threadIdx.x + 320 * 5] = a[5];
    nttc.coeffs[threadIdx.x + 320 * 6] = a[6];
    nttc.coeffs[threadIdx.x + 320 * 7] = a[7];

    __syncthreads();
    
    // if(threadIdx.x ==0){
    //     printf("look nttc\n");
    //     for(int i=0;i<N_N1277;i++){
    //         printf("%d,",nttc.coeffs[i]);
    //     }
    //     printf("\n");
    // }
    //1 - 320
    array_c[blockIdx.x].coeffs[threadIdx.x + 1] = fq_freeze(montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[threadIdx.x + 1] + nttc.coeffs[threadIdx.x + 1 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 1 + FPTRU_N])));
    
    //321 - 640
    array_c[blockIdx.x].coeffs[threadIdx.x + 321] = fq_freeze(montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[threadIdx.x + 321] + nttc.coeffs[threadIdx.x + 321 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 321 + FPTRU_N])));

    //641 - 960
    array_c[blockIdx.x].coeffs[threadIdx.x + 641] = fq_freeze(montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[threadIdx.x + 641] + nttc.coeffs[threadIdx.x + 641 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 641 + FPTRU_N])));
    
    
    //0,1276, 961-1275
    if(threadIdx.x <= 315){
        if(threadIdx.x == 315){
            array_c[blockIdx.x].coeffs[0] = fq_freeze(montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N])));
            array_c[blockIdx.x].coeffs[FPTRU_N - 1] = fq_freeze(montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2])));
        }
        else{
            array_c[blockIdx.x].coeffs[threadIdx.x + 961] = fq_freeze(montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[threadIdx.x + 961] + nttc.coeffs[threadIdx.x + 961 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 961 + FPTRU_N])));
        }
    }

}



__global__ void poly_mul_1277_batch_q2(poly * array_c,poly * array_a,poly * array_b){
    __shared__ nttpoly_n1277 ntta;
    __shared__ nttpoly_n1277 nttb;
    __shared__ nttpoly_n1277 nttc;

    int32_t a[8]={0};
    int32_t b[8]={0};

    int32_t zeta;
    int32_t t;

    a[0] = array_a[blockIdx.x].coeffs[threadIdx.x];
    a[1] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    a[2] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];
    

    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) a[3] = array_a[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];


    b[0] = array_b[blockIdx.x].coeffs[threadIdx.x];
    b[1] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x];
    b[2] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 2];
    
    if(threadIdx.x + blockDim.x * 3 < FPTRU_N) b[3] = array_b[blockIdx.x].coeffs[threadIdx.x + blockDim.x * 3];


    //1 基2
    special_CT_a(0,4);
    special_CT_a(1,5);
    special_CT_a(2,6);
    special_CT_a(3,7);

    special_CT_b(0,4);
    special_CT_b(1,5);
    special_CT_b(2,6);
    special_CT_b(3,7);

    //1 基2
    zeta = zetas_n1277[2];

    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    pse_CT_a(4,6);
    pse_CT_a(5,7);

    pse_CT_b(4,6);
    pse_CT_b(5,7);

    //3 基2
    zeta = zetas_n1277[4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n1277[5];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n1277[6];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    pse_CT_a(6,7);
    pse_CT_b(6,7);

    //放回共享内存
    ntta.coeffs[threadIdx.x] = a[0];
    ntta.coeffs[threadIdx.x + blockDim.x] = a[1];
    ntta.coeffs[threadIdx.x + blockDim.x * 2] = a[2];
    ntta.coeffs[threadIdx.x + blockDim.x * 3] = a[3];
    ntta.coeffs[threadIdx.x + blockDim.x * 4] = a[4];
    ntta.coeffs[threadIdx.x + blockDim.x * 5] = a[5];
    ntta.coeffs[threadIdx.x + blockDim.x * 6] = a[6];
    ntta.coeffs[threadIdx.x + blockDim.x * 7] = a[7];

    nttb.coeffs[threadIdx.x] = b[0];
    nttb.coeffs[threadIdx.x + blockDim.x] = b[1];
    nttb.coeffs[threadIdx.x + blockDim.x * 2] = b[2];
    nttb.coeffs[threadIdx.x + blockDim.x * 3] = b[3];
    nttb.coeffs[threadIdx.x + blockDim.x * 4] = b[4];
    nttb.coeffs[threadIdx.x + blockDim.x * 5] = b[5];
    nttb.coeffs[threadIdx.x + blockDim.x * 6] = b[6];
    nttb.coeffs[threadIdx.x + blockDim.x * 7] = b[7];

    __syncthreads();

    int group_num = threadIdx.x / 40;
    int group_idx = threadIdx.x % 40;
    int start = group_num * 320;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 40];
    a[2] = ntta.coeffs[start + group_idx + 40 * 2];
    a[3] = ntta.coeffs[start + group_idx + 40 * 3];
    a[4] = ntta.coeffs[start + group_idx + 40 * 4];
    a[5] = ntta.coeffs[start + group_idx + 40 * 5];
    a[6] = ntta.coeffs[start + group_idx + 40 * 6];
    a[7] = ntta.coeffs[start + group_idx + 40 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 40];
    b[2] = nttb.coeffs[start + group_idx + 40 * 2];
    b[3] = nttb.coeffs[start + group_idx + 40 * 3];
    b[4] = nttb.coeffs[start + group_idx + 40 * 4];
    b[5] = nttb.coeffs[start + group_idx + 40 * 5];
    b[6] = nttb.coeffs[start + group_idx + 40 * 6];
    b[7] = nttb.coeffs[start + group_idx + 40 * 7];


    //4 基2
    zeta = zetas_n1277[8 + group_num];
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);

    //5 基2
    zeta = zetas_n1277[16 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n1277[16 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    //6 基2
    zeta = zetas_n1277[32 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n1277[32 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n1277[32 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n1277[32 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 40] = a[1];
    ntta.coeffs[start + group_idx + 40 * 2] = a[2];
    ntta.coeffs[start + group_idx + 40 * 3] = a[3];
    ntta.coeffs[start + group_idx + 40 * 4] = a[4];
    ntta.coeffs[start + group_idx + 40 * 5] = a[5];
    ntta.coeffs[start + group_idx + 40 * 6] = a[6];
    ntta.coeffs[start + group_idx + 40 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 40] = b[1];
    nttb.coeffs[start + group_idx + 40 * 2] = b[2];
    nttb.coeffs[start + group_idx + 40 * 3] = b[3];
    nttb.coeffs[start + group_idx + 40 * 4] = b[4];
    nttb.coeffs[start + group_idx + 40 * 5] = b[5];
    nttb.coeffs[start + group_idx + 40 * 6] = b[6];
    nttb.coeffs[start + group_idx + 40 * 7] = b[7];

    __syncthreads();

    group_num = threadIdx.x / 5;
    group_idx = threadIdx.x % 5;
    start = group_num * 40;

    a[0] = ntta.coeffs[start + group_idx];
    a[1] = ntta.coeffs[start + group_idx + 5];
    a[2] = ntta.coeffs[start + group_idx + 5 * 2];
    a[3] = ntta.coeffs[start + group_idx + 5 * 3];
    a[4] = ntta.coeffs[start + group_idx + 5 * 4];
    a[5] = ntta.coeffs[start + group_idx + 5 * 5];
    a[6] = ntta.coeffs[start + group_idx + 5 * 6];
    a[7] = ntta.coeffs[start + group_idx + 5 * 7];

    b[0] = nttb.coeffs[start + group_idx];
    b[1] = nttb.coeffs[start + group_idx + 5];
    b[2] = nttb.coeffs[start + group_idx + 5 * 2];
    b[3] = nttb.coeffs[start + group_idx + 5 * 3];
    b[4] = nttb.coeffs[start + group_idx + 5 * 4];
    b[5] = nttb.coeffs[start + group_idx + 5 * 5];
    b[6] = nttb.coeffs[start + group_idx + 5 * 6];
    b[7] = nttb.coeffs[start + group_idx + 5 * 7];

    //7 基2
    zeta = zetas_n1277[64 + group_num];
    CT_a(0,4,zeta);
    CT_a(1,5,zeta);
    CT_a(2,6,zeta);
    CT_a(3,7,zeta);

    CT_b(0,4,zeta);
    CT_b(1,5,zeta);
    CT_b(2,6,zeta);
    CT_b(3,7,zeta);

    //8 基2
    zeta = zetas_n1277[128 + group_num * 2];
    CT_a(0,2,zeta);
    CT_a(1,3,zeta);

    CT_b(0,2,zeta);
    CT_b(1,3,zeta);

    zeta = zetas_n1277[128 + group_num * 2 + 1];
    CT_a(4,6,zeta);
    CT_a(5,7,zeta);

    CT_b(4,6,zeta);
    CT_b(5,7,zeta);

    //9 基2
    zeta = zetas_n1277[256 + group_num * 4];
    CT_a(0,1,zeta);
    CT_b(0,1,zeta);

    zeta = zetas_n1277[256 + group_num * 4 + 1];
    CT_a(2,3,zeta);
    CT_b(2,3,zeta);

    zeta = zetas_n1277[256 + group_num * 4 + 2];
    CT_a(4,5,zeta);
    CT_b(4,5,zeta);

    zeta = zetas_n1277[256 + group_num * 4 + 3];
    CT_a(6,7,zeta);
    CT_b(6,7,zeta);

    ntta.coeffs[start + group_idx] = a[0];
    ntta.coeffs[start + group_idx + 5] = a[1];
    ntta.coeffs[start + group_idx + 5 * 2] = a[2];
    ntta.coeffs[start + group_idx + 5 * 3] = a[3];
    ntta.coeffs[start + group_idx + 5 * 4] = a[4];
    ntta.coeffs[start + group_idx + 5 * 5] = a[5];
    ntta.coeffs[start + group_idx + 5 * 6] = a[6];
    ntta.coeffs[start + group_idx + 5 * 7] = a[7];

    nttb.coeffs[start + group_idx] = b[0];
    nttb.coeffs[start + group_idx + 5] = b[1];
    nttb.coeffs[start + group_idx + 5 * 2] = b[2];
    nttb.coeffs[start + group_idx + 5 * 3] = b[3];
    nttb.coeffs[start + group_idx + 5 * 4] = b[4];
    nttb.coeffs[start + group_idx + 5 * 5] = b[5];
    nttb.coeffs[start + group_idx + 5 * 6] = b[6];
    nttb.coeffs[start + group_idx + 5 * 7] = b[7];
    
    __syncthreads();
    
    //512 个 basemul

    //0-319个
    zeta = zetas_n1277[256 + threadIdx.x / 2];
    if(threadIdx.x & 1) zeta = -zeta;
    basemul_n1277_cuda(nttc.coeffs + 5 * threadIdx.x, ntta.coeffs + 5 * threadIdx.x,nttb.coeffs + 5 * threadIdx.x,zeta);


    if(threadIdx.x < 192){
        zeta = zetas_n1277[256 + threadIdx.x / 2 + 160];
        if(threadIdx.x & 1) zeta = -zeta;
        basemul_n1277_cuda(nttc.coeffs + 5 * threadIdx.x + 1600, ntta.coeffs + 5 * threadIdx.x + 1600, nttb.coeffs + 5 * threadIdx.x + 1600, zeta);
    }

    __syncthreads();


    //inv_ntt
    group_num = threadIdx.x / 5;
    group_idx = threadIdx.x % 5;
    start = group_num * 40;

    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 5];
    a[2] = nttc.coeffs[start + group_idx + 5 * 2];
    a[3] = nttc.coeffs[start + group_idx + 5 * 3];
    a[4] = nttc.coeffs[start + group_idx + 5 * 4];
    a[5] = nttc.coeffs[start + group_idx + 5 * 5];
    a[6] = nttc.coeffs[start + group_idx + 5 * 6];
    a[7] = nttc.coeffs[start + group_idx + 5 * 7];


    //1 基2 256
    zeta = zetas_inv_n1277[group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n1277[group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n1277[group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n1277[group_num * 4 + 3];
    GS_a(6,7,zeta);

    //2 基2 128
    zeta = zetas_inv_n1277[256 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n1277[256 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //3 基2 64
    zeta = zetas_inv_n1277[384 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 5] = a[1];
    nttc.coeffs[start + group_idx + 5 * 2] = a[2];
    nttc.coeffs[start + group_idx + 5 * 3] = a[3];
    nttc.coeffs[start + group_idx + 5 * 4] = a[4];
    nttc.coeffs[start + group_idx + 5 * 5] = a[5];
    nttc.coeffs[start + group_idx + 5 * 6] = a[6];
    nttc.coeffs[start + group_idx + 5 * 7] = a[7];

    __syncthreads();

    group_num = threadIdx.x / 40;
    group_idx = threadIdx.x % 40;
    start = group_num * 320;

    a[0] = nttc.coeffs[start + group_idx];
    a[1] = nttc.coeffs[start + group_idx + 40];
    a[2] = nttc.coeffs[start + group_idx + 40 * 2];
    a[3] = nttc.coeffs[start + group_idx + 40 * 3];
    a[4] = nttc.coeffs[start + group_idx + 40 * 4];
    a[5] = nttc.coeffs[start + group_idx + 40 * 5];
    a[6] = nttc.coeffs[start + group_idx + 40 * 6];
    a[7] = nttc.coeffs[start + group_idx + 40 * 7];

    //4 基2 32
    zeta = zetas_inv_n1277[448 + group_num * 4];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n1277[448 + group_num * 4 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n1277[448 + group_num * 4 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n1277[448 + group_num * 4 + 3];
    GS_a(6,7,zeta);

#pragma unroll
    for(int i=0;i<8;i++){
        pseudomersenne_reduce_single_n1277_cuda(a[i]);
        pseudomersenne_reduce_single_n1277_cuda(b[i]);
    }
    
    //5 基2 16
    zeta = zetas_inv_n1277[480 + group_num * 2];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n1277[480 + group_num * 2 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //6 基2 8
    zeta = zetas_inv_n1277[496 + group_num];
    GS_a(0,4,zeta);
    GS_a(1,5,zeta);
    GS_a(2,6,zeta);
    GS_a(3,7,zeta);

    nttc.coeffs[start + group_idx] = a[0];
    nttc.coeffs[start + group_idx + 40] = a[1];
    nttc.coeffs[start + group_idx + 40 * 2] = a[2];
    nttc.coeffs[start + group_idx + 40 * 3] = a[3];
    nttc.coeffs[start + group_idx + 40 * 4] = a[4];
    nttc.coeffs[start + group_idx + 40 * 5] = a[5];
    nttc.coeffs[start + group_idx + 40 * 6] = a[6];
    nttc.coeffs[start + group_idx + 40 * 7] = a[7];

    __syncthreads();
    

    a[0] = nttc.coeffs[threadIdx.x];
    a[1] = nttc.coeffs[threadIdx.x + 320];
    a[2] = nttc.coeffs[threadIdx.x + 320 * 2];
    a[3] = nttc.coeffs[threadIdx.x + 320 * 3];
    a[4] = nttc.coeffs[threadIdx.x + 320 * 4];
    a[5] = nttc.coeffs[threadIdx.x + 320 * 5];
    a[6] = nttc.coeffs[threadIdx.x + 320 * 6];
    a[7] = nttc.coeffs[threadIdx.x + 320 * 7];

    //7 基2 4
    zeta = zetas_inv_n1277[504];
    GS_a(0,1,zeta);

    zeta = zetas_inv_n1277[504 + 1];
    GS_a(2,3,zeta);

    zeta = zetas_inv_n1277[504 + 2];
    GS_a(4,5,zeta);

    zeta = zetas_inv_n1277[504 + 3];
    GS_a(6,7,zeta);

    //8 基2 2
    zeta = zetas_inv_n1277[508];
    GS_a(0,2,zeta);
    GS_a(1,3,zeta);

    zeta = zetas_inv_n1277[508 + 1];
    GS_a(4,6,zeta);
    GS_a(5,7,zeta);

    //9 基2 1
    //zeta = zetas_inv_n1277[510];
    pse_GS_a(0,4);
    pse_GS_a(1,5);
    pse_GS_a(2,6);
    pse_GS_a(3,7);

    nttc.coeffs[threadIdx.x] = a[0];
    nttc.coeffs[threadIdx.x + 320] = a[1];
    nttc.coeffs[threadIdx.x + 320 * 2] = a[2];
    nttc.coeffs[threadIdx.x + 320 * 3] = a[3];
    nttc.coeffs[threadIdx.x + 320 * 4] = a[4];
    nttc.coeffs[threadIdx.x + 320 * 5] = a[5];
    nttc.coeffs[threadIdx.x + 320 * 6] = a[6];
    nttc.coeffs[threadIdx.x + 320 * 7] = a[7];

    __syncthreads();
    
    //1 - 320
    array_c[blockIdx.x].coeffs[threadIdx.x + 1] = (montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[threadIdx.x + 1] + nttc.coeffs[threadIdx.x + 1 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 1 + FPTRU_N]))) & (FPTRU_Q2 - 1);
    //321 - 640
    array_c[blockIdx.x].coeffs[threadIdx.x + 321] = (montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[threadIdx.x + 321] + nttc.coeffs[threadIdx.x + 321 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 321 + FPTRU_N]))) & (FPTRU_Q2 - 1);

    //641 - 960
    array_c[blockIdx.x].coeffs[threadIdx.x + 641] = (montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[threadIdx.x + 641] + nttc.coeffs[threadIdx.x + 641 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 641 + FPTRU_N]))) & (FPTRU_Q2 - 1);
    
    
    //0,1276, 961-1275
    if(threadIdx.x <= 315){
        if(threadIdx.x == 315){
            array_c[blockIdx.x].coeffs[0] = (montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[0] + nttc.coeffs[FPTRU_N]))) & (FPTRU_Q2 - 1);;
            array_c[blockIdx.x].coeffs[FPTRU_N - 1] = (montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[FPTRU_N - 1] + nttc.coeffs[2 * FPTRU_N - 2]))) & (FPTRU_Q2 - 1);
        }
        else{
            array_c[blockIdx.x].coeffs[threadIdx.x + 961] = (montgomery_reduce_n1277_cuda((int64_t)FACTOR_DLXHNDNSHX * (nttc.coeffs[threadIdx.x + 961] + nttc.coeffs[threadIdx.x + 961 + FPTRU_N - 1] + nttc.coeffs[threadIdx.x + 961 + FPTRU_N]))) & (FPTRU_Q2 - 1);
        }
    }

}