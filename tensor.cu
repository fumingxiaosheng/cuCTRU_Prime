#include "hip/hip_runtime.h"
#include "kernel.h"
#include "tensor.h"
#include "poly.h"
#include <stdio.h>
#include <stdint.h>
#include <mma.h>
#include <stdlib.h> 
#include <time.h>

using namespace nvcuda;

__device__ int postive_mod(int x, int y){
    if(x >=0 ) return x;
    else return x + y;
}


#define V_BARRETT 929445

__device__ int16_t barrett_reduce_int32_t(int32_t a) //和%时间无差别
{
  int64_t t;
  t = V_BARRETT * ((int64_t) a);
  t >>= 32;
  t *= FPTRU_Q;
  //printf("%ld,%d,",t,a);
  if( a - t >= FPTRU_Q){
    return a - t - FPTRU_Q;
  }
  return a - t;
}

__global__ void wmma_ker_padding(int8_t *array_a, int8_t *array_b, int32_t *array_c) {
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, TC_X, TC_Y, TC_Z, int8_t, wmma::row_major> a_frag;
   // wklee, Read a in row major and b in column major
   wmma::fragment<wmma::matrix_b, TC_X, TC_Y, TC_Z, int8_t, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, TC_X, TC_Y, TC_Z, int32_t> c_frag;

   // Each warp compute 16 elements along index i
   uint32_t tot_warpID = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
   //每一个是41个Warp
   uint32_t warpgroup = tot_warpID / 41;
   uint32_t warpID = tot_warpID - warpgroup * 41;

   uint32_t ldA_offset, ldB_offset, row_idx, col_idx, st_offset;
   row_idx = warpID%((B_y)/TC_Z)*TC_Z;
   col_idx = warpID/((B_y)/TC_Z)*TC_X;//TODO:这里会不会有问题呢？
   st_offset = col_idx + row_idx * A_x ; 
   int8_t * a = &array_a[A_x * A_y * warpgroup];
   int8_t * b = &array_b[B_x * B_y * warpgroup];
   int32_t * c = &array_c[A_x * B_y * warpgroup];
   
   // Initialize the output to zero
   wmma::fill_fragment(c_frag, 0);  
   for (int i = 0; i < (A_y)/TC_Y; i ++)    
    {
     
      ldA_offset = col_idx*(A_y) + i*TC_Y; //TODO:明天继续改下面的，注意范围的限制
      ldB_offset = row_idx*(B_x) + i*TC_Y; 
      wmma::load_matrix_sync(a_frag, a + ldA_offset , A_y);   
      wmma::load_matrix_sync(b_frag, b + ldB_offset  , B_x);
      wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }    
    wmma::store_matrix_sync(c + st_offset , c_frag, A_x, wmma::mem_col_major);    
}

/*2024-6-26:
输入:BATCH个多项式a，将其转化为矩阵的形式
线程组织形式<<<FPTRU_N,FPTRU_N>>>*/
__global__ void init_matrix_a(poly* a,int8_t *ah,int8_t *al){
    int bid=blockIdx.x;
    int tid=threadIdx.x;
    /*ah[bid * A_y + tid] = 0;
    al[bid * A_y + tid] = 0;

    
    //__shared__ int16_t s_f[N];

    if(tid < N && bid < N){ */ //这里的分支是否是有必要的
        //s_f[tid] = f[tid];
        //__syncthreads();
        for(int num=0;num<BATCH_SIZE;num++){
            int16_t tmp = a[num].coeffs[postive_mod(bid - tid , N)];
            if (bid != 0 && tid - bid >= 0){
                tmp += a[num].coeffs[postive_mod(N - 1 - (tid -bid) , N)];
            }
            ah[num * A_x * A_y + bid * A_y + tid] = tmp >> 7;
            al[num * A_x * A_y + bid * A_y + tid] = tmp & (0x7f);
        }
    //}
}


__global__ void init_matrix_a_v3(poly* a,int8_t *ah,int8_t *al){
    int bidx=blockIdx.x;
    int tid=threadIdx.x;
    /*ah[bid * A_y + tid] = 0;
    al[bid * A_y + tid] = 0;

    
    //__shared__ int16_t s_f[N];

    if(tid < N && bid < N){ */ //这里的分支是否是有必要的
        //s_f[tid] = f[tid];
        //__syncthreads();

        __shared__ poly s_a;
        if(!tid){
            s_a = a[bidx];
        }
        __syncthreads();
        for(int bid=0;bid<FPTRU_N;bid++){
            int16_t tmp = s_a.coeffs[postive_mod(bid - tid , N)];
            if (bid != 0 && tid - bid >= 0){
                tmp += s_a.coeffs[postive_mod(N - 1 - (tid -bid) , N)];
            }
            ah[bidx * A_x * A_y + bid * A_y + tid] = tmp >> 7;
            al[bidx * A_x * A_y + bid * A_y + tid] = tmp & (0x7f);
        }
    //}
}

/*__global__ void init_matrix_a(poly* a,int8_t *ah,int8_t *al){
    int bid=blockIdx.x;
    int tid=threadIdx.x;

    for(int num=0;num<BATCH_SIZE;num++){
        int16_t tmp = a[num].coeffs[postive_mod(bid - tid , N)];
        if (bid != 0 && tid - bid >= 0){
            tmp += a[num].coeffs[postive_mod(N - 1 - (tid -bid) , N)];
        }
        ah[num * A_x * A_y + bid * A_y + tid] = tmp >> 7;
        al[num * A_x * A_y + bid * A_y + tid] = tmp & (0x7f);
    }
}*/


__global__ void init_matrix_a_v2(poly* a,int8_t *ah,int8_t *al){
    int bid=blockIdx.x;
    int tidx=threadIdx.x;
    int tidy=threadIdx.y;

    printf("%d,%d,%d,%d\n",bid,tidx,tidy,threadIdx.z);
    
    int16_t tmp = a[bid].coeffs[postive_mod(tidy - tidx , N)];
    if (tidy != 0 && tidx - tidy >= 0){
        tmp += a[bid].coeffs[postive_mod(N - 1 - (tidx -tidy) , N)];
    }
    ah[bid * A_x * A_y + tidy * A_y + tidx] = tmp >> 7;
    al[bid * A_x * A_y + tidy * A_y + tidx] = tmp & (0x7f);

}


/*2024-6-26:
将多项式b转化为**列主序**矩阵B
线程组织形式<<<BATCH_SIZE,653>>>*/
__global__ void init_matrix_b(poly * b,int8_t * B){
    int bid=blockIdx.x;//对应着多项式的下标
    int tid=threadIdx.x;
    B[bid * B_x * B_y +tid] = b[bid].coeffs[tid];
}


/*2024-6-26:
将结果的低高位组合得到相应的结果*/
__global__ void merge_hl(int32_t * r_h, int32_t * r_l , poly * p){
    int bid=blockIdx.x; //对应具体多项式
    int tid=threadIdx.x;

    int32_t res = (r_h[bid * A_x * B_y + tid] << 7) + r_l[bid * A_x * B_y+ tid]; //todo:改为64

    //r[tid] = barrett_reduce_int32_t(res);
    p[bid].coeffs[tid] = fq_freeze(res);
}

//#define looka
/*2024-6-26:
输入:多项式数组a、b、c，其中a对应着大系数多项式,b对应着小系数多项式
输出:c=a*b*/
void tensor_poly_mul(poly *c, poly *a,poly *b,hipStream_t stream){
    int32_t *C_l,*C_h;
    int8_t *A_h,*A_l,*B;

    HANDLE_ERROR(hipMalloc((void**)&C_h, A_x * B_y * sizeof(int32_t) * BATCH_SIZE));
    HANDLE_ERROR(hipMalloc((void**)&C_l, A_x * B_y * sizeof(int32_t) * BATCH_SIZE));

    HANDLE_ERROR(hipMalloc((void**)&A_h, A_x * A_y * sizeof(int8_t) * BATCH_SIZE));
    HANDLE_ERROR(hipMalloc((void**)&A_l, A_x * A_y * sizeof(int8_t) * BATCH_SIZE));

    HANDLE_ERROR(hipMalloc((void**)&B, B_x * B_y * sizeof(int8_t) * BATCH_SIZE));

    init_matrix_a<<<FPTRU_N,FPTRU_N,0,stream>>>(a,A_h,A_l);

    //init_matrix_a_v3<<<BATCH_SIZE,FPTRU_N,0,stream>>>(a,A_h,A_l);

    //dim3 threadsPerBlock(FPTRU_N, FPTRU_N);

    //init_matrix_a_v2<<<BATCH_SIZE,threadsPerBlock,0,stream>>>(a,A_h,A_l);
    //hipDeviceSynchronize();


#ifdef lookc
    int8_t h_al[A_x*A_y*BATCH_SIZE];
    int8_t h_ah[A_x*A_y*BATCH_SIZE];
    hipMemcpy(h_al,A_l, A_x * A_y * sizeof(int8_t) * BATCH_SIZE,hipMemcpyDeviceToHost);
    hipMemcpy(h_ah,A_h, A_x * A_y * sizeof(int8_t) * BATCH_SIZE,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    printf("looka\n");
    for(int num=0;num < BATCH_SIZE;num++){
        printf("num = %d\n",num);
        for(int i=0;i<A_x;i++){
        for(int j=0;j<A_y;j++){
            printf("(%d,%d)",h_ah[num * A_x * A_y + i*A_y + j] + h_al[num * A_x * A_y + i*A_y + j]);
        }
        printf("\n\n");
    }
    }
#endif

    init_matrix_b<<<BATCH_SIZE,653,0,stream>>>(b,B);


#ifdef lookb
    int8_t bb[B_x*B_y*BATCH_SIZE];
    
    hipMemcpy(bb,B, B_x * B_y * sizeof(int8_t) * BATCH_SIZE,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    printf("lookB\n");
    for(int num=0;num < BATCH_SIZE;num++){
        printf("num = %d\n",num);
        for(int i=0;i<A_x;i++){
        for(int j=0;j<B_y;j++){
            printf("%d,",bb[num * B_x * B_y + j*B_x + i]);
        }
        printf("\n\n");
    }
    }
#endif

    int blocks = 41 * (BATCH_SIZE / 8);
    int threads = 32 * 8;

    wmma_ker_padding<<<blocks,threads,0,stream>>>(A_h,B,C_h);
    wmma_ker_padding<<<blocks,threads,0,stream>>>(A_l,B,C_l);

#ifdef looka
    int32_t c_l[A_x*B_y*BATCH_SIZE];
    int32_t c_h[A_x*B_y*BATCH_SIZE];
    hipMemcpy(c_l,C_l, A_x * B_y * sizeof(int32_t) * BATCH_SIZE,hipMemcpyDeviceToHost);
    hipMemcpy(c_h,C_h, A_x * B_y * sizeof(int32_t) * BATCH_SIZE,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    printf("lookc\n");
    for(int num=0;num < BATCH_SIZE;num++){
        printf("num = %d\n",num);
        for(int i=0;i<A_x;i++){
        for(int j=0;j<B_y;j++){
            printf("(%d,%d)",c_h[num * A_x * B_y + j*A_x + i] , c_l[num * A_x * B_y + j*A_x + i]);
        }
        printf("\n\n");
    }
    }
#endif


    merge_hl<<<BATCH_SIZE,FPTRU_N,0,stream>>>(C_h,C_l,c);

    HANDLE_ERROR(hipFree(C_l));
    HANDLE_ERROR(hipFree(C_h));
    HANDLE_ERROR(hipFree(A_l));
    HANDLE_ERROR(hipFree(A_h));
    HANDLE_ERROR(hipFree(B));

}


