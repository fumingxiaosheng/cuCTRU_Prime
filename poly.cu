#include "hip/hip_runtime.h"
#include "params.h"
#include "kernel.h"
#include "cbd.h"
#include "reduce.h"
#include "poly.h"
#include "coding.h"

// #if (FPTRU_N == 653)
#include "./poly_mul_n653q/n653.h"
// #endif

#include <iostream>


__global__ void poly_sample(poly *array_a,unsigned char * array_coins){
    //printf("[%d %d]poly_sample start\n",blockIdx.x,threadIdx.x);
#if (FPTRU_N == 653) //FPTRU_N / 8 + 1个线程
    cbd3(&array_a[blockIdx.x], &array_coins[blockIdx.x * FPTRU_COIN_BYTES],0,0);
#elif (FPTRU_N == 761) //FPTRU_N / 4 个线程
    cbd2(&array_a[blockIdx.x], &array_coins[blockIdx.x * FPTRU_COIN_BYTES],0,0);
#elif (FPTRU_N == 1277) //FPTRU_N / 4 个线程
    cbd2(&array_a[blockIdx.x], &array_coins[blockIdx.x * FPTRU_COIN_BYTES],0,0);
#endif


}

__global__ void poly_double(poly *array_b,poly *array_a){
    if(threadIdx.x < FPTRU_N){
      //(array_b + blockIdx.x * sizeof(poly))->coeffs[threadIdx.x] = (array_a + blockIdx.x * sizeof(poly) )->coeffs[threadIdx.x] + (array_a + blockIdx.x * sizeof(poly))->coeffs[threadIdx.x];//注意这里得偏移一下多项式的位置

      array_b[blockIdx.x].coeffs[threadIdx.x]=array_a[blockIdx.x].coeffs[threadIdx.x] + array_a[blockIdx.x].coeffs[threadIdx.x];
    }
}

/*2024-4-25:
输入:array_a:全局内存,多项式数组，用于存储最终的sample结果
    array_cions:全局内存,种子数组,用于最为采样的随机数
    is_double:是否需要系数乘以2
    is_add1:是否需要第0个系数加上1

处理流程:在cbd的过程中，引入double的操作

思考:
    1.实验发现，在gpu代码上，额外的循环操作往往会引起时耗的增加，比如，对于cions的拷贝
    2.数据先分块，再处理，不要分块、处理，再分块处理，比如在cbd完了之后再进行double
    
时效:
    before:0.013312 ms
    after: 0.008192 ms*/
__global__ void poly_sample_and_double_v2(poly *array_a,unsigned char * array_coins,uint16_t is_double,uint16_t is_add1,int interval){
    //__shared__ poly a; //共享内存作为写入的地址
    /*
    unsigned char coins[FPTRU_COIN_BYTES];
    
    for(int i=0;i<FPTRU_COIN_BYTES;i++){ //TODO:首先读取到寄存器中
        coins[i]=array_coins[blockIdx.x * FPTRU_COIN_BYTES + i];
    }*/
#if (FPTRU_N == 653) //FPTRU_N / 8 + 1个线程
    cbd3(&array_a[blockIdx.x], &array_coins[blockIdx.x * interval],is_double,is_add1);
#elif (FPTRU_N == 761) //FPTRU_N / 4 个线程
    cbd2(&array_a[blockIdx.x],&array_coins[blockIdx.x * interval],is_double,is_add1);
    //cbd2_761(&array_a[blockIdx.x],&array_coins[blockIdx.x * interval],is_double,is_add1);
#elif (FPTRU_N == 1277) //FPTRU_N / 4 个线程
    cbd2(&array_a[blockIdx.x],&array_coins[blockIdx.x * interval],is_double,is_add1);
#endif
    //__syncthreads();//共享内存同步一下
}

/*2024-4-25:
输入:array_a:全局内存，多项式数组
    a:共享内存,多项式

处理流程:把计算得到的多项式a拷贝到全局内存array_a中

评价:单独拎出来会极大地减慢速度
*/
__device__ void poly_double_v2(poly *array_a,poly a){
#if(FPTRU_N == 653) //除以4个线程
    if(threadIdx.x == FPTRU_N/4){
        array_a[blockIdx.x].coeffs[threadIdx.x * 4]= a.coeffs[threadIdx.x * 4] + a.coeffs[threadIdx.x * 4];
    }
    else{
        array_a[blockIdx.x].coeffs[threadIdx.x * 4]= a.coeffs[threadIdx.x * 4] + a.coeffs[threadIdx.x * 4];

        array_a[blockIdx.x].coeffs[threadIdx.x * 4 + 1]= a.coeffs[threadIdx.x * 4 + 1] + a.coeffs[threadIdx.x * 4 + 1];

        array_a[blockIdx.x].coeffs[threadIdx.x * 4 + 2]= a.coeffs[threadIdx.x * 4 + 2] + a.coeffs[threadIdx.x * 4 + 2];

        array_a[blockIdx.x].coeffs[threadIdx.x * 4 + 3]= a.coeffs[threadIdx.x * 4 + 3] + a.coeffs[threadIdx.x * 4 + 3];
    }
#elif(FPTRU_N == 761) //除以8个线程
    if(threadIdx.x == FPTRU_N/8){
        array_a[blockIdx.x].coeffs[threadIdx.x * 8]= a.coeffs[threadIdx.x * 8] + a.coeffs[threadIdx.x * 8];
    }
    else{
        array_a[blockIdx.x].coeffs[threadIdx.x * 8]= a.coeffs[threadIdx.x * 8] + a.coeffs[threadIdx.x * 8];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 1]= a.coeffs[threadIdx.x * 8 + 1] + a.coeffs[threadIdx.x * 8 + 1];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 2]= a.coeffs[threadIdx.x * 8 + 2] + a.coeffs[threadIdx.x * 8 + 2];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 3]= a.coeffs[threadIdx.x * 8 + 3] + a.coeffs[threadIdx.x * 8 + 3];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 4]= a.coeffs[threadIdx.x * 8 + 4] + a.coeffs[threadIdx.x * 8 + 4];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 5]= a.coeffs[threadIdx.x * 8 + 5] + a.coeffs[threadIdx.x * 8 + 5];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 6]= a.coeffs[threadIdx.x * 8 + 6] + a.coeffs[threadIdx.x * 8 + 6];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 7]= a.coeffs[threadIdx.x * 8 + 7] + a.coeffs[threadIdx.x * 8 + 7];
    }

#elif(FPTRU_N == 1277)
    if(threadIdx.x == FPTRU_N/8){
        array_a[blockIdx.x].coeffs[threadIdx.x * 8]= a.coeffs[threadIdx.x * 8] + a.coeffs[threadIdx.x * 8];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 1]= a.coeffs[threadIdx.x * 8 + 1] + a.coeffs[threadIdx.x * 8 + 1];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 2]= a.coeffs[threadIdx.x * 8 + 2] + a.coeffs[threadIdx.x * 8 + 2];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 3]= a.coeffs[threadIdx.x * 8 + 3] + a.coeffs[threadIdx.x * 8 + 3];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 4]= a.coeffs[threadIdx.x * 8 + 4] + a.coeffs[threadIdx.x * 8 + 4];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 5]= a.coeffs[threadIdx.x * 8 + 5] + a.coeffs[threadIdx.x * 8 + 5];
    }
    else{
        array_a[blockIdx.x].coeffs[threadIdx.x * 8]= a.coeffs[threadIdx.x * 8] + a.coeffs[threadIdx.x * 8];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 1]= a.coeffs[threadIdx.x * 8 + 1] + a.coeffs[threadIdx.x * 8 + 1];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 2]= a.coeffs[threadIdx.x * 8 + 2] + a.coeffs[threadIdx.x * 8 + 2];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 3]= a.coeffs[threadIdx.x * 8 + 3] + a.coeffs[threadIdx.x * 8 + 3];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 4]= a.coeffs[threadIdx.x * 8 + 4] + a.coeffs[threadIdx.x * 8 + 4];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 5]= a.coeffs[threadIdx.x * 8 + 5] + a.coeffs[threadIdx.x * 8 + 5];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 6]= a.coeffs[threadIdx.x * 8 + 6] + a.coeffs[threadIdx.x * 8 + 6];

        array_a[blockIdx.x].coeffs[threadIdx.x * 8 + 7]= a.coeffs[threadIdx.x * 8 + 7] + a.coeffs[threadIdx.x * 8 + 7];
    }
#endif
}

__global__ void poly_copy(poly *dst, poly *src){
    if(threadIdx.x < FPTRU_N){
      dst->coeffs[threadIdx.x] = src->coeffs[threadIdx.x];
    }
}

/*2024-4-19:
输入:array_a:待处理的多项式数组

输出:存储到array_a中

处理流程:调整多项a的每个系数位于[0,FPTRU_Q-1]

线程组织形式:<<<BATCH_SIZE,1>>>
*/
__global__ void poly_fqcsubq_batch(poly *array_a){ 
    for (int i = 0; i < FPTRU_N; ++i){
        array_a[blockIdx.x].coeffs[i]=fqcsubq(array_a[blockIdx.x].coeffs[i]);
    }

}

/*2024-4-29:
输入:array_a:待处理的多项式数组

输出:存储到array_a中

线程组织形式:<<<BATCH_SIZE,FPTRU_N>>>*/
__global__ void poly_fqcsubq_batch_v2(poly *array_a){
    array_a[blockIdx.x].coeffs[threadIdx.x]=fqcsubq(array_a[blockIdx.x].coeffs[threadIdx.x]);
}

__global__ void poly_fqcsubq_batch_1277(poly *array_a){
    if(threadIdx.x == FPTRU_N / 4){
        array_a[blockIdx.x].coeffs[4 * threadIdx.x]=fqcsubq(array_a[blockIdx.x].coeffs[4 * threadIdx.x]);
    }
    else{
        array_a[blockIdx.x].coeffs[4 * threadIdx.x]=fqcsubq(array_a[blockIdx.x].coeffs[4 * threadIdx.x]);
        array_a[blockIdx.x].coeffs[4 * threadIdx.x + 1]=fqcsubq(array_a[blockIdx.x].coeffs[4 * threadIdx.x + 1]);
        array_a[blockIdx.x].coeffs[4 * threadIdx.x + 2]=fqcsubq(array_a[blockIdx.x].coeffs[4 * threadIdx.x + 2]);
        array_a[blockIdx.x].coeffs[4 * threadIdx.x + 3]=fqcsubq(array_a[blockIdx.x].coeffs[4 * threadIdx.x + 3]);
    }
}

/*2024-4-28:
线程组织形式:<<<1,FPTRU_N + 1>>>*/
__global__ void poly_inverse_once(poly * finv, poly * f){
    //printf("In poly_inverse_once\n");
    poly_inverse(finv,f);
}

/*2024-6-24:
线程组织形式:<<<1,FPTRU_N + 1>>>
*/
__global__ void poly_inv(poly * finv, poly * f){
    poly_inverse(&finv[blockIdx.x],&f[blockIdx.x]);
}


/*
输入:array_finv:设备端变量,用于存储计算得到的逆
     array_f:设备端变量,待计算逆的多项式数组
输出:存储到array_finv上

处理流程:使用蒙哥马利求逆的技巧,来同时快速计算多个多项式的逆

*/
void poly_inverse_batch_0(poly * array_finv,poly * array_f,hipStream_t stream){//TODO:加上stream
    poly *fmul;
    poly *fmul_inv;
    hipEvent_t start, stop;
    
    HANDLE_ERROR(hipMalloc(&fmul,sizeof(poly)));
    HANDLE_ERROR(hipMalloc(&fmul_inv,sizeof(poly)));


    //BEFORE_SPEED
    poly_copy<<<1,FPTRU_N>>>(fmul,&array_f[0]);

    //printf("[in poly_inverse]poly_copy done\n");
    for(int i = 1;i < BATCH_SIZE; i++ ){ //N个多项式乘法 TODO:这里能否有优化的点
        //poly_mul_q1_batch(fmul,fmul,&array_f[i],stream,1);//TODO:这里能否加上多个stream呢？这里是无法并行的，因为这是一个线性的结构->或者用之前简单的乘法的例子，但是此时ntt的算法结构就需要发生改变
        //printf("[in poly_inverse]poly_mul_q1_batch done\n");
        poly_mul_653_batch_q1_v2<<<1,N_N653/2,0,stream>>>(fmul,fmul,&array_f[i]);
    }

    //AFTER_SPEED("N_poly_MUL")

    //BEFORE_SPEED
    poly_inverse_once<<<1,1,0,stream>>>(fmul_inv,fmul);
    //AFTER_SPEED("poly_inverse_once v1")




    //BEFORE_SPEED
    for(int i=BATCH_SIZE-1;i>=2;i--){
        //poly_mul_q1_batch(&array_finv[i],fmul_inv,&array_f[i-1],stream,1); //TODO:poly_mul_q1应该直接写成函数的形式

        poly_mul_653_batch_q1_v2<<<1,N_N653/2,0,stream>>>(&array_finv[i],fmul_inv,&array_f[i-1]);
        for(int j= i-2; j >= 0; j--){
            //poly_mul_q1_batch(&array_finv[i],&array_finv[i],&array_f[j],stream,1);
            poly_mul_653_batch_q1_v2<<<1,N_N653/2,0,stream>>>(&array_finv[i],&array_finv[i],&array_f[j]);
        }
        //poly_mul_q1_batch(fmul_inv,fmul_inv,&array_f[i],stream,1);
        poly_mul_653_batch_q1_v2<<<1,N_N653/2,0,stream>>>(fmul_inv,fmul_inv,&array_f[i]);
    }

    //poly_mul_q1_batch(&array_finv[1],fmul_inv,&array_f[0],stream,1);
    poly_mul_653_batch_q1_v2<<<1,N_N653/2,0,stream>>>(&array_finv[1],fmul_inv,&array_f[0]);
    //poly_mul_q1_batch(&array_finv[0],fmul_inv,&array_f[1],stream,1);
    poly_mul_653_batch_q1_v2<<<1,N_N653/2,0,stream>>>(&array_finv[0],fmul_inv,&array_f[1]);

    //AFTER_SPEED("N_inverse_recover v1")
    hipStreamSynchronize(stream);//等待执行完成，再free掉申请的指针TODO:这里会没有并行性，导致CPU等待运行结束
    hipFree(fmul);
    hipFree(fmul_inv);

}

__device__ void device_function2(int * a){
    *a = 1;
}

__device__ void device_function3(int  a){
    a = 1;
}

__global__ void kernel2(){
    __shared__ int a;
    __shared__ int b;
    a=0;
    b=0;
    device_function2(&a);
    device_function3(b);

    printf("%d %d\n",a,b);
}

void poly_inverse_batch_0_v2(poly * array_finv,poly * array_f,hipStream_t stream){
    poly *fmul;
    poly *fmul_inv;
    hipEvent_t start, stop;

    
    HANDLE_ERROR(hipMalloc(&fmul,sizeof(poly)));
    HANDLE_ERROR(hipMalloc(&fmul_inv,sizeof(poly)));

    //printf("use kernel2 for test\n");
    //kernel2<<<1,1,0,stream>>>(); //DONE:测试表明，能够将共享内存作为参数传递

    //printf("N_poly_mul 2\n");
    //BEFORE_SPEED
    N_poly_mul<<<1,N_N653/2,0,stream>>>(fmul,array_f);//正确性验证
    //AFTER_SPEED("N_poly_mul")
    
    /*printf("N_poly_mul\n\n");
    poly *fmul_h;

    HANDLE_ERROR(hipHostAlloc((void**)&fmul_h, sizeof(poly),hipHostMallocDefault));


    HANDLE_ERROR(hipMemcpyAsync(fmul_h, fmul, sizeof(poly) , hipMemcpyDeviceToHost, stream));

    hipStreamSynchronize(stream);
    //hipDeviceSynchronize();
    for(int i=0;i<FPTRU_N;i++){
        printf("%d,",fmul_h->coeffs[i]);
    }
    printf("\n");*/

    //BEFORE_SPEED
    //printf("poly_inverse_once\n");
    poly_inverse_once<<<1,FPTRU_N + 1,0,stream>>>(fmul_inv,fmul);//正确性得以验证
    //AFTER_SPEED("poly_inverse_once v2")

    /*HANDLE_ERROR(hipMemcpyAsync(fmul_h, fmul_inv, sizeof(poly) , hipMemcpyDeviceToHost, stream));

    hipStreamSynchronize(stream);
    //hipDeviceSynchronize();
    for(int i=0;i<FPTRU_N;i++){
        printf("%d,",fmul_h->coeffs[i]);
    }
    printf("\n");*/


    //BEFORE_SPEED
    //printf("N_inverse_recover\n");
    N_inverse_recover<<<1,N_N653/2,0,stream>>>(fmul_inv,array_f,array_finv);
    //AFTER_SPEED("N_inverse_recover v2")


    hipStreamSynchronize(stream);//等待执行完成，再free掉申请的指针TODO:这里会没有并行性，导致CPU等待运行结束
    hipFree(fmul);
    hipFree(fmul_inv);
}

/*2024-6-23:
*/
void poly_inverse_batch_0_v3(poly * array_finv,poly * array_f,hipStream_t stream){
    poly *fmul;
    poly *fmul_inv;
    hipEvent_t start, stop;

    
    HANDLE_ERROR(hipMalloc(&fmul,sizeof(poly)));
    HANDLE_ERROR(hipMalloc(&fmul_inv,sizeof(poly)));

    //printf("use kernel2 for test\n");
    //kernel2<<<1,1,0,stream>>>(); //DONE:测试表明，能够将共享内存作为参数传递

    //printf("N_poly_mul 2\n");
    //BEFORE_SPEED
    N_poly_mul_v3<<<1,168,0,stream>>>(fmul,array_f);//正确性验证
    //AFTER_SPEED("N_poly_mul")
    
    /*printf("N_poly_mul\n\n");
    poly *fmul_h;

    HANDLE_ERROR(hipHostAlloc((void**)&fmul_h, sizeof(poly),hipHostMallocDefault));


    HANDLE_ERROR(hipMemcpyAsync(fmul_h, fmul, sizeof(poly) , hipMemcpyDeviceToHost, stream));

    hipStreamSynchronize(stream);
    //hipDeviceSynchronize();
    for(int i=0;i<FPTRU_N;i++){
        printf("%d,",fmul_h->coeffs[i]);
    }
    printf("\n");*/

    //BEFORE_SPEED
    //printf("poly_inverse_once\n");
    poly_inverse_once<<<1,FPTRU_N + 1,0,stream>>>(fmul_inv,fmul);//正确性得以验证
    //AFTER_SPEED("poly_inverse_once v2")

    /*HANDLE_ERROR(hipMemcpyAsync(fmul_h, fmul_inv, sizeof(poly) , hipMemcpyDeviceToHost, stream));

    hipStreamSynchronize(stream);
    //hipDeviceSynchronize();
    for(int i=0;i<FPTRU_N;i++){
        printf("%d,",fmul_h->coeffs[i]);
    }
    printf("\n");*/


    //BEFORE_SPEED
    //printf("N_inverse_recover\n");
    N_inverse_recover_v3<<<1,168,0,stream>>>(fmul_inv,array_f,array_finv);
    //AFTER_SPEED("N_inverse_recover v2")


    hipStreamSynchronize(stream);//等待执行完成，再free掉申请的指针TODO:这里会没有并行性，导致CPU等待运行结束
    hipFree(fmul);
    hipFree(fmul_inv);
}

/*2024-4-27:
输入:fmul:设备端变量，用于存储最终的相乘结果
    array_f:设备端变量,含BATCH_SIZE个多项式

处理流程:将BATCH_SIZE个多项式相乘的结果存储到fmul中

线程组织形式:<<<1,N_N653/2>>>*/
__global__ void N_poly_mul(poly * res_fmul, poly * array_f)
{   
    //printf("in N_poly_mul\n"); //TODO:这里为什么没有输出呢！
    __shared__ poly fmul;
    //__shared__ poly array_f[BATCH_SIZE];//考虑到对于全局内存的合并内存访问,这里还是将其设置为共享内存

    /*if(threadIdx.x < BATCH_SIZE){//BATCH_SIZE往往为32的整数倍，符合warp的形式，不会引起太多的时间浪费
        array_f[threadIdx.x] =src_array_f[threadIdx.x]; //DONE:这里是指针的拷贝还是完全的拷贝->这里应该是完全拷贝
    }

    __syncthreads();*///nvlink error   : Entry function '_Z10N_poly_mulP4polyS0_' uses too much shared data (0x1265a bytes, 0xc000 max) 共享内存数量限制，故删除

    /*if(threadIdx.x == 0){ //DONE:这里的array_f是正确的
        for(int j=0;j<BATCH_SIZE;j++){
            for(int i=0;i<FPTRU_N;i++){
                printf("%d,",array_f[j].coeffs[i]);
            }
            printf("\n");
        }
    }*/

    /*if(threadIdx.x == 0){
        for(int i=0;i<FPTRU_N;i++){
            printf("%d,",0);
        }
        printf("\n");
    }*/
    if(threadIdx.x < FPTRU_N){
        fmul.coeffs[threadIdx.x] = array_f[0].coeffs[threadIdx.x];
        //res_fmul->coeffs[threadIdx.x] = array_f[0].coeffs[threadIdx.x];
    }
    
    /*fmul.coeffs[2 * threadIdx.x] = array_f[0].coeffs[2 * threadIdx.x];
    fmul.coeffs[2 * threadIdx.x + 1] = array_f[0].coeffs[2 * threadIdx.x + 1];*/ //这里存在越界访问

    //TODO:改进为以下并进行速度的测量
    /*
    fmul.coeffs[threadIdx.x] = array_f[0].coeffs[threadIdx.x];
    fmul.coeffs[threadIdx.x + N_N653/2] = array_f[0].coeffs[threadIdx.x + N_N653/2];*/

    __syncthreads();
    /*if(threadIdx.x == 0){
        for(int i=0;i<FPTRU_N;i++){
            printf("%d,",fmul.coeffs[i]);
        }
        printf("\n");

        for(int i=0;i<FPTRU_N;i++){
            printf("%d,",array_f[0].coeffs[i]);
        }
        printf("\n");
    }*/

    for(int i = 1; i < BATCH_SIZE; i++){
        //poly_mul_653_batch_q1_v2_device(&fmul,&fmul,&array_f[i]);//DONE:是什么引起了共享内存的错误->共享内存取址会引发错误
        //poly_mul_653_batch_q1_v2_device(fmul,fmul,&array_f[i]);//应该是这个函数的问题

        /*if(threadIdx.x == 0){//TODO:看一下是不是poly_mul_653_batch_q1_v2_device函数改变了输入
            for(int i=0;i<FPTRU_N;i++){
                printf("%d,",array_f[i].coeffs[i]);
            }
            printf("\n");
        }*/

        /*if(threadIdx.x == 0){
            //printf("++++++++++%d++++++++++++\n",i);
            //printf("input\n");
            for(int j=0;j<FPTRU_N;j++){
                printf("%d,",fmul.coeffs[j]);
                //printf("%d,",res_fmul->coeffs[j]);
            }
            printf("\n");

            for(int j=0;j<FPTRU_N;j++){
                printf("%d,",array_f[i].coeffs[j]);
            }
            printf("\n");
        }*/

        poly_mul_653_batch_q1_v2_device(&fmul,&fmul,&array_f[i]);//TODO:两个共享内存同时做输入和输出是否存在问题，返回值赋给fmul会导致内存的访问冲突，但是此处用于验证是否是共享内存导致的问题
        //poly_mul_653_batch_q1_v2_device(res_fmul,res_fmul,&array_f[i]);
        
        __syncthreads();
        /*if(threadIdx.x == 0){
            //printf("output\n");
            for(int j=0;j<FPTRU_N;j++){
                printf("%d,",fmul.coeffs[j]);
            }
            printf("\n");
        }*/
    }

    /*if(threadIdx.x == 0){
        printf("++++++end+++++\n");
    }*/
    
    if(threadIdx.x < FPTRU_N){
        res_fmul->coeffs[threadIdx.x] = fmul.coeffs[threadIdx.x];
    }
    /*
    res_fmul->coeffs[threadIdx.x] = fmul.coeffs[threadIdx.x];
    res_fmul->coeffs[threadIdx.x + N_N653/2 ] = fmul.coeffs[threadIdx.x + N_N653/2]; //存在越界访问
    printf("[%d]%d [%d]%d\n",threadIdx.x,threadIdx.x + N_N653/2,res_fmul->coeffs[threadIdx.x],res_fmul->coeffs[threadIdx.x + N_N653/2 ]);*/
}


/*2024-6-22:
输入:fmul:设备端变量，用于存储最终的相乘结果
    array_f:设备端变量,含BATCH_SIZE个多项式

处理流程:将BATCH_SIZE个多项式相乘的结果存储到fmul中

线程组织形式:<<<1,168>>>*/
__global__ void N_poly_mul_v3(poly * res_fmul, poly * array_f)
{   

    __shared__ poly fmul;
    int tid = threadIdx.x;
    
    if(threadIdx.x == 0){
        fmul = array_f[0];
    }
    

    __syncthreads();

    for(int i = 1; i < BATCH_SIZE; i++){
        poly_mul_653_batch_q1_v3_device(&fmul,&fmul,&array_f[i]);//TODO:两个共享内存同时做输
        
        __syncthreads();
    }
    if(threadIdx.x == 0){
        res_fmul[0] = fmul;
    }
}

/*2024-4-28:
输入:src_fmul_inv:先前计算的BATCH_SIZE个多项式相乘的结果，全局变量
     array_f:BATCH_SIZE个多项式，全局变量
     array_finv:BATCH_SIZE个多项式，用于存储每个多项式求逆的结果

输出:存储到array_f中

线程组织形式:<<<1,N_N653/2>>>

处理流程:将fmul_inv和finv设置为共享内存，每计算一个，就将其写入共享内存中
*/
__global__ void N_inverse_recover(poly * src_fmul_inv,poly * array_f ,poly * array_finv){
    //printf("in N_inverse_recover\n");
    __shared__ poly fmul_inv;
    __shared__ poly finv;
    fmul_inv = src_fmul_inv[0];

    for(int i=BATCH_SIZE-1;i>=2;i--){
        poly_mul_653_batch_q1_v2_device(&finv,&fmul_inv,&array_f[i-1]);
        __syncthreads();
        for(int j=i-2;j >= 0; j--){
            poly_mul_653_batch_q1_v2_device(&finv,&finv,&array_f[j]);
            __syncthreads();
        }

        //将计算的结果赋值给全局内存
        if(threadIdx.x == 0 ){
            array_finv[i]=finv;
        }
        
        //更正fmul_inv的值
        poly_mul_653_batch_q1_v2_device(&fmul_inv,&fmul_inv,&array_f[i]);
        __syncthreads();
    }

    poly_mul_653_batch_q1_v2_device(&finv,&fmul_inv,&array_f[0]);
    __syncthreads();

    if(threadIdx.x == 0 ){
        array_finv[1]=finv;
    }

    __syncthreads();

    poly_mul_653_batch_q1_v2_device(&finv,&fmul_inv,&array_f[1]);
    __syncthreads();

    if(threadIdx.x == 0 ){
        array_finv[0]=finv;
    }
    __syncthreads();
}


/*2024-6-24:
输入:src_fmul_inv:先前计算的BATCH_SIZE个多项式相乘的结果，全局变量
     array_f:BATCH_SIZE个多项式，全局变量
     array_finv:BATCH_SIZE个多项式，用于存储每个多项式求逆的结果

输出:存储到array_f中

线程组织形式:<<<1,168>>>

处理流程:将fmul_inv和finv设置为共享内存，每计算一个，就将其写入共享内存中
*/
__global__ void N_inverse_recover_v3(poly * src_fmul_inv,poly * array_f ,poly * array_finv){
    //printf("in N_inverse_recover\n");
    __shared__ poly fmul_inv;
    __shared__ poly finv;
    if(threadIdx.x == 0){
        fmul_inv = src_fmul_inv[0];
    }
    

    for(int i=BATCH_SIZE-1;i>=2;i--){
        poly_mul_653_batch_q1_v3_device(&finv,&fmul_inv,&array_f[i-1]);
        __syncthreads();
        for(int j=i-2;j >= 0; j--){
            poly_mul_653_batch_q1_v3_device(&finv,&finv,&array_f[j]);
            __syncthreads();
        }

        //将计算的结果赋值给全局内存
        if(threadIdx.x == 0 ){
            array_finv[i]=finv;
        }
        
        //更正fmul_inv的值
        poly_mul_653_batch_q1_v3_device(&fmul_inv,&fmul_inv,&array_f[i]);
        __syncthreads();
    }

    poly_mul_653_batch_q1_v3_device(&finv,&fmul_inv,&array_f[0]);
    __syncthreads();

    if(threadIdx.x == 0 ){
        array_finv[1]=finv;
    }

    __syncthreads();

    poly_mul_653_batch_q1_v3_device(&finv,&fmul_inv,&array_f[1]);
    __syncthreads();

    if(threadIdx.x == 0 ){
        array_finv[0]=finv;
    }
    __syncthreads();
}


/*
输入:array_finv:设备端变量,用于存储计算得到的逆
     array_f:设备端变量,待计算逆的多项式数组
输出:存储到array_finv上

处理流程:每个线程处理一个poly_inverse
*/
__global__ void poly_inverse_batch_1(poly * array_finv,poly * array_f){
    poly_inverse(&array_finv[threadIdx.x * sizeof(poly)],&array_f[threadIdx.x * sizeof(poly)]);
}

/*TODO:
1.poly_inverse内部是什么样的呢?
2.poly_mul_q1应该如何进行分装
*/

__device__ void poly_inverse(poly *b, const poly *a)
{
    //printf("in poly_inverse\n");
// #if(NEW == 0)
//   rq_inverse(b->coeffs, a->coeffs);
// #endif

#if(NEW == 1)
  rq_inverse_clean(b->coeffs,a->coeffs);
  /*printf("v2 result\n");
  for(int i=0 ;i <FPTRU_N;i++){
    printf("%d,",b->coeffs[i]);
  }
  printf("\n");
  rq_inverse(b->coeffs, a->coeffs);
  printf("v1 result\n");
  for(int i=0 ;i <FPTRU_N;i++){
    printf("%d,",b->coeffs[i]);
  }
  printf("\n");*/
#endif
}

/*2024-4-8:
输入:y:存储x/m
    r:存储x mod m
    x:无符号被除数
    m:无符号除数
输出:将x/m保存在y中,将x mod m保存在r中
处理流程:将输入的32位整数 x 除以14位整数 m，并将结果保存在 y 中，余数保存在 r 中*/
__device__ void uint32_divmod_uint14(uint32_t *y, uint16_t *r, uint32_t x, uint16_t m)
{
    uint32_t w = 0x80000000;
    uint32_t qpart;
    uint32_t mask;

    w /= m;

    *y = 0;
    qpart = (x * (uint64_t)w) >> 31;
    x -= qpart * m;
    *y += qpart;

    qpart = (x * (uint64_t)w) >> 31;
    x -= qpart * m;
    *y += qpart;

    x -= m;
    *y += 1;
    mask = -(x >> 31);
    x += mask & (uint32_t)m;
    *y += mask;

    *r = x;
}

void uint32_divmod_uint14_cpu(uint32_t *y, uint16_t *r, uint32_t x, uint16_t m)
{
    uint32_t w = 0x80000000;
    uint32_t qpart;
    uint32_t mask;

    w /= m;

    *y = 0;
    qpart = (x * (uint64_t)w) >> 31;
    x -= qpart * m;
    *y += qpart;

    qpart = (x * (uint64_t)w) >> 31;
    x -= qpart * m;
    *y += qpart;

    x -= m;
    *y += 1;
    mask = -(x >> 31);
    x += mask & (uint32_t)m;
    *y += mask;

    *r = x;
}

/*2024-4-8:
输入:y:存储x/m
    r:存储x mod m（要求r一定是正数）
    x:有符号被除数
    m:无符号除数
输出:将x/m保存在y中,将x mod m保存在r中，例如x=-31,m=6,y=-6,r=5
处理流程:将输入的32位整数 x 除以14位整数 m，并将结果保存在 y 中，余数保存在 r 中*/
__device__ void int32_divmod_uint14(int32_t *y, uint16_t *r, int32_t x, uint16_t m)
{
    uint32_t uq, uq2;
    uint16_t ur, ur2;
    uint32_t mask;

    uint32_divmod_uint14(&uq, &ur, 0x80000000 + (uint32_t)x, m);
    uint32_divmod_uint14(&uq2, &ur2, 0x80000000, m);

    ur -= ur2;
    uq -= uq2;

    mask = -(uint32_t)(ur >> 15);
    ur += mask & m;
    uq += mask;
    *r = ur;
    *y = uq;
}

/*2024-4-8:
输入:x:32bit的无符号数
    m:32bit的无符号数
输出:x mod m
处理流程:调用uint32_divmod_uint14来计算x/m和x mod m,并返回结果x mod m
*/
__device__ uint16_t uint32_mod_uint14(uint32_t x, uint16_t m)
{
    uint32_t q;
    uint16_t r;
    uint32_divmod_uint14(&q, &r, x, m);
    return r;
}

uint16_t uint32_mod_uint14_cpu(uint32_t x, uint16_t m)
{
    uint32_t q;
    uint16_t r;
    uint32_divmod_uint14_cpu(&q, &r, x, m);
    return r;
}

/*2024-4-8:
输入:x:32bit的有符号数
    m:32bit的无符号数
输出:x mod m
处理流程:调用int32_divmod_uint14来计算x/m和x mod m,并返回结果x mod m*/
__device__ uint16_t int32_mod_uint14(int32_t x, uint16_t m)
{
    int32_t y;
    uint16_t r;
    int32_divmod_uint14(&y, &r, x, m);
    return r;
}
/*2024-4-8:TODO:如何通过half_q来控制相应的范围
输入:x:
输出:x mod FPTRU_Q，且控制了mod的范围*/
__device__ int16_t fq_freeze(int32_t x)
{

    const int16_t half_q = (FPTRU_Q - 1) >> 1;
    return int32_mod_uint14(x + half_q, FPTRU_Q) - half_q;
}

__device__ int int16_nonzero_mask(int16_t x)
{
    uint16_t u = x;
    uint32_t w = u;
    w = -w;
    w >>= 31;
    return -w;
}

__device__ int int16_negative_mask(int16_t x)
{
    uint16_t u = x;
    u >>= 15;
    return -(int)u;
}

#if (FPTRU_Q == 4091)

__device__ int16_t fq_inverse(int16_t a0)
{
    int16_t a = a0;
    int16_t t = a0;

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    return t;
}

#elif (FPTRU_Q == 4591)

__device__ int16_t fq_inverse(int16_t a0)
{
    int16_t a = a0;
    int16_t t = a0;

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    return t;
}

#elif (FPTRU_Q == 4621)

__device__ int16_t fq_inverse(int16_t a0)
{
    int16_t a = a0;
    int16_t t = a0;

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    return t;
}
#elif (FPTRU_Q == 7879)

__device__ int16_t fq_inverse(int16_t a0)
{
    int16_t a = a0;
    int16_t t = a0;

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    a = fq_freeze(a * (int32_t)a);
    t = fq_freeze(t * (int32_t)a);

    return t;
}
#endif

__device__ int rq_inverse(int16_t *finv, const int16_t *f)
{
    int16_t Phi[FPTRU_N + 1], F[FPTRU_N + 1], V[FPTRU_N + 1], S[FPTRU_N + 1];
    int i, loop, Delta, swap, t;
    int32_t Phi0, F0;
    int16_t scale;

    for (i = 0; i < FPTRU_N; ++i)
        Phi[i] = 0;
    Phi[0] = 1;
    Phi[FPTRU_N - 1] = Phi[FPTRU_N] = -1;

    for (i = 0; i < FPTRU_N; ++i)
        F[FPTRU_N - 1 - i] = f[i];//i会越界访问
    F[FPTRU_N] = 0;

    Delta = 1;

    for (i = 0; i < FPTRU_N + 1; ++i)
        V[i] = 0;

    for (i = 0; i < FPTRU_N + 1; ++i)
        S[i] = 0;
    S[0] = 1;

    for (loop = 0; loop < 2 * FPTRU_N - 1; ++loop)
    {
        for (i = FPTRU_N; i > 0; --i)
            V[i] = V[i - 1];
        V[0] = 0;

        swap = int16_negative_mask(-Delta) & int16_nonzero_mask(F[0]);

        for (i = 0; i < FPTRU_N + 1; ++i)
        {
            t = swap & (Phi[i] ^ F[i]);
            Phi[i] ^= t;
            F[i] ^= t;
            t = swap & (V[i] ^ S[i]);
            V[i] ^= t;
            S[i] ^= t;
        }

        Delta ^= swap & (Delta ^ -Delta);
        Delta++;

        Phi0 = Phi[0];
        F0 = F[0];

        for (i = 0; i < FPTRU_N + 1; ++i)
            F[i] = fq_freeze(Phi0 * F[i] - F0 * Phi[i]);
        for (i = 0; i < FPTRU_N; ++i)
            F[i] = F[i + 1];
        F[FPTRU_N] = 0;

        for (i = 0; i < FPTRU_N + 1; ++i)
            S[i] = fq_freeze(Phi0 * S[i] - F0 * V[i]);
    }

    scale = fq_inverse(Phi[0]);
    for (i = 0; i < FPTRU_N; ++i)
        finv[i] = fq_freeze(scale * (int32_t)V[FPTRU_N - 1 - i]);

    return int16_nonzero_mask(Delta);
}

/*2024-4-28:
输入:finv:FPTRU_N个int16_t组成的数组
    f:FPTRU_N个int16_t组成的数组

输出:存储到finv中

线程组织形式:<<<1,FPTRU_N + 1>>> 
*/
__device__ void rq_inverse_v2(int16_t *finv,const int16_t *f){
    __shared__ int16_t Phi[FPTRU_N + 1], F[FPTRU_N + 1], V[FPTRU_N + 1], S[FPTRU_N + 1];

    __shared__ int Delta;

    __shared__ int swap;
    int  loop, t;
    __shared__ int32_t Phi0, F0;
    __shared__ int16_t scale;

    int16_t tmp;

    //TODO:输入是否需要变为共享内存
    Phi[threadIdx.x] = ((!(int16_t)threadIdx.x) & 1 ) - ((!((int16_t)threadIdx.x - FPTRU_N)) & (1)) - ((!((int16_t)threadIdx.x - FPTRU_N + 1)) & (1));


    if(threadIdx.x < FPTRU_N){//存在越界的问题，需要使用条件判断语句
        F[FPTRU_N - 1 - threadIdx.x] = f[threadIdx.x];
    }
    else {
        F[threadIdx.x] = 0;
        Delta = 1;
    }


    V[threadIdx.x] = 0;
    S[threadIdx.x] = (!threadIdx.x) & 1;//线程为0时，对应的值为1

    __syncthreads();

    //下面进入循环
    for(loop = 0;loop < 2 * FPTRU_N - 1; ++loop){ //主要的问题在于循环的次数过多
        if(threadIdx.x){ //存在越界问题，需要使用条件判断语句
            tmp = V[threadIdx.x - 1];
        }
        else{
            tmp = 0;
        }
        __syncthreads();//防止读写冲突
        V[threadIdx.x] = tmp;


        if(!threadIdx.x){
            swap = int16_negative_mask(-Delta) & int16_nonzero_mask(F[0]);
        }

        __syncthreads();//等待线程0计算完swap再向后继续

        t = swap & (Phi[threadIdx.x] ^ F[threadIdx.x]);
        Phi[threadIdx.x] ^= t;
        F[threadIdx.x] ^= t;
        t = swap & (V[threadIdx.x] ^ S[threadIdx.x]);
        V[threadIdx.x] ^= t;
        S[threadIdx.x] ^= t;
        
        if(!threadIdx.x){//对Phi[0]和F[0]的选择都是由线程编号为0的线程来执行的，因此此处不需要线程的同步
            Delta ^= swap & (Delta ^ -Delta);
            Delta++;

            Phi0 = Phi[0];//这里更新了吗
            F0 = F[0];
        }

        __syncthreads();//等待线程0计算结束

        F[threadIdx.x] = fq_freeze(Phi0 * F[threadIdx.x] - F0 * Phi[threadIdx.x]);

        __syncthreads();//等待F更新结束

        if(threadIdx.x < FPTRU_N){
            tmp = F[threadIdx.x + 1];
        }
        else{
            tmp = 0;
        }

        __syncthreads();
        F[threadIdx.x] = tmp;

        S[threadIdx.x] = fq_freeze(Phi0 * S[threadIdx.x] - F0 * V[threadIdx.x]);
    }

    if(!threadIdx.x){
        scale = fq_inverse(Phi[0]);
    }

    __syncthreads();//等待线程0计算结束
    
    if(threadIdx.x < FPTRU_N){
        finv[threadIdx.x] = fq_freeze(scale * (int32_t)V[FPTRU_N - 1 - threadIdx.x]);
    }


}


/*2024-7-21:
过大的循环体导致核函数的较为耗时，可能是由于错误的内存访问模式:
循环判断的开销等造成的;
输入:finv:FPTRU_N个int16_t组成的数组
    f:FPTRU_N个int16_t组成的数组

输出:存储到finv中

线程组织形式:<<<1,FPTRU_N + 1>>> 
*/
__device__ void rq_inverse_v3(int16_t *finv,const int16_t *f){
    __shared__ int16_t Phi[FPTRU_N + 1], F[FPTRU_N + 1], V[FPTRU_N + 1], S[FPTRU_N + 1];//这里应该页不是很多吧

    __shared__ int Delta;

    //int Delta;
    __shared__ int swap;
    int  loop, t;
    __shared__ int32_t Phi0, F0;
    __shared__ int16_t scale;

    int16_t tmp;

    //TODO:输入是否需要变为共享内存
    Phi[threadIdx.x] = ((!(int16_t)threadIdx.x) & 1 ) - ((!((int16_t)threadIdx.x - FPTRU_N)) & (1)) - ((!((int16_t)threadIdx.x - FPTRU_N + 1)) & (1));


    if(threadIdx.x < FPTRU_N){//存在越界的问题，需要使用条件判断语句
        F[FPTRU_N - 1 - threadIdx.x] = f[threadIdx.x];
    }
    else{
        F[threadIdx.x] = 0;
        Delta = 1;
    }
    
    

    V[threadIdx.x] = 0;
    S[threadIdx.x] = (!threadIdx.x) & 1;//线程为0时，对应的值为1

    __syncthreads();


    //TODO:删掉
    __syncthreads();
    if(threadIdx.x == 1){
        printf("\n\nPhi\n");
        
        for(int i=0;i<=FPTRU_N;i++){
            printf("%d,",Phi[i]);
        }

        printf("\n\nS\n");
        for(int i=0;i<=FPTRU_N;i++){
            printf("%d,",S[i]);
        }

        printf("\n\nF\n");
        for(int i=0;i<=FPTRU_N;i++){
            printf("%d,",F[i]);
        }

        printf("\n\nV\n");
        for(int i=0;i<=FPTRU_N;i++){
            printf("%d,",V[i]);
        }
    } //以上构建的输入的正确性已经得到验证
    
    //下面进入循环
#pragma unroll//循环展开
    for(loop = 0;loop < 2 * FPTRU_N - 1; ++loop){ //主要的问题在于循环的次数过多
        if(threadIdx.x){ //存在越界问题，需要使用条件判断语句
            tmp = V[threadIdx.x - 1];
        }
        else{
            tmp = 0;
        }
        __syncthreads();//防止读写冲突
        V[threadIdx.x] = tmp;


        if(!threadIdx.x){
            swap = int16_negative_mask(-Delta) & int16_nonzero_mask(F[0]);
        }

        __syncthreads();//等待线程0计算完swap再向后继续
        if(threadIdx.x == 0){
            printf("ff0=%d,vv0=%d,Phi0=%d,swap=%d,S0=%d\n",F[0],V[0],Phi[0],swap,S[0]);
        }
        t = swap & (Phi[threadIdx.x] ^ F[threadIdx.x]);
        Phi[threadIdx.x] ^= t;
        F[threadIdx.x] ^= t;
        t = swap & (V[threadIdx.x] ^ S[threadIdx.x]);
        V[threadIdx.x] ^= t;
        S[threadIdx.x] ^= t;

        /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/
        __syncthreads();
        if(threadIdx.x == 1){
            printf("\n\nPhi\n");
            
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",Phi[i]);
            }

            printf("\n\nS\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",S[i]);
            }

            printf("\n\nF\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",F[i]);
            }

            printf("\n\nV\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",V[i]);
            }
        } //以上构建的输入的正确性已经得到验证

        /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/
        break;
    
        
        if(!threadIdx.x){//对Phi[0]和F[0]的选择都是由线程编号为0的线程来执行的，因此此处不需要线程的同步
            Delta ^= swap & (Delta ^ -Delta);
            Delta++;

            Phi0 = Phi[0];
            F0 = F[0];
        }

        __syncthreads();//等待线程0计算结束

        F[threadIdx.x] = fq_freeze(Phi0 * F[threadIdx.x] - F0 * Phi[threadIdx.x]);

        __syncthreads();//等待F更新结束

        if(threadIdx.x < FPTRU_N){
            tmp = F[threadIdx.x + 1];
        }
        else{
            tmp = 0;
        }

        __syncthreads();
        F[threadIdx.x] = tmp;

        S[threadIdx.x] = fq_freeze(Phi0 * S[threadIdx.x] - F0 * V[threadIdx.x]);


        /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/
        __syncthreads();
        if(threadIdx.x == 1){
            printf("\n\nPhi\n");
            
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",Phi[i]);
            }

            printf("\n\nS\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",S[i]);
            }

            printf("\n\nF\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",F[i]);
            }

            printf("\n\nV\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",V[i]);
            }
        } //以上构建的输入的正确性已经得到验证

    /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/
    }

    if(!threadIdx.x){
        scale = fq_inverse(Phi[0]);
    }

    __syncthreads();//等待线程0计算结束
    
    if(threadIdx.x < FPTRU_N){
        finv[threadIdx.x] = fq_freeze(scale * (int32_t)V[FPTRU_N - 1 - threadIdx.x]);
    }


}

//使用FPTRU_N个线程：都使用共享内存的话，会导致频繁地syncthreads,所以要使用寄存器,都维护起来F0和Phi0。每个线程负责维护threadIdx.x+1的位置，同时自己维护下标为0的
//线程组织形式为<<<BATCH_SIZE,FPTRU_N>>>
/*2024-7-23:完成正确性检验*/
__device__ void rq_inverse_v4_test(int16_t *finv,const int16_t *f){
    __shared__ int16_t V[FPTRU_N + 1],F[FPTRU_N + 1];
    int idx = threadIdx.x + 1;//向后移动1个,每一个都计算一边F0
    int16_t Phi,S,Phi0,S0;

    int Delta,swap,t;
    int16_t scale;
    int16_t tmp,ff0,vv0;

    int32_t phi0,f0;


    Phi = ((!(int16_t)idx) & 1 ) - ((!((int16_t)idx - FPTRU_N)) & (1)) - ((!((int16_t)idx - FPTRU_N + 1)) & (1));
    Phi0 = 1;

    F[FPTRU_N - 1 - threadIdx.x] = f[threadIdx.x];//TODO：F[FPTRU_N]是否要重新赋值为0
    
    //printf("f[%d]=%d,",threadIdx.x,f[threadIdx.x]);
    //根据实验结果,共享内存区域并不是初始就赋值为0,因此需要进行0赋值
    
    V[idx] = 0;

    //实验结果:分支执行比存储体冲突带来的开销更大
    /*if(idx == 1){
        F[FPTRU_N] = 0;
        V[0] = 0;
    }*/
    
    F[FPTRU_N] = 0;
    V[0] = 0;

    __syncthreads();//确保第一次关于V的读取是正确的->加了之后没有效果 TODO:正确之后把这个给删了

    //++++++++++++++++++++++++++++++++

    /*if(idx == 1) printf("F[FPTRU_N] = %d\n,V[0]=%d",F[FPTRU_N],V[0]);
    printf("V[%d]=%d\n",idx,V[idx]);*/

    Delta = 1;

    //TODO:是否需要为V赋值为0
    S = 0;
    S0 = 1;
    

    /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/
    //__shared__ int16_t test_phi[FPTRU_N + 1],test_s[FPTRU_N + 1];
    /*test_phi[idx] = Phi;
    test_s[idx] = S;

    __syncthreads();
    if(idx == 1){
        printf("\n\nPhi\n");
        printf("%d,",Phi0);
        for(int i=1;i<=FPTRU_N;i++){
            printf("%d,",test_phi[i]);
        }

        printf("\n\nS\n");
        printf("%d,",S0);
        for(int i=1;i<=FPTRU_N;i++){
            printf("%d,",test_s[i]);
        }

        printf("\n\nF\n");
        for(int i=0;i<=FPTRU_N;i++){
            printf("%d,",F[i]);
        }

        printf("\n\nV\n");
        for(int i=0;i<=FPTRU_N;i++){
            printf("%d,",V[i]);
        }
    } //以上构建的输入的正确性已经得到验证*/

    /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/

    //下面进入循环
#pragma unroll//循环展开
    for(int loop = 0;loop < 2 * FPTRU_N - 1; ++loop){ //主要的问题在于循环的次数过多
        
        tmp = V[idx-1];
        __syncthreads();//防止读写冲突
        V[idx] = tmp;
        V[0] = 0; //TODO:V[0]的值没有发生过改变 是否会造成冲突
        //__syncthreads();
        
        swap = int16_negative_mask(-Delta) & int16_nonzero_mask(F[0]);
       

        t = swap & (Phi ^ F[idx]);
        Phi ^= t;
        F[idx] ^= t;
        t = swap & (V[idx] ^ S);
        V[idx] ^= t;
        S ^= t;
        
        ff0 = F[0];
        vv0 = V[0];
        __syncthreads();

        //printf("ff0=%d,vv0=%d,Phi0=%d,swap=%d,S0=%d\n",ff0,vv0,Phi0,swap,S0);//输入是完全一样的

        t = swap & (Phi0 ^ ff0); 
        Phi0 ^= t;
        //__syncthreads();
        ff0 ^= t; //是否会造成存储体冲突
        t = swap & (vv0 ^ S0);
        //__syncthreads();
        vv0 ^= t;
        S0 ^= t; //TODO:删掉t0

        F[0] = ff0;
        V[0] = vv0;

        //TODO:先看到这里的正确性,F和V的第一个有问题

        /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/
        /*test_phi[idx] = Phi;
        test_s[idx] = S;

        __syncthreads();
        if(idx == 1){
            printf("\n正确性测试结果 loop = %d\n",loop);
            printf("\n\nPhi\n");
            printf("%d,",Phi0);
            for(int i=1;i<=FPTRU_N;i++){
                printf("%d,",test_phi[i]);
            }

            printf("\n\nS\n");
            printf("%d,",S0);
            for(int i=1;i<=FPTRU_N;i++){
                printf("%d,",test_s[i]);
            }

            printf("\n\nF\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",F[i]);
            }

            printf("\n\nV\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",V[i]);
            }
        } //以上构建的输入的正确性已经得到验证*/
        /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/

        
        Delta ^= swap & (Delta ^ -Delta);
        Delta++;

        
        phi0 = Phi0;
        f0 = F[0];
        ff0 = F[0];

        __syncthreads();//读了F[0],为了防止其余线程把这个给覆盖掉

        F[idx] = fq_freeze(phi0 * F[idx] - f0 * Phi);
        //F[0] = fq_freeze(phi0 * F[0] - f0 * Phi0); //这里会出问题，因为有些数值算了F[0]，然后又迭代进去了
        F[0] = fq_freeze(phi0 * ff0 - f0 * Phi0);
        //__syncthreads();
        tmp = F[idx];
        __syncthreads();
        F[threadIdx.x] = tmp;
        F[FPTRU_N] = 0; //TODO:存储体冲突
        

        S = fq_freeze(phi0 * S - f0 * V[idx]);
        S0 = fq_freeze(phi0 * S0 - f0 * V[0]); //S0可能是由于F的不同导致的V[0]的不同导致的

        /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/
        /*test_phi[idx] = Phi;
        test_s[idx] = S;

        __syncthreads();
        if(idx == 1){
            printf("\n正确性测试结果 loop = %d\n",loop);
            printf("\n\nPhi\n");
            printf("%d,",Phi0);
            for(int i=1;i<=FPTRU_N;i++){
                printf("%d,",test_phi[i]);
            }

            printf("\n\nS\n");
            printf("%d,",S0);
            for(int i=1;i<=FPTRU_N;i++){
                printf("%d,",test_s[i]);
            }

            printf("\n\nF\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",F[i]);
            }

            printf("\n\nV\n");
            for(int i=0;i<=FPTRU_N;i++){
                printf("%d,",V[i]);
            }
        } //以上构建的输入的正确性已经得到验证
        break;*/
        /*+++++++++++++++++++做正确性测试使用++++++++++++++++++++++++++++*/
        
    }

    scale = fq_inverse(Phi0);

    finv[threadIdx.x] = fq_freeze(scale * (int32_t)V[FPTRU_N - 1 - threadIdx.x]);

}

__device__ void rq_inverse_v4(int16_t *finv,const int16_t *f){
    __shared__ int16_t V[FPTRU_N + 1],F[FPTRU_N + 1];
    int idx = threadIdx.x + 1;//向后移动1个,每一个都计算一边F0
    int16_t Phi,S,Phi0,S0,tmpV,tmpF;

    int Delta,swap,t;
    int16_t scale;
    int16_t tmp,ff0,vv0;

    int32_t phi0,f0;


    Phi = ((!(int16_t)idx) & 1 ) - ((!((int16_t)idx - FPTRU_N)) & (1)) - ((!((int16_t)idx - FPTRU_N + 1)) & (1));
    Phi0 = 1;

    F[FPTRU_N - 1 - threadIdx.x] = f[threadIdx.x];
    
    
    V[idx] = 0;    
    F[FPTRU_N] = 0;
    V[0] = 0;

    __syncthreads();

    Delta = 1;

    S = 0;
    S0 = 1;
    

    //下面进入循环
#pragma unroll//循环展开
    //v5版本,速度为900.74
    for(int loop = 0;loop < 2 * FPTRU_N - 1; ++loop){
        
        tmpV = V[idx-1];
        //__syncthreads();//防止读写冲突
        //V[idx] = tmp;
        //V[0] = 0;
        tmpF = F[idx];
        ff0 = F[0];
        swap = int16_negative_mask(-Delta) & int16_nonzero_mask(ff0);
       
        t = swap & (Phi ^ tmpF);
        Phi ^= t;
        tmpF ^= t;
        t = swap & (tmpV ^ S);//读共享内存
        tmpV ^= t;//写共享内存
        S ^= t;
        

        //vv0 = V[0];
        vv0 = 0;//V】
        //__syncthreads();


        t = swap & (Phi0 ^ ff0); 
        Phi0 ^= t;
        ff0 ^= t;
        t = swap & (vv0 ^ S0);
        vv0 ^= t;
        S0 ^= t;

        //F[0] = ff0;
        
        
        Delta ^= swap & (Delta ^ -Delta);
        Delta++;

        
        phi0 = Phi0;
        f0 = ff0;//F[0];
        //ff0 = F[0];

        //__syncthreads();

        tmpF = fq_freeze(phi0 * tmpF - f0 * Phi);
        //F[0] = fq_freeze(phi0 * ff0 - f0 * Phi0);
        
        //tmp = F[idx];
        S = fq_freeze(phi0 * S - f0 * tmpV);
        S0 = fq_freeze(phi0 * S0 - f0 * vv0);

        __syncthreads();
        F[threadIdx.x] = tmpF;
        F[FPTRU_N] = 0; //TODO:存储体冲突

        V[idx] = tmpV;
        V[0] = vv0;
        __syncthreads();
    }

    
    /*//v4版本速度为1257.343071
    for(int loop = 0;loop < 2 * FPTRU_N - 1; ++loop){
        
        tmp = V[idx-1];
        __syncthreads();//防止读写冲突
        V[idx] = tmp;
        V[0] = 0;
        
        swap = int16_negative_mask(-Delta) & int16_nonzero_mask(F[0]);
       
        t = swap & (Phi ^ F[idx]);
        Phi ^= t;
        F[idx] ^= t;
        t = swap & (V[idx] ^ S);//读共享内存
        V[idx] ^= t;//写共享内存
        S ^= t;
        
        ff0 = F[0];
        vv0 = V[0];
        __syncthreads();


        t = swap & (Phi0 ^ ff0); 
        Phi0 ^= t;
        ff0 ^= t;
        t = swap & (vv0 ^ S0);
        vv0 ^= t;
        S0 ^= t;

        F[0] = ff0;
        V[0] = vv0;
        
        Delta ^= swap & (Delta ^ -Delta);
        Delta++;

        
        phi0 = Phi0;
        f0 = F[0];
        ff0 = F[0];

        __syncthreads();

        F[idx] = fq_freeze(phi0 * F[idx] - f0 * Phi);
        F[0] = fq_freeze(phi0 * ff0 - f0 * Phi0);
        
        tmp = F[idx];
        __syncthreads();
        F[threadIdx.x] = tmp;
        F[FPTRU_N] = 0; //TODO:存储体冲突
        

        S = fq_freeze(phi0 * S - f0 * V[idx]);
        S0 = fq_freeze(phi0 * S0 - f0 * V[0]);
    }*/

    scale = fq_inverse(Phi0);

    finv[threadIdx.x] = fq_freeze(scale * (int32_t)V[FPTRU_N - 1 - threadIdx.x]);

}

/*2024-7-24:
已经验证了正确性
TODO:为什么防止内存栅栏会存在区别*/
__device__ void rq_inverse_clean(int16_t *finv,const int16_t *f){
    __shared__ int16_t V[FPTRU_N + 1],F[FPTRU_N + 1];
    int idx = threadIdx.x + 1;//向后移动1个,每一个都计算一边F0
    int16_t Phi,S,Phi0,S0,tmpV,tmpF;

    int Delta,swap,t;
    int16_t scale;
    int16_t tmp,ff0,vv0;

    int32_t phi0,f0;


    Phi = ((!(int16_t)idx) & 1 ) - ((!((int16_t)idx - FPTRU_N)) & (1)) - ((!((int16_t)idx - FPTRU_N + 1)) & (1));
    Phi0 = 1;

    F[FPTRU_N - 1 - threadIdx.x] = f[threadIdx.x];
    
    
    V[idx] = 0;    
    F[FPTRU_N] = 0;
    V[0] = 0;

    __syncthreads();

    Delta = 1;

    S = 0;
    S0 = 1;
    

    //下面进入循环
#pragma unroll//循环展开
    for(int loop = 0;loop < 2 * FPTRU_N - 1; ++loop){
        
        tmpV = V[idx-1];
        tmpF = F[idx];
        ff0 = F[0];
        
        //__syncthreads();//全读完才能写 这有什么样的区别呢

        swap = int16_negative_mask(-Delta) & int16_nonzero_mask(ff0);
       
        t = swap & (Phi ^ tmpF);
        Phi ^= t;
        tmpF ^= t;
        t = swap & (tmpV ^ S);//读共享内存
        tmpV ^= t;//写共享内存
        S ^= t;
        
        vv0 = 0;

        t = swap & (Phi0 ^ ff0); 
        Phi0 ^= t;
        ff0 ^= t;
        t = swap & (vv0 ^ S0);
        vv0 ^= t;
        S0 ^= t;

        Delta ^= swap & (Delta ^ -Delta);
        Delta++;

        
        phi0 = Phi0;
        f0 = ff0;

        tmpF = fq_freeze(phi0 * tmpF - f0 * Phi);
        
        S = fq_freeze(phi0 * S - f0 * tmpV);
        S0 = fq_freeze(phi0 * S0 - f0 * vv0);
        __syncthreads();//全读完才能写
        F[threadIdx.x] = tmpF;
        F[FPTRU_N] = 0; //TODO:存储体冲突

        V[idx] = tmpV;
        V[0] = vv0;
        __syncthreads();//全写完才能读
        
    }


    scale = fq_inverse(Phi0);

    finv[threadIdx.x] = fq_freeze(scale * (int32_t)V[FPTRU_N - 1 - threadIdx.x]);

}

/*2024-10-15:
线程组织形式:<<<BATCH_SIZE,639>>>*/
__device__ void rq_inverse_clean_1277(int16_t *finv,const int16_t *f){
    __shared__ int16_t V[FPTRU_N + 1],F[FPTRU_N + 1],S[FPTRU_N + 1],Phi[FPTRU_N + 1];//n太大，因此可以尽情使用共享内存
    int16_t tmp1,tmp2;

    int i, loop, Delta, swap, t;
    int32_t Phi0, F0;
    int16_t scale;

    int idx = threadIdx.x;

    
    Delta = 1;
    Phi[idx * 2] = 0;
    Phi[idx * 2 + 1] = 0;

    V[idx * 2] = 0;
    V[idx * 2 + 1] = 0;

    S[idx * 2] = 0;
    S[idx * 2 + 1] = 0;

    if(idx == FPTRU_N / 2){ //0-638   
        F[FPTRU_N - 1 - idx * 2] = f[idx * 2];
    }
    else{
        F[FPTRU_N - 1 - idx * 2] = f[idx * 2];
        F[FPTRU_N - 1 - (idx * 2 + 1)] = f[idx * 2 + 1];
    }

    if(idx == FPTRU_N / 2){
        Phi[0] = 1;
        Phi[FPTRU_N - 1] = Phi[FPTRU_N] = -1;
        S[0] = 1;
        F[FPTRU_N] = 0;
    }
    
    //初始赋值无问题
   

    for(loop = 0;loop < 2* FPTRU_N - 1; ++loop){
        tmp1 = V[idx * 2];
        tmp2 = V[(idx * 2 + 1) * (idx != FPTRU_N/2)];
        __syncthreads();
        V[idx * 2 + 1] = tmp1;
        V[(idx * 2 + 2) * (idx != FPTRU_N/2)] = tmp2 * (idx != FPTRU_N/2); //减少分支语句
        __syncthreads();

        // if(idx == FPTRU_N / 2 && (loop == 1 || loop == 2)){
        //     printf("in kernel\n");
        //     for(int i =0;i<=FPTRU_N;i++) printf("%d ",V[i]);
        //     printf("\n");
        // }

        // __syncthreads();

        swap = int16_negative_mask(-Delta) & int16_nonzero_mask(F[0]);
        __syncthreads();//防止对于F[0]的读写

        t = swap & (Phi[idx * 2] ^ F[idx * 2]);
        Phi[idx * 2] ^= t;
        F[idx * 2] ^= t;
        t = swap & (V[idx * 2] ^ S[idx * 2]);
        V[idx * 2] ^= t;
        S[idx * 2] ^= t;

        t = swap & (Phi[idx * 2 + 1] ^ F[idx * 2 + 1]);
        Phi[idx * 2 + 1] ^= t;
        F[idx * 2 + 1] ^= t;
        t = swap & (V[idx * 2 + 1] ^ S[idx * 2 + 1]);
        V[idx * 2 + 1] ^= t;
        S[idx * 2 + 1] ^= t;

        Delta ^= swap & (Delta ^ -Delta);
        Delta++;

        __syncthreads();
        Phi0 = Phi[0];
        F0 = F[0];
        __syncthreads();
        F[idx * 2] = fq_freeze(Phi0 * F[idx * 2] - F0 * Phi[idx * 2]);
        F[idx * 2 + 1] = fq_freeze(Phi0 * F[idx * 2 + 1] - F0 * Phi[idx * 2 + 1]);

        S[idx * 2] = fq_freeze(Phi0 * S[idx * 2] - F0 * V[idx * 2]);
        S[idx * 2 + 1] = fq_freeze(Phi0 * S[idx * 2 + 1] - F0 * V[idx * 2 + 1]);

        __syncthreads();

        tmp1 = F[idx * 2 + 1];
        tmp2 = F[(idx * 2 + 2) * (idx != FPTRU_N/2)];

        __syncthreads();
        F[idx * 2] = tmp1;
        F[idx * 2 + 1] = tmp2 * (idx != FPTRU_N/2);
    }

    scale = fq_inverse(Phi[0]);
    finv[idx * 2] = fq_freeze(scale * (int32_t)V[FPTRU_N - 1 - idx * 2]);
    if(idx != FPTRU_N/2) finv[idx * 2 + 1] = fq_freeze(scale * (int32_t)V[FPTRU_N - 1 - (idx * 2 + 1)]);
    
}



__device__ void poly_inverse_1277(poly *b, const poly *a)
{
  rq_inverse_clean_1277(b->coeffs,a->coeffs);
}

__global__ void poly_inv_1277(poly * finv, poly * f){
    poly_inverse_1277(&finv[blockIdx.x],&f[blockIdx.x]);
}

/*2024-7-8
输入:c:待输出的多项式
    sigma:待加上的多项式
    msg:待encode的明文
输出:多项式c

概述:完成一次poly_encode_compress操作,线程组织形式为<<<1,FPTRU_N>>>*/
__device__ void poly_encode_compress_1(poly * c, poly * sigma, unsigned char * msg){
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    if(tid >= 16 * FPTRU_MSGBYTES){
        int16_t t = (int32_t)((fqcsubq(sigma->coeffs[tid]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
        c->coeffs[tid] = t & (FPTRU_Q2 - 1);
    }
    else{
        int i = tid >> 3;//tid / 8;
        int j = tid & 7 ;//tid % 8;
        int k = i >> 1;//i / 2;

        uint8_t tmp = (msg[k] >> ((i %2 ) << 2)) & 0xF;
        uint8_t mh = encode_e8(tmp);

        int16_t mask = -(int16_t)((mh >> j) & 1);
        int16_t t = (int32_t)((fqcsubq(sigma->coeffs[tid]) << FPTRU_LOGQ2) + (FPTRU_Q >> 1)) / FPTRU_Q;
        t = t + (mask & (FPTRU_Q2 >> 1));//使得大于16 * FPTRU_MSGBYTES 为0
        c->coeffs[tid] = t & (FPTRU_Q2 - 1);
    }
}

__global__ void poly_fqcsubq_encode_compress_batch(poly * c, poly * sigma, unsigned char * msg, int interval){
    poly_encode_compress_1(&c[blockIdx.x],&sigma[blockIdx.x],&msg[interval * blockIdx.x]);
}

__device__ int32_t sqr(int32_t x)
{
    return x * x;
}

__device__ uint32_t abs_q2(uint32_t x)
{
    uint32_t mask = -(((FPTRU_Q2 - (x << 1)) >> 31) & 1);
    return (mask & (FPTRU_Q2 - x)) | ((~mask) & x);
}

__device__ uint32_t const_abs(int32_t x)
{
    uint32_t mask = x >> 31;
    return (x ^ mask) - mask;
}

__device__ uint8_t decode_d8_00(uint32_t *cost, uint32_t tmp_cost[8][2])
{
    uint8_t m[2] = {0, 0}, xor_sum = 0, res;
    uint32_t min_diff = ~0U >> 2, r;
    uint32_t min_i = 0;
    int i;

    *cost = 0;
    for (i = 0; i < 4; i++)
    {
        uint32_t c[2];
        c[0] = tmp_cost[i << 1][0] + tmp_cost[i << 1 | 1][0];
        c[1] = tmp_cost[i << 1][1] + tmp_cost[i << 1 | 1][1];
        r = ((c[1] - c[0]) >> 31) & 1;
        m[0] |= r << i;
        xor_sum ^= r;

        uint32_t tmp = ((-(r ^ 1)) & (uint32_t)c[0]) ^ ((-(r & 1)) & (uint32_t)c[1]);
        uint32_t tmp_xor = ((-(r & 1)) & (uint32_t)c[0]) ^ ((-(r ^ 1)) & (uint32_t)c[1]);

        *cost += tmp;
        uint32_t diff = tmp_xor - tmp;

        r = ((diff - min_diff) >> 31) & 1;
        min_diff = ((-r) & diff) | ((-(r ^ 1)) & min_diff);
        min_i = ((-r) & i) | ((-(r ^ 1)) & min_i);
    }
    m[1] = m[0] ^ (1 << min_i);
    res = ((-(xor_sum ^ 1)) & (uint32_t)m[0]) ^ ((-(xor_sum & 1)) & (uint32_t)m[1]);
    *cost += (-xor_sum) & min_diff;
    return res;
}

__device__ uint8_t decode_d8_10(uint32_t *cost, uint32_t tmp_cost[8][2])
{
    uint8_t m[2] = {0, 0}, xor_sum = 0, res;
    uint32_t min_diff = ~0U >> 2, r;
    uint32_t min_i = 0;
    int i;

    *cost = 0;
    for (i = 0; i < 4; i++)
    {
        uint32_t c[2];
        c[0] = tmp_cost[i << 1][1] + tmp_cost[i << 1 | 1][0];
        c[1] = tmp_cost[i << 1][0] + tmp_cost[i << 1 | 1][1];
        r = ((c[1] - c[0]) >> 31) & 1;
        m[0] |= r << i;
        xor_sum ^= r;

        uint32_t tmp = ((-(r ^ 1)) & (uint32_t)c[0]) ^ ((-(r & 1)) & (uint32_t)c[1]);
        uint32_t tmp_xor = ((-(r & 1)) & (uint32_t)c[0]) ^ ((-(r ^ 1)) & (uint32_t)c[1]);

        *cost += tmp;
        uint32_t diff = tmp_xor - tmp;
        r = ((diff - min_diff) >> 31) & 1;
        min_diff = ((-r) & diff) | ((-(r ^ 1)) & min_diff);
        min_i = ((-r) & i) | ((-(r ^ 1)) & min_i);
    }
    m[1] = m[0] ^ (1 << min_i);
    res = ((-(xor_sum ^ 1)) & (uint32_t)m[0]) ^ ((-(xor_sum & 1)) & (uint32_t)m[1]);
    *cost += (-xor_sum) & min_diff;
    return res;
}

__device__ uint8_t decode_e8(uint32_t vec[8])
{
    uint32_t cost[2], r;
    uint8_t m[2], res;
    uint32_t tmp_cost[8][2];
    int i;

    for (i = 0; i < 8; i++)
    {
        tmp_cost[i][0] = sqr(abs_q2(vec[i]));
        tmp_cost[i][1] = sqr(const_abs(vec[i] - (FPTRU_Q2 >> 1)));
    }

    m[0] = decode_d8_00(cost + 0, tmp_cost);
    m[1] = decode_d8_10(cost + 1, tmp_cost);
    r = ((cost[1] - cost[0]) >> 31) & 1;

    res = ((-(r ^ 1)) & (uint32_t)m[0]) ^ ((-(r & 1)) & (uint32_t)m[1]);
    res = ((((res ^ (res << 1)) & 0x3) | ((res >> 1) & 4)) << 1) | r;

    return res;
}

__device__ void poly_decode(unsigned char * m, poly *mp){
    int idx = threadIdx.x;
    uint32_t tmp_mp[8];
    
    uint8_t msg_l,msg_h;

    tmp_mp[0] = (uint32_t)mp->coeffs[16 * idx];
    tmp_mp[1] = (uint32_t)mp->coeffs[16 * idx + 1];
    tmp_mp[2] = (uint32_t)mp->coeffs[16 * idx + 2];
    tmp_mp[3] = (uint32_t)mp->coeffs[16 * idx + 3];
    tmp_mp[4] = (uint32_t)mp->coeffs[16 * idx + 4];
    tmp_mp[5] = (uint32_t)mp->coeffs[16 * idx + 5];
    tmp_mp[6] = (uint32_t)mp->coeffs[16 * idx + 6];
    tmp_mp[7] = (uint32_t)mp->coeffs[16 * idx + 7];

    msg_l = decode_e8(tmp_mp);

    tmp_mp[0] = (uint32_t)mp->coeffs[16 * idx + 8];
    tmp_mp[1] = (uint32_t)mp->coeffs[16 * idx + 9];
    tmp_mp[2] = (uint32_t)mp->coeffs[16 * idx + 10];
    tmp_mp[3] = (uint32_t)mp->coeffs[16 * idx + 11];
    tmp_mp[4] = (uint32_t)mp->coeffs[16 * idx + 12];
    tmp_mp[5] = (uint32_t)mp->coeffs[16 * idx + 13];
    tmp_mp[6] = (uint32_t)mp->coeffs[16 * idx + 14];
    tmp_mp[7] = (uint32_t)mp->coeffs[16 * idx + 15];

    msg_h = decode_e8(tmp_mp);

    m[threadIdx.x] = (msg_h << 4) + msg_l;
}

__global__ void poly_decode_batch(unsigned char *m, poly *mp){
    poly_decode(&m[blockIdx.x * (FPTRU_PREFIXHASHBYTES + FPTRU_MSGBYTES)],&mp[blockIdx.x]);
}


